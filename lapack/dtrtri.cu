#include "hip/hip_runtime.h"
#include "blas.h"

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

template <CBlasUplo uplo, CBlasDiag diag, unsigned int bx>
__global__ void dtrti2(double * A, int * info, int lda, int n) {
  // info parameter cached in shared memory for fast access by all threads in the block
  __shared__ int sinfo;

  // thread 0 is the only thread to write to info in shared or global memory
  if (threadIdx.x == 0)
    *info = sinfo = 0;  // initialise info to zero and cache

  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 32 threads are needed but this would
   * use 8192 bytes (32 * 32 * sizeof(double)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 4224 bytes((32 * (32 + 1)) / 2 * sizeof(double))
   * of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ double a[(bx * (bx + 1)) / 2];
  __shared__ double ajj;

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = 0; j < n; j++) {
      double temp;
      // Read current column into registers
      if (threadIdx.x <= j)
        temp = a[upper(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (temp == 0.0) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[upper(threadIdx.x, j)] = 1.0 / temp;
          ajj = -a[upper(threadIdx.x, j)];
        }
        else
          ajj = -1.0;
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x < j) {
        if (diag == CBlasNonUnit)
          temp *= a[upper(threadIdx.x, threadIdx.x)];
        for (int k = threadIdx.x + 1; k < j; k++)
          temp += a[upper(threadIdx.x, k)] * a[upper(k, j)];
      }

      __syncthreads();

      if (threadIdx.x < j)
        a[upper(threadIdx.x, j)] = temp * ajj;

      __syncthreads();
    }

    // Write the upper triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j)
        A[j * lda + threadIdx.x] = a[upper(threadIdx.x, j)];
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = n - 1; j >= 0; j--) {
      double temp;
      // Read current column into registers
      if (threadIdx.x >= j)
        temp = a[lower<bx>(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (temp == 0.0) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[lower<bx>(threadIdx.x, j)] = 1.0 / temp;
          ajj = -a[lower<bx>(threadIdx.x, j)];
        }
        else
          ajj = -1.0;
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x > j) {
        if (diag == CBlasNonUnit)
          temp *= a[lower<bx>(threadIdx.x, threadIdx.x)];
        for (int k = j + 1; k < threadIdx.x; k++)
          temp += a[lower<bx>(threadIdx.x, k)] * a[lower<bx>(k, j)];
      }

      __syncthreads();

      if (threadIdx.x > j)
        a[lower<bx>(threadIdx.x, j)] = temp * ajj;

      __syncthreads();
    }

    // Write the lower triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        A[j * lda + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template __global__ void dtrti2<CBlasUpper, CBlasUnit, 32>(double *, int *, int, int);
template __global__ void dtrti2<CBlasUpper, CBlasNonUnit, 32>(double *, int *, int, int);
template __global__ void dtrti2<CBlasLower, CBlasUnit, 32>(double *, int *, int, int);
template __global__ void dtrti2<CBlasLower, CBlasNonUnit, 32>(double *, int *, int, int);

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void dtrti2_(const char *, const char *, const int *, double *, const int *, int *);
static inline void dtrti2(CBlasUplo uplo, CBlasDiag diag, int n, double * A, int lda, int * info) {
  if (uplo == CBlasUpper) {
    if (diag == CBlasNonUnit)
      dtrti2<CBlasUpper, CBlasNonUnit, 32><<<1,32>>>(A, info, lda, n);
    else
      dtrti2<CBlasUpper, CBlasUnit, 32><<<1,32>>>(A, info, lda, n);
  }
  else {
    if (diag == CBlasNonUnit)
      dtrti2<CBlasLower, CBlasNonUnit, 32><<<1,32>>>(A, info, lda, n);
    else
      dtrti2<CBlasLower, CBlasUnit, 32><<<1,32>>>(A, info, lda, n);
  }
}

static int cond(int, double, double *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  CBlasDiag diag;
  int n;

  if (argc != 4) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <diag>  is 'U' or 'u' for CBlasUnit or 'N' or 'n' for CBlasNonUnit\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  char d;
  if (sscanf(argv[2], "%c", &d) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[2]);
    return 2;
  }
  switch (d) {
    case 'u': case 'U': diag = CBlasUnit; break;
    case 'n': case 'N': diag = CBlasNonUnit; break;
    default: fprintf(stderr, "Unknown diag '%c'\n", u); return 1;
  }

  if (sscanf(argv[3], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[3]);
    return 3;
  }

  double * A, * dA, * refA;
  size_t lda = (n + 1u) & ~1u, dlda;
  int * dinfo;
  if ((A = (double *)malloc(lda * n * sizeof(double))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (double *)malloc(lda * n * sizeof(double))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(double), n));
  CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
  dlda /= sizeof(double);

  cond(n, 2.0, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(double));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(double), A, lda * sizeof(double), n * sizeof(double), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  int info = 0, refInfo = 0;
  dtrti2_((const char *)&uplo, (const char *)&diag, &n, refA, (const int *)&lda, &refInfo);
  dtrti2(uplo, diag, n, dA, dlda, dinfo);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(double), dA, dlda * sizeof(double), n * sizeof(double), n, hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", refA[j * lda + i]);
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  double error = 0.0;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      double diff = fabs(refA[j * lda + i] - A[j * lda + i]);
      if (diff > error)
        error = diff;
    }
  }

  fprintf(stdout, "Info = %d, refInfo = %d, Error = %6.3e\n", info, refInfo, error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));
  CUDA_ERROR_CHECK(hipFree(dinfo));

  return info;
}

static int cond(int n, double c, double * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  double * u, * v, * w;
  size_t offset = (n + 1u) & ~1u;

  if ((u = (double *)malloc(3 * offset * sizeof(double))) == NULL)
    return 1;

  v = &u[offset];
  w = &v[offset];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] = 0.0;
  }

  A[0] = 1.0;
  A[lda + 1] = c;
  for (int j = 2; j < n; j++)
    A[j * lda + j] = ((double) rand() / (double)RAND_MAX) * (c - 1.0) + 1.0;

  double t = 0.0, s = 0.0;
  for (int j = 0; j < n; j++) {
    // u is a random vector
    u[j] = (double)rand() / (double)RAND_MAX;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += u[j] * u[j];
    // s = t^2 u'v / 2
    s += u[j] * v[j];
  }
  t = 2.0 / t;
  s = t * t * s / 2.0;

  // w = tv - su
  for (int j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * w[j] + w[i] * u[j];
  }

  free(u);

  return 0;
}
