#include "hip/hip_runtime.h"
#include "blas.h"

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

template <CBlasUplo uplo, unsigned int bx>
__global__ void slauu2(float * A, int lda, int n) {
  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 64 threads are needed but this would
   * use all 16384 bytes (64 * 64 * sizeof(float)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 8320 bytes((64 * (64 + 1)) / 2 * sizeof(float))
   * of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ float a[(bx * (bx + 1)) / 2];

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the cholesky decomposition
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j) {
        float temp = a[upper(threadIdx.x, j)] * a[upper(j, j)];
        for (int k = j + 1; k < n; k++)
          temp += a[upper(threadIdx.x, k)] * a[upper(j, k)];
        A[j * lda + threadIdx.x] = temp;
      }
      __syncthreads();
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the cholesky decomposition
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int i = 0; i < n; i++) {
      if (threadIdx.x <= i) {
        float temp = a[lower<bx>(i, threadIdx.x)] * a[lower<bx>(i, i)];
        for (int k = i + 1; k < n; k++)
          temp += a[lower<bx>(k, i)] * a[lower<bx>(k, threadIdx.x)];
        a[lower<bx>(i, threadIdx.x)] = temp;
      }
      __syncthreads();
    }

    // Write the lower triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        A[j * lda + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template __global__ void slauu2<CBlasUpper, 64>(float *, int, int);
template __global__ void slauu2<CBlasLower, 64>(float *, int, int);

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void slauu2_(const char *, const int *, float *, const int *, int *);
static inline void slauu2(CBlasUplo uplo, int n, float * A, int lda) {
  if (uplo == CBlasUpper)
    slauu2<CBlasUpper, 64><<<1,64>>>(A, lda, n);
  else
    slauu2<CBlasLower, 64><<<1,64>>>(A, lda, n);
}

static int cond(int, float, float *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  int n;

  if (argc != 3) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  if (sscanf(argv[2], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[2]);
    return 2;
  }

  float * A, * dA, * refA;
  size_t lda = (n + 3u) & ~3u, dlda;
  if ((A = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(float), n));
  dlda /= sizeof(float);

  cond(n, 2.0f, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(float));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(float), A, lda * sizeof(float), n * sizeof(float), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  int refInfo;
  slauu2_((const char *)&uplo, &n, refA, (const int *)&lda, &refInfo);
  slauu2(uplo, n, dA, dlda);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(float), dA, dlda * sizeof(float), n * sizeof(float), n, hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", refA[j * lda + i]);
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  float error = 0.0f;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      float diff = fabsf(refA[j * lda + i] - A[j * lda + i]);
      if (diff > error)
        error = diff;
    }
  }

  fprintf(stdout, "refInfo = %d, Error = %6.3e\n", refInfo, error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));

  return refInfo;
}

static int cond(int n, float c, float * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  float * u, * v, * w;
  size_t offset = (n + 3u) & ~3u;

  if ((u = (float *)malloc(3 * offset * sizeof(float))) == NULL)
    return 1;

  v = &u[offset];
  w = &v[offset];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] = 0.0f;
  }

  A[0] = 1.0f;
  A[lda + 1] = c;
  for (int j = 2; j < n; j++)
    A[j * lda + j] = ((float) rand() / (float)RAND_MAX) * (c - 1.0f) + 1.0f;

  float t = 0.0f, s = 0.0f;
  for (int j = 0; j < n; j++) {
    // u is a random vector
    u[j] = (float)rand() / (float)RAND_MAX;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += u[j] * u[j];
    // s = t^2 u'v / 2
    s += u[j] * v[j];
  }
  t = 2.0f / t;
  s = t * t * s / 2.0f;

  // w = tv - su
  for (int j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * w[j] + w[i] * u[j];
  }

  free(u);

  return 0;
}
