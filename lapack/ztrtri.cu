#include "hip/hip_runtime.h"
#include "blas.h"
#include <hip/hip_complex.h>

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

template <CBlasUplo uplo, CBlasDiag diag, unsigned int bx>
__global__ void ztrti2(hipDoubleComplex * A, int * info, int lda, int n) {
  // info parameter cached in shared memory for fast access by all threads in the block
  __shared__ int sinfo;

  // thread 0 is the only thread to write to info in shared or global memory
  if (threadIdx.x == 0)
    *info = sinfo = 0;  // initialise info to zero and cache

  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 16 threads are needed but this would
   * use 4096 bytes (16 * 16 * sizeof(hipDoubleComplex)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 2176 bytes((16 * (16 + 1)) / 2 *
   * sizeof(hipDoubleComplex)) of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ hipDoubleComplex a[(bx * (bx + 1)) / 2];
  __shared__ hipDoubleComplex ajj;

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = 0; j < n; j++) {
      hipDoubleComplex temp;
      // Read current column into registers
      if (threadIdx.x <= j)
        temp = a[upper(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (hipCreal(temp) == 0.0 && hipCimag(temp) == 0.0) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[upper(threadIdx.x, j)] = hipCdiv(make_hipDoubleComplex(1.0, 0.0), temp);
          ajj = make_hipDoubleComplex(-hipCreal(a[upper(threadIdx.x, j)]), -hipCimag(a[upper(threadIdx.x, j)]));
        }
        else
          ajj = make_hipDoubleComplex(-1.0, 0.0);
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x < j) {
        if (diag == CBlasNonUnit)
          temp = hipCmul(temp, a[upper(threadIdx.x, threadIdx.x)]);
        for (int k = threadIdx.x + 1; k < j; k++)
          temp = hipCfma(a[upper(threadIdx.x, k)], a[upper(k, j)], temp);
      }

      __syncthreads();

      if (threadIdx.x < j)
        a[upper(threadIdx.x, j)] = hipCmul(temp, ajj);

      __syncthreads();
    }

    // Write the upper triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j)
        A[j * lda + threadIdx.x] = a[upper(threadIdx.x, j)];
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = n - 1; j >= 0; j--) {
      hipDoubleComplex temp;
      // Read current column into registers
      if (threadIdx.x >= j)
        temp = a[lower<bx>(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (hipCreal(temp) == 0.0 && hipCimag(temp) == 0.0) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[lower<bx>(threadIdx.x, j)] = hipCdiv(make_hipDoubleComplex(1.0, 0.0), temp);
          ajj = make_hipDoubleComplex(-hipCreal(a[lower<bx>(threadIdx.x, j)]), -hipCimag(a[lower<bx>(threadIdx.x, j)]));
        }
        else
          ajj = make_hipDoubleComplex(-1.0, 0.0);
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x > j) {
        if (diag == CBlasNonUnit)
          temp = hipCmul(temp, a[lower<bx>(threadIdx.x, threadIdx.x)]);
        for (int k = j + 1; k < threadIdx.x; k++)
          temp = hipCfma(a[lower<bx>(threadIdx.x, k)], a[lower<bx>(k, j)], temp);
      }

      __syncthreads();

      if (threadIdx.x > j)
        a[lower<bx>(threadIdx.x, j)] = hipCmul(temp, ajj);

      __syncthreads();
    }

    // Write the lower triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        A[j * lda + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template __global__ void ztrti2<CBlasUpper, CBlasUnit, 16>(hipDoubleComplex *, int *, int, int);
template __global__ void ztrti2<CBlasUpper, CBlasNonUnit, 16>(hipDoubleComplex *, int *, int, int);
template __global__ void ztrti2<CBlasLower, CBlasUnit, 16>(hipDoubleComplex *, int *, int, int);
template __global__ void ztrti2<CBlasLower, CBlasNonUnit, 16>(hipDoubleComplex *, int *, int, int);

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void ztrti2_(const char *, const char *, const int *, void *, const int *, int *);
static inline void ztrti2(CBlasUplo uplo, CBlasDiag diag, int n, hipDoubleComplex * A, int lda, int * info) {
  if (uplo == CBlasUpper) {
    if (diag == CBlasNonUnit)
      ztrti2<CBlasUpper, CBlasNonUnit, 16><<<1,16>>>(A, info, lda, n);
    else
      ztrti2<CBlasUpper, CBlasUnit, 16><<<1,16>>>(A, info, lda, n);
  }
  else {
    if (diag == CBlasNonUnit)
      ztrti2<CBlasLower, CBlasNonUnit, 16><<<1,16>>>(A, info, lda, n);
    else
      ztrti2<CBlasLower, CBlasUnit, 16><<<1,16>>>(A, info, lda, n);
  }
}

static int ccond(int, double, double complex *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  CBlasDiag diag;
  int n;

  if (argc != 4) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <diag>  is 'U' or 'u' for CBlasUnit or 'N' or 'n' for CBlasNonUnit\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  char d;
  if (sscanf(argv[2], "%c", &d) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[2]);
    return 2;
  }
  switch (d) {
    case 'u': case 'U': diag = CBlasUnit; break;
    case 'n': case 'N': diag = CBlasNonUnit; break;
    default: fprintf(stderr, "Unknown diag '%c'\n", u); return 1;
  }

  if (sscanf(argv[3], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[3]);
    return 3;
  }

  double complex * A, * refA;
  hipDoubleComplex * dA;
  size_t lda = n, dlda;
  int * dinfo;
  if ((A = (double complex *)malloc(lda * n * sizeof(double complex))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (double complex *)malloc(lda * n * sizeof(double complex))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(hipDoubleComplex), n));
  CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
  dlda /= sizeof(hipDoubleComplex);

  ccond(n, 2.0, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(double complex));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(hipDoubleComplex), A, lda * sizeof(double complex), n * sizeof(hipDoubleComplex), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", creal(A[j * lda + i]), cimag(A[j * lda + i]));
    fprintf(stderr, "\n");
  }

  int info = 0, refInfo = 0;
  ztrti2_((const char *)&uplo, (const char *)&diag, &n, refA, (const int *)&lda, &refInfo);
  ztrti2(uplo, diag, n, dA, dlda, dinfo);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(double complex), dA, dlda * sizeof(hipDoubleComplex), n * sizeof(hipDoubleComplex), n, hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", creal(refA[j * lda + i]), cimag(refA[j * lda + i]));
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", creal(A[j * lda + i]), cimag(A[j * lda + i]));
    fprintf(stderr, "\n");
  }

  double real_error = 0.0, imag_error = 0.0;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      double diff = fabs(creal(refA[j * lda + i]) - creal(A[j * lda + i]));
      if (diff > real_error)
        real_error = diff;
      diff = fabs(cimag(refA[j * lda + i]) - cimag(A[j * lda + i]));
      if (diff > imag_error)
        imag_error = diff;
    }
  }

  fprintf(stdout, "Info = %d, refInfo = %d, Error = %6.3e + %6.3ei\n", info, refInfo, real_error, imag_error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));
  CUDA_ERROR_CHECK(hipFree(dinfo));

  return info;
}

static int ccond(int n, double c, double complex * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  double complex * u, * v, * w;

  if ((u = (double complex *)malloc(3 * n * sizeof(double complex))) == NULL)
    return 1;

  v = &u[n];
  w = &v[n];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (size_t j = 0; j < n; j++) {
    for (size_t i = 0; i < n; i++)
      A[j * lda + i] = 0.0 + 0.0 * I;
  }

  A[0] = 1.0 + 0.0 * I;
  A[lda + 1] = c + 0.0 * I;
  for (size_t j = 2; j < n; j++)
    A[j * lda + j] = ((double) rand() / (double)RAND_MAX) * (c - 1.0) + 1.0 + 0.0 * I;

  double t = 0.0;
  double complex s = 0.0 + 0.0 * I;
  for (size_t j = 0; j < n; j++) {
    // u is a complex precision random vector
    u[j] = ((double)rand() / (double)RAND_MAX) + ((double)rand() / (double)RAND_MAX) * I;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += creal(conj(u[j]) * u[j]);
    // s = t^2 u'v / 2
    s += conj(u[j]) * v[j];
  }
  t = 2.0 / t;
  s = t * t * s / (2.0 + 0.0 * I);

  // w = tv - su
  for (size_t j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (size_t j = 0; j < n; j++) {
    for (size_t i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * conj(w[j]) + w[i] * conj(u[j]);
  }

  free(u);

  return 0;
}
