#include "hip/hip_runtime.h"
#include "../blas/strmm.cu"

#ifndef __UPLO_H
#define __UPLO_H

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

#endif

/**
 * In-place triangular packed unblocked triangular inverse device function.
 *
 * @param n     the size of the matrix
 * @param A     the matrix stored using upper or lower triangular packed storage
 *                mode
 * @param info  info
 *
 * A and info are expected to be in shared memory but may still work (slower) if
 * in global memory.
 */
template <CBlasUplo uplo, CBlasDiag diag, unsigned int nb, unsigned int bx>
__device__ void stpti2(int n, float * __restrict__ A, int * __restrict__ info) {
  const int i = threadIdx.y * bx + threadIdx.x;

  // thread 0 is the only thread to write to info
  if (i == 0)
    *info = 0;  // initialise info to zero

  // Copy of diagonal element in shared memory prior to updating
  __shared__ float ajj;

  if (uplo == CBlasUpper) {
    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = 0; j < n; j++) {
      float temp;
      // Read current column into registers
      if (i <= j)
        temp = A[upper(i, j)];

      // Thread j calculates the diagonal element
      if (i == j) {
        if (diag == CBlasNonUnit) {
          if (temp == 0.0f)
            *info = j + 1;
          else {
            A[upper(j, j)] = 1.0f / temp;
            ajj = -A[upper(j, j)];
          }
        }
        else
          ajj = -1.0f;
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (*info != 0)
        return;

      if (i < j) {
        if (diag == CBlasNonUnit)
          temp *= A[upper(i, i)];
        for (int k = i + 1; k < j; k++)
          temp += A[upper(i, k)] * A[upper(k, j)];
      }

      __syncthreads();

      if (i < j)
        A[upper(i, j)] = temp * ajj;

      __syncthreads();
    }
  }
  else {
    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also avoids bank
    // conflicts.
    if (i < n) {
      for (int j = n - 1; j >= 0; j--) {
        float temp;
        // Read current column into registers
        if (i >= j)
          temp = A[lower<nb>(i, j)];

        // Thread j calculates the diagonal element
        if (i == j) {
          if (diag == CBlasNonUnit) {
            if (temp == 0.0f)
              *info = j + 1;
            else {
              A[lower<nb>(j, j)] = 1.0f / temp;
              ajj = -A[lower<nb>(j, j)];
            }
          }
          else
            ajj = -1.0f;
        }

        __syncthreads();

        // If info != 0 return (matrix is singular)
        if (*info != 0)
          return;

        if (i > j) {
          if (diag == CBlasNonUnit)
            temp *= A[lower<nb>(i, i)];
          for (int k = j + 1; k < i; k++)
            temp += A[lower<nb>(i, k)] * A[lower<nb>(k, j)];
        }

        __syncthreads();

        if (i > j)
          A[lower<nb>(i, j)] = temp * ajj;

        __syncthreads();
      }
    }
  }
}

template <CBlasUplo uplo, CBlasDiag diag, unsigned int bx>
__global__ void strti2(const float * A, float * B, int * info, int lda, int ldb, int n) {
  // info parameter cached in shared memory for fast access by all threads in the block
  __shared__ int sinfo;

  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 64 threads are needed but this would
   * use all 16384 bytes (64 * 64 * sizeof(float)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 8320 bytes((64 * (64 + 1)) / 2 * sizeof(float))
   * of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ float a[(bx * (bx + 1)) / 2];

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse using the packed device function
    stpti2<CBlasUpper, diag, bx, bx>(n, a, &sinfo);

    // Write info back to global memory
    if (threadIdx.x == 0)
      *info = sinfo;

    // Write the upper triangle of A back to B in global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j)
        B[j * ldb + threadIdx.x] = a[upper(threadIdx.x, j)];
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse using the packed device function
    stpti2<CBlasLower, diag, bx, bx>(n, a, &sinfo);

    // Write info back to global memory
    if (threadIdx.x == 0)
      *info = sinfo;

    // Write the lower triangle of A back to B in global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j && threadIdx.x < n)
        B[j * ldb + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template <CBlasUplo uplo,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void strtimm2(float * __restrict__ A, float * __restrict__ B,
                         int * __restrict__ info, int lda, int ldb, int j, int jb, int n) {
  // info parameter cached in shared memory for fast access by all threads in the block
  __shared__ int sinfo;

  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 64 threads are needed but this would
   * use all 16384 bytes (64 * 64 * sizeof(float)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 8320 bytes((64 * (64 + 1)) / 2 * sizeof(float))
   * of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ float a[(nb * (nb + 1)) / 2];

  const int i = threadIdx.y * bx + threadIdx.x;

  if (uplo == CBlasUpper) {
    if (blockIdx.x == gridDim.x - 1) {
      if (blockIdx.y == 0) {
        A += j * lda + j;
        // Read upper triangle of A into shared memory
        #pragma unroll
        for (int k = 0; k < nb; k++) {
          if (i <= k)
            a[upper(i, k)] = A[k * lda + i];
        }

        __syncthreads();

        // Perform the triangular inverse using the packed device function
        stpti2<CBlasUpper, CBlasNonUnit, nb, bx>(jb, a, &sinfo);

        // Write info back to global memory
        if (i == 0)
          *info = sinfo;

        // Write the upper triangle of A back to global memory
        for (int k = 0; k < jb; k++) {
          if (i <= k)
            A[k * lda + i] = a[upper(i, k)];
        }
      }
    }
    else
      strmm2LUN<CBlasNonUnit, mb, nb, kb, bx, by>(j, jb, 1.0f, A, lda, &A[j * lda], lda, B, ldb);
  }
  else {
    if (blockIdx.x == gridDim.x - 1) {
      if (blockIdx.y == 0) {
        // Read lower triangle of A into shared memory
        #pragma unroll
        for (int k = 0; k < nb; k++) {
          if (i >= k)
            a[lower<nb>(i, k)] = A[k * lda + i];
        }

        __syncthreads();

        // Perform the triangular inverse using the packed device function
        stpti2<CBlasLower, CBlasNonUnit, nb, bx>(jb, a, &sinfo);

        // Write info back to global memory
        if (i == 0)
          *info = sinfo;

        // Write the lower triangle of A back to global memory
        for (int k = 0; k < jb; k++) {
          if (i >= k && i < jb)
            A[k * lda + i] = a[lower<nb>(i, k)];
        }
      }
    }
    else
      strmm2LLN<CBlasNonUnit, mb, nb, kb, bx, by>(n - j - jb, jb, 1.0f,
                                                  &A[jb * lda + jb], lda,
                                                  &A[jb], lda, B, ldb);
  }
}

template __global__ void strti2<CBlasUpper, CBlasUnit, 64>(const float * __restrict__, float * __restrict__, int * __restrict__, int, int, int);
template __global__ void strti2<CBlasUpper, CBlasNonUnit, 64>(const float * __restrict__, float * __restrict__, int * __restrict__, int, int, int);
template __global__ void strti2<CBlasLower, CBlasUnit, 64>(const float * __restrict__, float * __restrict__, int * __restrict__, int, int, int);
template __global__ void strti2<CBlasLower, CBlasNonUnit, 64>(const float * __restrict__, float * __restrict__, int * __restrict__, int, int, int);

template __global__ void strtimm2<CBlasUpper, 64, 16, 16, 16, 4>(float * __restrict__, float * __restrict__, int * __restrict__, int, int, int, int, int);
template __global__ void strtimm2<CBlasLower, 64, 16, 16, 16, 4>(float * __restrict__, float * __restrict__, int * __restrict__, int, int, int, int, int);

#if 0
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void strti2_(const char *, const char *, const int *, float *, const int *, int *);
static inline void strti2(CBlasUplo uplo, CBlasDiag diag, int n, float * A, int lda, int * info) {
  if (uplo == CBlasUpper) {
    if (diag == CBlasNonUnit)
      strti2<CBlasUpper, CBlasNonUnit, 64><<<1,64>>>(A, A, info, lda, lda, n);
    else
      strti2<CBlasUpper, CBlasUnit, 64><<<1,64>>>(A, A, info, lda, lda, n);
  }
  else {
    if (diag == CBlasNonUnit)
      strti2<CBlasLower, CBlasNonUnit, 64><<<1,64>>>(A, A, info, lda, lda, n);
    else
      strti2<CBlasLower, CBlasUnit, 64><<<1,64>>>(A, A, info, lda, lda, n);
  }
}

static int cond(int, float, float *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  CBlasDiag diag;
  int n;

  if (argc != 4) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <diag>  is 'U' or 'u' for CBlasUnit or 'N' or 'n' for CBlasNonUnit\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  char d;
  if (sscanf(argv[2], "%c", &d) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[2]);
    return 2;
  }
  switch (d) {
    case 'u': case 'U': diag = CBlasUnit; break;
    case 'n': case 'N': diag = CBlasNonUnit; break;
    default: fprintf(stderr, "Unknown diag '%c'\n", u); return 1;
  }

  if (sscanf(argv[3], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[3]);
    return 3;
  }

  float * A, * dA, * refA;
  size_t lda = (n + 3) & ~3, dlda;
  int * dinfo;
  if ((A = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(float), n));
  CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
  dlda /= sizeof(float);

  cond(n, 2.0f, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(float));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(float), A, lda * sizeof(float), n * sizeof(float), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  int info = 0, refInfo = 0;
  strti2_((const char *)&uplo, (const char *)&diag, &n, refA, (const int *)&lda, &refInfo);
  strti2(uplo, diag, n, dA, dlda, dinfo);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(float), dA, dlda * sizeof(float), n * sizeof(float), n, hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", refA[j * lda + i]);
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  float error = 0.0f;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      float diff = fabsf(refA[j * lda + i] - A[j * lda + i]);
      if (diff > error)
        error = diff;
    }
  }

  fprintf(stdout, "Info = %d, refInfo = %d, Error = %6.3e\n", info, refInfo, error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));
  CUDA_ERROR_CHECK(hipFree(dinfo));

  return info;
}

static int cond(int n, float c, float * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  float * u, * v, * w;
  size_t offset = (n + 3u) & ~3u;

  if ((u = (float *)malloc(3 * offset * sizeof(float))) == NULL)
    return 1;

  v = &u[offset];
  w = &v[offset];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] = 0.0f;
  }

  A[0] = 1.0f;
  A[lda + 1] = c;
  for (int j = 2; j < n; j++)
    A[j * lda + j] = ((float) rand() / (float)RAND_MAX) * (c - 1.0f) + 1.0f;

  float t = 0.0f, s = 0.0f;
  for (int j = 0; j < n; j++) {
    // u is a random vector
    u[j] = (float)rand() / (float)RAND_MAX;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += u[j] * u[j];
    // s = t^2 u'v / 2
    s += u[j] * v[j];
  }
  t = 2.0f / t;
  s = t * t * s / 2.0f;

  // w = tv - su
  for (int j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * w[j] + w[i] * u[j];
  }

  free(u);

  return 0;
}
#endif

#if 1
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void strmm_(const char *, const char *, const char *, const char *,
                       const int *, const int *,
                       const float *, const float *, const int *, float *, const int *);
extern "C" void strti2_(const char *, const char *, const int *, float *, const int *, int *);

static inline void strtimm2(CBlasUplo uplo, int j, int jb, int n, float * A, int lda, float * B, int ldb, int * info) {
  const unsigned int mb = 64;
  const unsigned int nb = 16;
  const unsigned int kb = 16;
  const unsigned int bx = 16;
  const unsigned int by =  4;

  if (jb > nb) {
    fputs("On entry to strtimm2 parameter 3 had an invalid value\n", stderr);
    return;
  }

  if (uplo == CBlasUpper) {
    const unsigned int gx = (j + mb - 1) / mb;
    const unsigned int gy = (jb + nb - 1) / nb;
    strtimm2<CBlasUpper, mb, nb, kb, bx, by><<<dim3(gx + 1, max(gy, 1)), dim3(bx, by)>>>(A, B, info, lda, ldb, j, jb, n);
  }
  else {
    const unsigned int gx = (n - j - jb + mb - 1) / mb;
    const unsigned int gy = (jb + nb - 1) / nb;
    strtimm2<CBlasLower, mb, nb, kb, bx, by><<<dim3(gx + 1, max(gy, 1)), dim3(bx, by)>>>(A, B, info, lda, ldb, j, jb, n);
  }
}

static void strtimm2_(CBlasUplo uplo, int j, int jb, int n, float * A, int lda, float * B, int ldb, int * info) {
  const float one = 1.0f;
  if (uplo == CBlasUpper) {
    strti2_("Upper", "Non-Unit", &jb, &A[j * lda + j], &lda, info);
    for (int k = 0; k < jb; k++)
      memcpy(&B[k * ldb], &A[(j + k) * lda], j * sizeof(float));
    strmm_("Left", "Upper", "No Transpose", "Non-Unit", &j, &jb, &one, A, &lda, B, &ldb);
  }
  else {
    const int n_j_jb = n - j - jb;
    strti2_("Lower", "Non-Unit", &jb, A, &lda, info);
    for (int k = 0; k < jb; k++)
      memcpy(&B[k * ldb], &A[k * lda + jb], (n - j - jb) * sizeof(float));
    strmm_("Left", "Lower", "No Transpose", "Non-Unit", &n_j_jb, &jb, &one, &A[jb * lda + jb], &lda, B, &ldb);
  }
}

static int cond(int, float, float *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  int j, nb, n;

  if (argc != 5) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <j>     is the current index\n"
                    "  <nb>    is the block size\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  if (sscanf(argv[2], "%d", &j) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[2]);
    return 2;
  }

  if (sscanf(argv[3], "%d", &nb) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[3]);
    return 3;
  }

  if (sscanf(argv[4], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[4]);
    return 4;
  }

  if (j + nb > n) {
    fputs("n is too small\n", stderr);
    return 5;
  }

  int jb = min(nb, n - j);

  float * A, * B, * dA, * dB, * refA, * refB;
  size_t lda, ldb, dlda, dldb;
  int * dinfo;
  if (uplo == CBlasUpper) {
    if ((A = (float *)malloc((lda = (j + jb + 3u) & ~3u) * (j + jb) * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate A\n");
      return -1;
    }
    if ((B = (float *)calloc((ldb = (j + 3u) & ~3u), jb * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate B\n");
      return -2;
    }
    if ((refA = (float *)malloc(lda * (j + jb) * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate refA\n");
      return -3;
    }
    if ((refB = (float *)calloc(ldb, jb * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate refB\n");
      return -4;
    }
    CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, (j + jb) * sizeof(float), j + jb));
    CUDA_ERROR_CHECK(hipMallocPitch((void **)&dB, &dldb, j * sizeof(float), jb));
    CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
    dlda /= sizeof(float);
    dldb /= sizeof(float);

    cond(j + jb, 2.0f, A, lda);

    for (int k = 0; k < j + jb; k++)
      memcpy(&refA[k * lda], &A[k * lda], (j + jb) * sizeof(float));

    CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(float), A, lda * sizeof(float), (j + jb) * sizeof(float), j + jb, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy2D(dB, dldb * sizeof(float), B, ldb * sizeof(float), j * sizeof(float), jb, hipMemcpyHostToDevice));

#ifdef PRINT
    for (int i = 0; i < j + jb; i++) {
      for (int k = 0; k < j + jb; k++)
        fprintf(stderr, "%15.6f", A[k * lda + i]);
      fprintf(stderr, "\n");
    }
#endif
  }
  else {
    if ((A = (float *)malloc((lda = (n - j + 3u) & ~3u) * (n - j) * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate A\n");
      return -1;
    }
    if ((B = (float *)calloc((ldb = ((n - j - jb) + 3u) & ~3u), jb * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate B\n");
      return -2;
    }
    if ((refA = (float *)malloc(lda * (n - j) * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate refA\n");
      return -3;
    }
    if ((refB = (float *)calloc(ldb, jb * sizeof(float))) == NULL) {
      fprintf(stderr, "Failed to allocate refB\n");
      return -4;
    }
    CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, (n - j) * sizeof(float), n - j));
    CUDA_ERROR_CHECK(hipMallocPitch((void **)&dB, &dldb, (n - j - jb) * sizeof(float), jb));
    CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
    dlda /= sizeof(float);
    dldb /= sizeof(float);

    cond(n - j, 2.0f, A, lda);

    for (int k = 0; k < n - j; k++)
      memcpy(&refA[k * lda], &A[k * lda], (n - j) * sizeof(float));

    CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(float), A, lda * sizeof(float), (n - j) * sizeof(float), n - j, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy2D(dB, dldb * sizeof(float), B, ldb * sizeof(float), (n - j - jb) * sizeof(float), jb, hipMemcpyHostToDevice));

#ifdef PRINT
    for (int i = 0; i < n - j; i++) {
      for (int k = 0; k < n - j; k++)
        fprintf(stderr, "%15.6f", A[k * lda + i]);
      fprintf(stderr, "\n");
    }
#endif
  }

  int info = 0, refInfo = 0;
  strtimm2(uplo, j, jb, n, dA, dlda, dB, dldb, dinfo);
  strtimm2_(uplo, j, jb, n, refA, lda, refB, ldb, &refInfo);
  CUDA_ERROR_CHECK(hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost));

  float error = 0.0f;
  if (uplo == CBlasUpper) {
    CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(float), dA, dlda * sizeof(float), (j + jb) * sizeof(float), j + jb, hipMemcpyDeviceToHost));
    CUDA_ERROR_CHECK(hipMemcpy2D(B, ldb * sizeof(float), dB, dldb * sizeof(float), j * sizeof(float), jb, hipMemcpyDeviceToHost));

#ifdef PRINT
    fputs("\nrefA:\n", stderr);
    for (int i = 0; i < j + jb; i++) {
      for (int k = 0; k < j + jb; k++)
        fprintf(stderr, "%15.6f", refA[k * lda + i]);
      fprintf(stderr, "\n");
    }
    fputs("\nrefB:\n", stderr);
    for (int i = 0; i < j; i++) {
      for (int k = 0; k < jb; k++)
        fprintf(stderr, "%15.6f", refB[k * ldb + i]);
      fprintf(stderr, "\n");
    }

    fputs("\nA:\n", stderr);
    for (int i = 0; i < j + jb; i++) {
      for (int k = 0; k < j + jb; k++)
        fprintf(stderr, "%15.6f", A[k * lda + i]);
      fprintf(stderr, "\n");
    }
    fputs("\nB:\n", stderr);
    for (int i = 0; i < j; i++) {
      for (int k = 0; k < jb; k++)
        fprintf(stderr, "%15.6f", B[k * ldb + i]);
      fprintf(stderr, "\n");
    }
#endif

    for (int k = 0; k < j + jb; k++) {
      for (int i = 0; i < j + jb; i++) {
        float diff = fabsf(A[k * lda + i] - refA[k * lda + i]);
        if (diff > error)
          error = diff;
      }
    }

    for (int k = 0; k < jb; k++) {
      for (int i = 0; i < j; i++) {
        float diff = fabsf(B[k * ldb + i] - refB[k * ldb + i]);
        if (diff > error)
          error = diff;
      }
    }
  }
  else {
    CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(float), dA, dlda * sizeof(float), (n - j) * sizeof(float), n - j, hipMemcpyDeviceToHost));
    CUDA_ERROR_CHECK(hipMemcpy2D(B, ldb * sizeof(float), dB, dldb * sizeof(float), (n - j - jb) * sizeof(float), jb, hipMemcpyDeviceToHost));

#ifdef PRINT
    fputs("\nrefA:\n", stderr);
    for (int i = 0; i < n - j; i++) {
      for (int k = 0; k < n - j; k++)
        fprintf(stderr, "%15.6f", refA[k * lda + i]);
      fprintf(stderr, "\n");
    }
    fputs("\nrefB:\n", stderr);
    for (int i = 0; i < n - j - jb; i++) {
      for (int k = 0; k < jb; k++)
        fprintf(stderr, "%15.6f", refB[k * ldb + i]);
      fprintf(stderr, "\n");
    }

    fputs("\nA:\n", stderr);
    for (int i = 0; i < n - j; i++) {
      for (int k = 0; k < n - j; k++)
        fprintf(stderr, "%15.6f", A[k * lda + i]);
      fprintf(stderr, "\n");
    }
    fputs("\nB:\n", stderr);
    for (int i = 0; i < n - j - jb; i++) {
      for (int k = 0; k < jb; k++)
        fprintf(stderr, "%15.6f", B[k * ldb + i]);
      fprintf(stderr, "\n");
    }
#endif

    for (int k = 0; k < n - j; k++) {
      for (int i = 0; i < n - j; i++) {
        float diff = fabsf(A[k * lda + i] - refA[k * lda + i]);
        if (diff > error)
          error = diff;
      }
    }

    for (int k = 0; k < jb; k++) {
      for (int i = 0; i < n - j - jb; i++) {
        float diff = fabsf(B[k * ldb + i] - refB[k * ldb + i]);
        if (diff > error)
          error = diff;
      }
    }
  }

  fprintf(stdout, "Info = %d, refInfo = %d, Error = %6.3e\n", info, refInfo, error);

  free(A);
  free(B);
  free(refA);
  free(refB);
  CUDA_ERROR_CHECK(hipFree(dA));
  CUDA_ERROR_CHECK(hipFree(dB));
  CUDA_ERROR_CHECK(hipFree(dinfo));

  return info;
}

static int cond(int n, float c, float * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  float * u, * v, * w;
  size_t offset = (n + 3u) & ~3u;

  if ((u = (float *)malloc(3 * offset * sizeof(float))) == NULL)
    return 1;

  v = &u[offset];
  w = &v[offset];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] = 0.0f;
  }

  A[0] = 1.0f;
  A[lda + 1] = c;
  for (int j = 2; j < n; j++)
    A[j * lda + j] = ((float) rand() / (float)RAND_MAX) * (c - 1.0f) + 1.0f;

  float t = 0.0f, s = 0.0f;
  for (int j = 0; j < n; j++) {
    // u is a random vector
    u[j] = (float)rand() / (float)RAND_MAX;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += u[j] * u[j];
    // s = t^2 u'v / 2
    s += u[j] * v[j];
  }
  t = 2.0f / t;
  s = t * t * s / 2.0f;

  // w = tv - su
  for (int j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * w[j] + w[i] * u[j];
  }

  free(u);

  return 0;
}
#endif
