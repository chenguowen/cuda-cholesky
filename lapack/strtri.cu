#include "hip/hip_runtime.h"
#include "blas.h"

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

template <CBlasUplo uplo, CBlasDiag diag, unsigned int bx>
__global__ void strti2(float * A, int * info, int lda, int n) {
  // info parameter cached in shared memory for fast access by all threads in the block
  __shared__ int sinfo;

  // thread 0 is the only thread to write to info in shared or global memory
  if (threadIdx.x == 0)
    *info = sinfo = 0;  // initialise info to zero and cache

  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 64 threads are needed but this would
   * use all 16384 bytes (64 * 64 * sizeof(float)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 8320 bytes((64 * (64 + 1)) / 2 * sizeof(float))
   * of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ float a[(bx * (bx + 1)) / 2];
  __shared__ float ajj;

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = 0; j < n; j++) {
      float temp;
      // Read current column into registers
      if (threadIdx.x <= j)
        temp = a[upper(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (temp == 0.0f) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[upper(threadIdx.x, j)] = 1.0f / temp;
          ajj = -a[upper(threadIdx.x, j)];
        }
        else
          ajj = -1.0f;
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x < j) {
        if (diag == CBlasNonUnit)
          temp *= a[upper(threadIdx.x, threadIdx.x)];
        for (int k = threadIdx.x + 1; k < j; k++)
          temp += a[upper(threadIdx.x, k)] * a[upper(k, j)];
      }

      __syncthreads();

      if (threadIdx.x < j)
        a[upper(threadIdx.x, j)] = temp * ajj;

      __syncthreads();
    }

    // Write the upper triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j)
        A[j * lda + threadIdx.x] = a[upper(threadIdx.x, j)];
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = n - 1; j >= 0; j--) {
      float temp;
      // Read current column into registers
      if (threadIdx.x >= j)
        temp = a[lower<bx>(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (temp == 0.0f) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[lower<bx>(threadIdx.x, j)] = 1.0f / temp;
          ajj = -a[lower<bx>(threadIdx.x, j)];
        }
        else
          ajj = -1.0f;
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x > j) {
        if (diag == CBlasNonUnit)
          temp *= a[lower<bx>(threadIdx.x, threadIdx.x)];
        for (int k = j + 1; k < threadIdx.x; k++)
          temp += a[lower<bx>(threadIdx.x, k)] * a[lower<bx>(k, j)];
      }

      __syncthreads();

      if (threadIdx.x > j)
        a[lower<bx>(threadIdx.x, j)] = temp * ajj;

      __syncthreads();
    }

    // Write the lower triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        A[j * lda + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template __global__ void strti2<CBlasUpper, CBlasUnit, 64>(float *, int *, int, int);
template __global__ void strti2<CBlasUpper, CBlasNonUnit, 64>(float *, int *, int, int);
template __global__ void strti2<CBlasLower, CBlasUnit, 64>(float *, int *, int, int);
template __global__ void strti2<CBlasLower, CBlasNonUnit, 64>(float *, int *, int, int);

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void strti2_(const char *, const char *, const int *, float *, const int *, int *);
static inline void strti2(CBlasUplo uplo, CBlasDiag diag, int n, float * A, int lda, int * info) {
  if (uplo == CBlasUpper) {
    if (diag == CBlasNonUnit)
      strti2<CBlasUpper, CBlasNonUnit, 64><<<1,64>>>(A, info, lda, n);
    else
      strti2<CBlasUpper, CBlasUnit, 64><<<1,64>>>(A, info, lda, n);
  }
  else {
    if (diag == CBlasNonUnit)
      strti2<CBlasLower, CBlasNonUnit, 64><<<1,64>>>(A, info, lda, n);
    else
      strti2<CBlasLower, CBlasUnit, 64><<<1,64>>>(A, info, lda, n);
  }
}

static int cond(int, float, float *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  CBlasDiag diag;
  int n;

  if (argc != 4) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <diag>  is 'U' or 'u' for CBlasUnit or 'N' or 'n' for CBlasNonUnit\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  char d;
  if (sscanf(argv[2], "%c", &d) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[2]);
    return 2;
  }
  switch (d) {
    case 'u': case 'U': diag = CBlasUnit; break;
    case 'n': case 'N': diag = CBlasNonUnit; break;
    default: fprintf(stderr, "Unknown diag '%c'\n", u); return 1;
  }

  if (sscanf(argv[3], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[3]);
    return 3;
  }

  float * A, * dA, * refA;
  size_t lda = (n + 3) & ~3, dlda;
  int * dinfo;
  if ((A = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(float), n));
  CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
  dlda /= sizeof(float);

  cond(n, 2.0f, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(float));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(float), A, lda * sizeof(float), n * sizeof(float), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  int info = 0, refInfo = 0;
  strti2_((const char *)&uplo, (const char *)&diag, &n, refA, (const int *)&lda, &refInfo);
  strti2(uplo, diag, n, dA, dlda, dinfo);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(float), dA, dlda * sizeof(float), n * sizeof(float), n, hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", refA[j * lda + i]);
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  float error = 0.0f;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      float diff = fabsf(refA[j * lda + i] - A[j * lda + i]);
      if (diff > error)
        error = diff;
    }
  }

  fprintf(stdout, "Info = %d, refInfo = %d, Error = %6.3e\n", info, refInfo, error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));
  CUDA_ERROR_CHECK(hipFree(dinfo));

  return info;
}

static int cond(int n, float c, float * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  float * u, * v, * w;
  size_t offset = (n + 3u) & ~3u;

  if ((u = (float *)malloc(3 * offset * sizeof(float))) == NULL)
    return 1;

  v = &u[offset];
  w = &v[offset];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] = 0.0f;
  }

  A[0] = 1.0f;
  A[lda + 1] = c;
  for (int j = 2; j < n; j++)
    A[j * lda + j] = ((float) rand() / (float)RAND_MAX) * (c - 1.0f) + 1.0f;

  float t = 0.0f, s = 0.0f;
  for (int j = 0; j < n; j++) {
    // u is a random vector
    u[j] = (float)rand() / (float)RAND_MAX;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += u[j] * u[j];
    // s = t^2 u'v / 2
    s += u[j] * v[j];
  }
  t = 2.0f / t;
  s = t * t * s / 2.0f;

  // w = tv - su
  for (int j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * w[j] + w[i] * u[j];
  }

  free(u);

  return 0;
}
