#include "hip/hip_runtime.h"
#include "blas.h"
#include <hip/hip_complex.h>

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

template <CBlasUplo uplo, CBlasDiag diag, unsigned int bx>
__global__ void ctrti2(hipComplex * A, int * info, int lda, int n) {
  // info parameter cached in shared memory for fast access by all threads in the block
  __shared__ int sinfo;

  // thread 0 is the only thread to write to info in shared or global memory
  if (threadIdx.x == 0)
    *info = sinfo = 0;  // initialise info to zero and cache

  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 32 threads are needed but this would
   * use 8192 bytes (32 * 32 * sizeof(hipComplex)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 4224 bytes((32 * (32 + 1)) / 2 * sizeof(hipComplex))
   * of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ hipComplex a[(bx * (bx + 1)) / 2];
  __shared__ hipComplex ajj;

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = 0; j < n; j++) {
      hipComplex temp;
      // Read current column into registers
      if (threadIdx.x <= j)
        temp = a[upper(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (hipCrealf(temp) == 0.0f && hipCimagf(temp) == 0.0f) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[upper(threadIdx.x, j)] = hipCdivf(make_hipComplex(1.0f, 0.0f), temp);
          ajj = make_hipComplex(-hipCrealf(a[upper(threadIdx.x, j)]), -hipCimagf(a[upper(threadIdx.x, j)]));
        }
        else
          ajj = make_hipComplex(-1.0f, 0.0f);
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x < j) {
        if (diag == CBlasNonUnit)
          temp = hipCmulf(temp, a[upper(threadIdx.x, threadIdx.x)]);
        for (int k = threadIdx.x + 1; k < j; k++)
          temp = hipCfmaf(a[upper(threadIdx.x, k)], a[upper(k, j)], temp);
      }

      __syncthreads();

      if (threadIdx.x < j)
        a[upper(threadIdx.x, j)] = hipCmulf(temp, ajj);

      __syncthreads();
    }

    // Write the upper triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j)
        A[j * lda + threadIdx.x] = a[upper(threadIdx.x, j)];
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the triangular inverse
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = n - 1; j >= 0; j--) {
      hipComplex temp;
      // Read current column into registers
      if (threadIdx.x >= j)
        temp = a[lower<bx>(threadIdx.x, j)];

      // Thread j calculates the diagonal element
      if (threadIdx.x == j) {
        if (diag == CBlasNonUnit) {
          if (hipCrealf(temp) == 0.0f && hipCimagf(temp) == 0.0f) {
            *info = sinfo = j + 1;        // update info in shared and global memory
            break;
          }
          a[lower<bx>(threadIdx.x, j)] = hipCdivf(make_hipComplex(1.0f, 0.0f), temp);
          ajj = make_hipComplex(-hipCrealf(a[lower<bx>(threadIdx.x, j)]), -hipCimagf(a[lower<bx>(threadIdx.x, j)]));
        }
        else
          ajj = make_hipComplex(-1.0f, 0.0f);
      }

      __syncthreads();

      // If info != 0 return (matrix is singular)
      if (sinfo != 0)
        return;

      if (threadIdx.x > j) {
        if (diag == CBlasNonUnit)
          temp = hipCmulf(temp, a[lower<bx>(threadIdx.x, threadIdx.x)]);
        for (int k = j + 1; k < threadIdx.x; k++)
          temp = hipCfmaf(a[lower<bx>(threadIdx.x, k)], a[lower<bx>(k, j)], temp);
      }

      __syncthreads();

      if (threadIdx.x > j)
        a[lower<bx>(threadIdx.x, j)] = hipCmulf(temp, ajj);

      __syncthreads();
    }

    // Write the lower triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        A[j * lda + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template __global__ void ctrti2<CBlasUpper, CBlasUnit, 32>(hipComplex *, int *, int, int);
template __global__ void ctrti2<CBlasUpper, CBlasNonUnit, 32>(hipComplex *, int *, int, int);
template __global__ void ctrti2<CBlasLower, CBlasUnit, 32>(hipComplex *, int *, int, int);
template __global__ void ctrti2<CBlasLower, CBlasNonUnit, 32>(hipComplex *, int *, int, int);

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void ctrti2_(const char *, const char *, const int *, void *, const int *, int *);
static inline void ctrti2(CBlasUplo uplo, CBlasDiag diag, int n, hipComplex * A, int lda, int * info) {
  if (uplo == CBlasUpper) {
    if (diag == CBlasNonUnit)
      ctrti2<CBlasUpper, CBlasNonUnit, 32><<<1,32>>>(A, info, lda, n);
    else
      ctrti2<CBlasUpper, CBlasUnit, 32><<<1,32>>>(A, info, lda, n);
  }
  else {
    if (diag == CBlasNonUnit)
      ctrti2<CBlasLower, CBlasNonUnit, 32><<<1,32>>>(A, info, lda, n);
    else
      ctrti2<CBlasLower, CBlasUnit, 32><<<1,32>>>(A, info, lda, n);
  }
}

static int ccond(int, float, float complex *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  CBlasDiag diag;
  int n;

  if (argc != 4) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <diag>  is 'U' or 'u' for CBlasUnit or 'N' or 'n' for CBlasNonUnit\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  char d;
  if (sscanf(argv[2], "%c", &d) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[2]);
    return 2;
  }
  switch (d) {
    case 'u': case 'U': diag = CBlasUnit; break;
    case 'n': case 'N': diag = CBlasNonUnit; break;
    default: fprintf(stderr, "Unknown diag '%c'\n", u); return 1;
  }

  if (sscanf(argv[3], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[3]);
    return 3;
  }

  float complex * A, * refA;
  hipComplex * dA;
  size_t lda = (n + 1u) & ~1u, dlda;
  int * dinfo;
  if ((A = (float complex *)malloc(lda * n * sizeof(float complex))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (float complex *)malloc(lda * n * sizeof(float complex))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(hipComplex), n));
  CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
  dlda /= sizeof(hipComplex);

  ccond(n, 2.0f, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(float complex));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(hipComplex), A, lda * sizeof(float complex), n * sizeof(hipComplex), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", crealf(A[j * lda + i]), cimagf(A[j * lda + i]));
    fprintf(stderr, "\n");
  }

  int info = 0, refInfo = 0;
  ctrti2_((const char *)&uplo, (const char *)&diag, &n, refA, (const int *)&lda, &refInfo);
  ctrti2(uplo, diag, n, dA, dlda, dinfo);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(float complex), dA, dlda * sizeof(hipComplex), n * sizeof(hipComplex), n, hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", crealf(refA[j * lda + i]), cimagf(refA[j * lda + i]));
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", crealf(A[j * lda + i]), cimagf(A[j * lda + i]));
    fprintf(stderr, "\n");
  }

  float real_error = 0.0f, imag_error = 0.0f;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      float diff = fabsf(crealf(refA[j * lda + i]) - crealf(A[j * lda + i]));
      if (diff > real_error)
        real_error = diff;
      diff = fabsf(cimagf(refA[j * lda + i]) - cimagf(A[j * lda + i]));
      if (diff > imag_error)
        imag_error = diff;
    }
  }

  fprintf(stdout, "Info = %d, refInfo = %d, Error = %6.3e + %6.3ei\n", info, refInfo, real_error, imag_error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));
  CUDA_ERROR_CHECK(hipFree(dinfo));

  return info;
}

static int ccond(int n, float c, float complex * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0f)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  float complex * u, * v, * w;
  size_t offset = (n + 1u) & ~1u;

  if ((u = (float complex *)malloc(3 * offset * sizeof(float complex))) == NULL)
    return 1;

  v = &u[offset];
  w = &v[offset];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (size_t j = 0; j < n; j++) {
    for (size_t i = 0; i < n; i++)
      A[j * lda + i] = 0.0f + 0.0f * I;
  }

  A[0] = 1.0f + 0.0f * I;
  A[lda + 1] = c + 0.0f * I;
  for (size_t j = 2; j < n; j++)
    A[j * lda + j] = ((float) rand() / (float)RAND_MAX) * (c - 1.0f) + 1.0f + 0.0f * I;

  float t = 0.0;
  float complex s = 0.0f + 0.0f * I;
  for (size_t j = 0; j < n; j++) {
    // u is a complex precision random vector
    u[j] = ((float)rand() / (float)RAND_MAX) + ((float)rand() / (float)RAND_MAX) * I;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += crealf(conjf(u[j]) * u[j]);
    // s = t^2 u'v / 2
    s += conjf(u[j]) * v[j];
  }
  t = 2.0f / t;
  s = t * t * s / (2.0f + 0.0f * I);

  // w = tv - su
  for (size_t j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (size_t j = 0; j < n; j++) {
    for (size_t i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * conjf(w[j]) + w[i] * conjf(u[j]);
  }

  free(u);

  return 0;
}
