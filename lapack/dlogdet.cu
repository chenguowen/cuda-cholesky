
#include <hip/hip_runtime.h>
/**
 * Copied from CUDA SDK version 5.0.35 with the following changes:
 *  - reduceBlocks is inlined to reduceSinglePass
 *  - reduceMultipass is removed
 *  - reduceSinglePass is renamed reduce
 *
 * To calculate the log of the determinant (log(prod(x)^2) = 2 * sum(log(x))):
 *  - incx is added to sum elements that are not contiguous (i.e. along the
 *    diagonal of A)
 *  - elements of x are calculated as log(x) when reading from global memory
 *  - the final sum is written as 2 * sum(log(x)) to compute the determinant
 */

template <unsigned int bs>
__device__ void reduceBlock(volatile double * sdata, double sum, const unsigned int i) {
  sdata[i] = sum;
  __syncthreads();

  // do reduction in shared mem
  if (bs >= 512) { if (i < 256) { sdata[i] = sum = sum + sdata[i + 256]; } __syncthreads(); }
  if (bs >= 256) { if (i < 128) { sdata[i] = sum = sum + sdata[i + 128]; } __syncthreads(); }
  if (bs >= 128) { if (i <  64) { sdata[i] = sum = sum + sdata[i +  64]; } __syncthreads(); }

  if (i < 32) {
    if (bs >=  64) { sdata[i] = sum = sum + sdata[i + 32]; }
    if (bs >=  32) { sdata[i] = sum = sum + sdata[i + 16]; }
    if (bs >=  16) { sdata[i] = sum = sum + sdata[i +  8]; }
    if (bs >=   8) { sdata[i] = sum = sum + sdata[i +  4]; }
    if (bs >=   4) { sdata[i] = sum = sum + sdata[i +  2]; }
    if (bs >=   2) { sdata[i] = sum = sum + sdata[i +  1]; }
  }
}

// Global variable used by reduce to count how many blocks have finished
__device__ unsigned int retirementCount = 0;

// This reduction kernel reduces an arbitrary size array in a single kernel invocation
// It does so by keeping track of how many blocks have finished.  After each thread
// block completes the reduction of its own block of data, it "takes a ticket" by
// atomically incrementing a global counter.  If the ticket value is equal to the number
// of thread blocks, then the block holding the ticket knows that it is the last block
// to finish.  This last block is responsible for summing the results of all the other
// blocks.
//
// In order for this to work, we must be sure that before a block takes a ticket, all
// of its memory transactions have completed.  This is what __threadfence() does -- it
// blocks until the results of all outstanding memory transactions within the
// calling thread are visible to all other threads.
//
// For more details on the reduction algorithm (notably the multi-pass approach), see
// the "reduction" sample in the CUDA SDK.
template <unsigned int bs, bool nIsPow2>
__global__ void reduce(const double *g_idata, double *g_odata, int inc, int n) {
  __shared__ double sdata[bs];

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  const unsigned int tid = threadIdx.x;
  const unsigned int gs = bs * 2 * gridDim.x;
  double sum = 0.0f;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gs and therefore fewer elements per thread
  for (int i = blockIdx.x * (bs * 2) + threadIdx.x; i < n; i += gs) {
    sum += log(g_idata[i * inc]);

    // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
    if (nIsPow2 || i + bs < n)
      sum += log(g_idata[(i + bs) * inc]);
  }

  // do reduction in shared mem
  reduceBlock<bs>(sdata, sum, tid);

  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] = 2.0 * sdata[0];

  if (gridDim.x > 1) {
    __shared__ bool amLast;

    // wait until all outstanding memory instructions in this thread are finished
    __threadfence();

    // Thread 0 takes a ticket
    if (tid == 0) {
      unsigned int ticket = atomicInc(&retirementCount, gridDim.x);
      // If the ticket ID is equal to the number of blocks, we are the last block!
      amLast = (ticket == gridDim.x-1);
    }

    __syncthreads();

    // The last block sums the results of all other blocks
    if (amLast) {
      double sum = 0.0f;

      for (int i = tid; i < gridDim.x; i += bs)
        sum += g_odata[i];

      reduceBlock<bs>(sdata, sum, tid);

      if (tid == 0) {
        g_odata[0] = sdata[0];

        // reset retirement count so that next run succeeds
        retirementCount = 0;
      }
    }
  }
}

template __global__ void reduce<512,  true>(const double *, double *, int, int);
template __global__ void reduce<256,  true>(const double *, double *, int, int);
template __global__ void reduce<128,  true>(const double *, double *, int, int);
template __global__ void reduce< 64,  true>(const double *, double *, int, int);
template __global__ void reduce< 32,  true>(const double *, double *, int, int);
template __global__ void reduce< 16,  true>(const double *, double *, int, int);
template __global__ void reduce<  8,  true>(const double *, double *, int, int);
template __global__ void reduce<  4,  true>(const double *, double *, int, int);
template __global__ void reduce<  2,  true>(const double *, double *, int, int);
template __global__ void reduce<  1,  true>(const double *, double *, int, int);

template __global__ void reduce<512, false>(const double *, double *, int, int);
template __global__ void reduce<256, false>(const double *, double *, int, int);
template __global__ void reduce<128, false>(const double *, double *, int, int);
template __global__ void reduce< 64, false>(const double *, double *, int, int);
template __global__ void reduce< 32, false>(const double *, double *, int, int);
template __global__ void reduce< 16, false>(const double *, double *, int, int);
template __global__ void reduce<  8, false>(const double *, double *, int, int);
template __global__ void reduce<  4, false>(const double *, double *, int, int);
template __global__ void reduce<  2, false>(const double *, double *, int, int);
template __global__ void reduce<  1, false>(const double *, double *, int, int);
