#include "hip/hip_runtime.h"
#include "blas.h"

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

template <CBlasUplo uplo, unsigned int bx>
__device__ void spptf2(int n, float * __restrict__ A, int * __restrict__ info) {
  // thread 0 is the only thread to write to info in shared or global memory
  if (threadIdx.x == 0)
    *info = 0;  // initialise info to zero

  if (uplo == CBlasUpper) {
    // Perform the cholesky decomposition
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also avoids bank
    // conflicts.
    for (int j = 0; j < n; j++) {
      float temp;
      if (threadIdx.x >= j) {
        // SGEMV/SSYRK
        temp = A[upper(j, threadIdx.x)];
        for (int k = 0; k < j; k++)
          temp -= A[upper(k, j)] * A[upper(k, threadIdx.x)];

        // Thread j calculates the diagonal element
        if (threadIdx.x == j) {
          if (temp <= 0.0f || isnan(temp)) {
            *info = j + 1;
            A[upper(j, threadIdx.x)] = temp;
          }
          else
            A[upper(j, threadIdx.x)] = sqrtf(temp);
        }
      }

      __syncthreads();

      // If info != 0 return (matrix is not positive definite)
      if (*info != 0)
        return;

      // SSCAL
      if (threadIdx.x > j)
        A[upper(j, threadIdx.x)] = temp / A[upper(j, j)];

      __syncthreads();
    }
  }
  else {
    // Perform the cholesky decomposition
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also avoids bank
    // conflicts.
    for (int j = 0; j < n; j++) {
      float temp;
      if (threadIdx.x >= j) {
        // SGEMV/SSYRK
        temp = A[lower<bx>(threadIdx.x, j)];
        for (int k = 0; k < j; k++)
          temp -= A[lower<bx>(j, k)] * A[lower<bx>(threadIdx.x, k)];

        // Thread j calculates the diagonal element
        if (threadIdx.x == j) {
          if (temp <= 0.0f || isnan(temp)) {
            *info = j + 1;
            A[lower<bx>(threadIdx.x, j)] = temp;
          }
          else
            A[lower<bx>(threadIdx.x, j)] = sqrtf(temp);
        }
      }

      __syncthreads();

      // If info != 0 return (matrix is not positive definite)
      if (*info != 0)
        return;

      // SSCAL
      if (threadIdx.x > j)
        A[lower<bx>(threadIdx.x, j)] = temp / A[lower<bx>(j, j)];

      __syncthreads();
    }
  }
}

template <CBlasUplo uplo, unsigned int bx>
__global__ void spotf2(float * A, int * info, int lda, int n) {
  // info parameter cached in shared memory for fast access by all threads in the block
  __shared__ int sinfo;

  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 64 threads are needed but this would
   * use all 16384 bytes (64 * 64 * sizeof(float)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 8320 bytes((64 * (64 + 1)) / 2 * sizeof(float))
   * of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ float a[(bx * (bx + 1)) / 2];

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the cholesky decomposition using the packed device function
    spptf2<CBlasUpper, bx>(n, A, &sinfo);

    // Write info back to global memory
    if (threadIdx.x == 0)
      *info = sinfo;

    // Write the upper triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j)
        A[j * lda + threadIdx.x] = a[upper(threadIdx.x, j)];
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the cholesky decomposition using the packed device function
    spptf2<CBlasLower, bx>(n, A, &sinfo);

    // Write info back to global memory
    if (threadIdx.x == 0)
      *info = sinfo;

    // Write the lower triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        A[j * lda + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template __global__ void spotf2<CBlasUpper, 64>(float *, int *, int, int);
template __global__ void spotf2<CBlasLower, 64>(float *, int *, int, int);

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void spotf2_(const char *, const int *, float *, const int *, int *);
static inline void spotf2(CBlasUplo uplo, int n, float * A, int lda, int * info) {
  if (uplo == CBlasUpper)
    spotf2<CBlasUpper, 64><<<1,64>>>(A, info, lda, n);
  else
    spotf2<CBlasLower, 64><<<1,64>>>(A, info, lda, n);
}

static int cond(int, float, float *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  int n;

  if (argc != 3) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  if (sscanf(argv[2], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[2]);
    return 2;
  }

  float * A, * dA, * refA;
  size_t lda = (n + 3) & ~3, dlda;
  int * dinfo;
  if ((A = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (float *)malloc(lda * n * sizeof(float))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(float), n));
  CUDA_ERROR_CHECK(hipMalloc((void **)&dinfo, sizeof(int)));
  dlda /= sizeof(float);

  cond(n, 2.0f, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(float));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(float), A, lda * sizeof(float), n * sizeof(float), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  int info = 0, refInfo = 0;
  spotf2_((const char *)&uplo, &n, refA, (const int *)&lda, &refInfo);
  spotf2(uplo, n, dA, dlda, dinfo);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(float), dA, dlda * sizeof(float), n * sizeof(float), n, hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", refA[j * lda + i]);
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f", A[j * lda + i]);
    fprintf(stderr, "\n");
  }

  float error = 0.0f;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      float diff = fabsf(refA[j * lda + i] - A[j * lda + i]);
      if (diff > error)
        error = diff;
    }
  }

  fprintf(stdout, "Info = %d, refInfo = %d, Error = %6.3e\n", info, refInfo, error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));
  CUDA_ERROR_CHECK(hipFree(dinfo));

  return info;
}

static int cond(int n, float c, float * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  float * u, * v, * w;
  size_t offset = (n + 3u) & ~3u;

  if ((u = (float *)malloc(3 * offset * sizeof(float))) == NULL)
    return 1;

  v = &u[offset];
  w = &v[offset];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] = 0.0f;
  }

  A[0] = 1.0f;
  A[lda + 1] = c;
  for (int j = 2; j < n; j++)
    A[j * lda + j] = ((float) rand() / (float)RAND_MAX) * (c - 1.0f) + 1.0f;

  float t = 0.0f, s = 0.0f;
  for (int j = 0; j < n; j++) {
    // u is a random vector
    u[j] = (float)rand() / (float)RAND_MAX;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += u[j] * u[j];
    // s = t^2 u'v / 2
    s += u[j] * v[j];
  }
  t = 2.0f / t;
  s = t * t * s / 2.0f;

  // w = tv - su
  for (int j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * w[j] + w[i] * u[j];
  }

  free(u);

  return 0;
}
