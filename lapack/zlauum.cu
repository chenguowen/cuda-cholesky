#include "hip/hip_runtime.h"
#include "blas.h"
#include <hip/hip_complex.h>

/*
 * Indexing function for upper triangular packed storage mode.  Only works when
 * i <= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
__device__ int upper(int i, int j) {
  return ((j * (j + 1)) / 2) + i;
}

/*
 * Indexing function for lower triangular packed storage mode.  Only works when
 * i >= j otherwise generates an out-of-bounds access in shared memory and CUDA
 * will segfault.
 */
template <unsigned int bx>
__device__ int lower(int i, int j) {
  return ((2 * bx - j - 1) * j) / 2 + i;
}

template <CBlasUplo uplo, unsigned int bx>
__global__ void zlauu2(hipDoubleComplex * A, int lda, int n) {
  /*
   * For efficient data reuse A needs to be cached in shared memory.  In order
   * to get maximum instruction throughput 16 threads are needed but this would
   * use 4096 bytes (16 * 16 * sizeof(hipDoubleComplex)) of shared memory to store A.
   * Triangular packed storage mode is therefore used to store only the
   * triangle of A being updated using 2176 bytes((16 * (16 + 1)) / 2 *
   * sizeof(hipDoubleComplex)) of shared memory.
   * Since this is only ever going to be run using one thread block shared
   * memory and register use can be higher than when trying to fit multiple
   * thread blocks onto each multiprocessor.
   */
  __shared__ hipDoubleComplex a[(bx * (bx + 1)) / 2];

  if (uplo == CBlasUpper) {
    // Read upper triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x <= j)
        a[upper(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the cholesky decomposition
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int j = 0; j < n; j++) {
      if (threadIdx.x <= j) {
        hipDoubleComplex temp = hipCmul(a[upper(threadIdx.x, j)], hipConj(a[upper(j, j)]));
        for (int k = j + 1; k < n; k++)
          temp = hipCfma(a[upper(threadIdx.x, k)], hipConj(a[upper(j, k)]), temp);
        A[j * lda + threadIdx.x] = temp;
      }
      __syncthreads();
    }
  }
  else {
    // Read lower triangle of A into shared memory
    #pragma unroll
    for (int j = 0; j < bx; j++) {
      if (threadIdx.x >= j)
        a[lower<bx>(threadIdx.x, j)] = A[j * lda + threadIdx.x];
    }

    __syncthreads();

    // Perform the cholesky decomposition
    // Accesses do not have to be coalesced or aligned as they would if A were
    // in global memory.  Using triangular packed storage also neatly avoids
    // bank conflicts.
    for (int i = 0; i < n; i++) {
      if (threadIdx.x <= i) {
        hipDoubleComplex temp = hipCmul(a[lower<bx>(i, threadIdx.x)], hipConj(a[lower<bx>(i, i)]));
        for (int k = i + 1; k < n; k++)
          temp = hipCfma(hipConj(a[lower<bx>(k, i)]), a[lower<bx>(k, threadIdx.x)], temp);
        a[lower<bx>(i, threadIdx.x)] = temp;
      }
      __syncthreads();
    }

    // Write the lower triangle of A back to global memory
    for (int j = 0; j < n; j++) {
      if (threadIdx.x >= j)
        A[j * lda + threadIdx.x] = a[lower<bx>(threadIdx.x, j)];
    }
  }
}

template __global__ void zlauu2<CBlasUpper, 16>(hipDoubleComplex *, int, int);
template __global__ void zlauu2<CBlasLower, 16>(hipDoubleComplex *, int, int);

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <ctype.h>
#include <float.h>
#include <math.h>
#include <complex.h>

#define CUDA_ERROR_CHECK(call) \
  do { \
    hipError_t error = (call); \
    if (error != hipSuccess) { \
      fprintf(stderr, "CUDA Runtime error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(error)); \
      return error; \
    } \
  } while (false)

#define xerbla(info) \
  fprintf(stderr, "On entry to %s parameter %d had an invalid value\n", __func__, (info))

extern "C" void zlauu2_(const char *, const int *, void *, const int *, int *);
static inline void zlauu2(CBlasUplo uplo, int n, hipDoubleComplex * A, int lda) {
  if (uplo == CBlasUpper)
    zlauu2<CBlasUpper, 16><<<1,16>>>(A, lda, n);
  else
    zlauu2<CBlasLower, 16><<<1,16>>>(A, lda, n);
}

static int ccond(int, double, double complex *, size_t);

int main(int argc, char * argv[]) {
  CBlasUplo uplo;
  int n;

  if (argc != 3) {
    fprintf(stderr, "Usage %s <uplo> <diag> <n>\n"
                    "where:\n"
                    "  <uplo>  is 'U' or 'u' for CBlasUpper or 'L' or 'l' for CBlasLower\n"
                    "  <n>     is the size of the matrix\n", argv[0]);
    return -1;
  }

  char u;
  if (sscanf(argv[1], "%c", &u) != 1) {
    fprintf(stderr, "Unable to parse character from '%s'\n", argv[1]);
    return 1;
  }
  switch (u) {
    case 'u': case 'U': uplo = CBlasUpper; break;
    case 'l': case 'L': uplo = CBlasLower; break;
    default: fprintf(stderr, "Unknown uplo '%c'\n", u); return 1;
  }

  if (sscanf(argv[2], "%d", &n) != 1) {
    fprintf(stderr, "Unable to parse integer from '%s'\n", argv[2]);
    return 2;
  }

  double complex * A, * refA;
  hipDoubleComplex * dA;
  size_t lda = n, dlda;
  if ((A = (double complex *)malloc(lda * n * sizeof(double complex))) == NULL) {
    fprintf(stderr, "Failed to allocate A\n");
    return -1;
  }
  if ((refA = (double complex *)malloc(lda * n * sizeof(double complex))) == NULL) {
    fprintf(stderr, "Failed to allocate refA\n");
    return -2;
  }
  CUDA_ERROR_CHECK(hipMallocPitch((void **)&dA, &dlda, n * sizeof(hipDoubleComplex), n));
  dlda /= sizeof(hipDoubleComplex);

  ccond(n, 2.0, A, lda);

  for (int j = 0; j < n; j++)
    memcpy(&refA[j * lda], &A[j * lda], n * sizeof(double complex));

  CUDA_ERROR_CHECK(hipMemcpy2D(dA, dlda * sizeof(hipDoubleComplex), A, lda * sizeof(double complex), n * sizeof(hipDoubleComplex), n, hipMemcpyHostToDevice));

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", creal(A[j * lda + i]), cimag(A[j * lda + i]));
    fprintf(stderr, "\n");
  }

  int refInfo;
  zlauu2_((const char *)&uplo, &n, refA, (const int *)&lda, &refInfo);
  zlauu2(uplo, n, dA, dlda);
  CUDA_ERROR_CHECK(hipMemcpy2D(A, lda * sizeof(double complex), dA, dlda * sizeof(hipDoubleComplex), n * sizeof(hipDoubleComplex), n, hipMemcpyDeviceToHost));

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", creal(refA[j * lda + i]), cimag(refA[j * lda + i]));
    fprintf(stderr, "\n");
  }

  fprintf(stderr, "\n");
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++)
      fprintf(stderr, "%15.6f + %15.6fi", creal(A[j * lda + i]), cimag(A[j * lda + i]));
    fprintf(stderr, "\n");
  }

  double real_error = 0.0, imag_error = 0.0;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      double diff = fabs(creal(refA[j * lda + i]) - creal(A[j * lda + i]));
      if (diff > real_error)
        real_error = diff;
      diff = fabs(cimag(refA[j * lda + i]) - cimag(A[j * lda + i]));
      if (diff > imag_error)
        imag_error = diff;
    }
  }

  fprintf(stdout, "refInfo = %d, Error = %6.3e + %6.3ei\n", refInfo, real_error, imag_error);

  free(A);
  free(refA);
  CUDA_ERROR_CHECK(hipFree(dA));

  return refInfo;
}

static int ccond(int n, double c, double complex * A, size_t lda) {
  int info = 0;
  if (n < 2)
    info = -1;
  else if (c < 1.0)
    info = -2;
  else if (lda < n)
    info = -4;
  if (info != 0) {
    xerbla(-info);
    return info;
  }

  double complex * u, * v, * w;

  if ((u = (double complex *)malloc(3 * n * sizeof(double complex))) == NULL)
    return 1;

  v = &u[n];
  w = &v[n];

  // Initialise A as a diagonal matrix whose diagonal consists of numbers from
  // [1,c] with 1 and c chosen at least once (here in the top left)
  for (size_t j = 0; j < n; j++) {
    for (size_t i = 0; i < n; i++)
      A[j * lda + i] = 0.0 + 0.0 * I;
  }

  A[0] = 1.0 + 0.0 * I;
  A[lda + 1] = c + 0.0 * I;
  for (size_t j = 2; j < n; j++)
    A[j * lda + j] = ((double) rand() / (double)RAND_MAX) * (c - 1.0) + 1.0 + 0.0 * I;

  double t = 0.0;
  double complex s = 0.0 + 0.0 * I;
  for (size_t j = 0; j < n; j++) {
    // u is a complex precision random vector
    u[j] = ((double)rand() / (double)RAND_MAX) + ((double)rand() / (double)RAND_MAX) * I;
    // v = Au
    v[j] = A[j * lda + j] * u[j];
    // t = 2/u'u
    t += creal(conj(u[j]) * u[j]);
    // s = t^2 u'v / 2
    s += conj(u[j]) * v[j];
  }
  t = 2.0 / t;
  s = t * t * s / (2.0 + 0.0 * I);

  // w = tv - su
  for (size_t j = 0; j < n; j++)
    w[j] = t * v[j] - s * u[j];

  // A -= uw' + wu'
  for (size_t j = 0; j < n; j++) {
    for (size_t i = 0; i < n; i++)
      A[j * lda + i] -= u[i] * conj(w[j]) + w[i] * conj(u[j]);
  }

  free(u);

  return 0;
}
