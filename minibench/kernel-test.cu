#include "hip/hip_runtime.h"
#include <stdio.h>

#define ITERATIONS 1E6

#define CUDA_ERROR_CHECK(call) \
 do { \
   hipError_t __error__; \
   if ((__error__ = (call)) != hipSuccess) { \
     fprintf(stderr, "CUDA error: %s\n\t at %s(%s:%d)\n", hipGetErrorString(__error__), __func__, __FILE__, __LINE__); \
     return (int)__error__; \
   } \
 } while (false)

/**
 * This is a minimal kernel used to measure kernel launch overhead on the GPU.
 *
 * It copies one single precision 32-bit floating point value from one area of
 * global memory to another.
 */
extern "C" __global__ void kernel(const float * in, float * out) {
  *out = *in;
}

/**
 * This measures the overhead in launching a kernel function on each GPU in the
 * system.
 *
 * It does this by executing a small kernel (copying 1 value in global memory) a
 * very large number of times and taking the average execution time.  This
 * program uses the CUDA runtime API.
 */
int main() {
  int count;
  CUDA_ERROR_CHECK(hipGetDeviceCount(&count));

  float x = 5.0f;
  for (int d = 0; d < count; d++) {
    CUDA_ERROR_CHECK(hipSetDevice(d));

    float * in, * out;
    CUDA_ERROR_CHECK(hipMalloc((void **)&in, sizeof(float)));
    CUDA_ERROR_CHECK(hipMalloc((void **)&out, sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(in, &x, sizeof(float), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&stop));

    CUDA_ERROR_CHECK(hipEventRecord(start));
    for (int i = 0; i < ITERATIONS; i++)
      kernel<<<1, 1>>>(in, out);
    CUDA_ERROR_CHECK(hipEventRecord(stop));
    CUDA_ERROR_CHECK(hipEventSynchronize(stop));

    float time;
    CUDA_ERROR_CHECK(hipEventElapsedTime(&time, start, stop));

    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(stop));

    CUDA_ERROR_CHECK(hipFree(in));
    CUDA_ERROR_CHECK(hipFree(out));

    fprintf(stdout, "Device %d: %fms\n", d, (time / (double)ITERATIONS));

    CUDA_ERROR_CHECK(hipDeviceReset());
  }

  return 0;
}
