#include "matrix.cu"
#include "../hipComplex.cuh"

template void      all<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t, size_t);
template void identity<hipComplex<float> >(hipComplex<float> *, size_t, size_t, size_t);
template void           add<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t, size_t);
template void      addConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t, size_t);
template void      subtract<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t, size_t);
template void subtractConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t, size_t);
template void      multiply<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t, size_t);
template void multiplyConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t, size_t);
template void        divide<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t, size_t);
template void   divideConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t, size_t);
