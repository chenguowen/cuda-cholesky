#include "hip/hip_runtime.h"
#include "blas.h"

__device__ void saxpy(float alpha, const float * x, float * y) {
  y[ 0] += alpha * x[ 0]; y[ 1] += alpha * x[ 1]; y[ 2] += alpha * x[ 2]; y[ 3] += alpha * x[ 3];
  y[ 4] += alpha * x[ 4]; y[ 5] += alpha * x[ 5]; y[ 6] += alpha * x[ 6]; y[ 7] += alpha * x[ 7];
  y[ 8] += alpha * x[ 8]; y[ 9] += alpha * x[ 9]; y[10] += alpha * x[10]; y[11] += alpha * x[11];
  y[12] += alpha * x[12]; y[13] += alpha * x[13]; y[14] += alpha * x[14]; y[15] += alpha * x[15];
}

__device__ int min(int a, int b) { return (a < b) ? a : b; }

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void strmm2L(int m, int n,
                        float alpha, const float * __restrict__ A, int lda, const float * __restrict__ B, int ldb,
                        float * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;
  int tj = 0;
  if (trans != CBlasNoTrans) {
    tj = 16 * (ti / mb);
    ti = ti % mb;
  }

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? bi * lda + bi + ti : bi + ti;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + bi + threadIdx.x : (bj + threadIdx.y) * ldb + threadIdx.x;
  }
  else {
    A += (uplo == CBlasUpper) ? (bi + threadIdx.y) * lda + threadIdx.x : (bi + threadIdx.y) * lda + bi + threadIdx.x;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + threadIdx.x : (bj + threadIdx.y) * ldb + bi + threadIdx.x;
  }
  X += (bj + tj) * ldx + bi + ti;

  __shared__ float a[mb][kb + 1];
  __shared__ float b[kb][nb];

  float x[] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

  // For Upper/NoTrans and Lower/Trans process diagonal first
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {
    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int l = 0; l < kb; l += bx) {
#pragma unroll
          for (int i = 0; i < mb; i += by)
            a[i + threadIdx.y][l + threadIdx.x] = A[i * lda + l];
        }
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti <= l++)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti < l)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti <= l++)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti < l)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for SGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? m - bi - mb : bi)
                                  : ((uplo == CBlasUpper) ? bi : m - bi - mb);
  while (k > 0) {
    if (trans != CBlasNoTrans) {
#pragma unroll
      for (int l = 0; l < kb; l += bx) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][l + threadIdx.x] = A[i * lda + l];
      }
      A += kb;
    }

#pragma unroll
    for (int j = 0; j < nb; j += by)
      b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

    __syncthreads();

    if (k < kb) break;

    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int l = 0; l < kb; l++) {
        saxpy(A[0], b[l], x);
        A += lda;
      }
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l++)
        saxpy(a[ti][l], &b[l][tj], x);
    }

    __syncthreads();

    B += kb;
    k -= kb;
  }

  if (trans == CBlasNoTrans) {
    for (int l = 0; l < k; l++) {
      saxpy(A[0], b[l], x);
      A += lda;
    }
  }
  else {
    for (int l = 0; l < k; l++)
      saxpy(a[ti][l], &b[l][tj], x);
  }

  // For Upper/Trans and Lower/NoTrans process diagonal last
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {

    __syncthreads();

    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][threadIdx.x] = A[i * lda];
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti >= l++)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti > l)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti >= l++)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti > l)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }
  }

  n -= bj + tj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7]; if ( 8 >= n) return; X += ldx;
  X[0] = alpha * x[ 8]; if ( 9 >= n) return; X += ldx;
  X[0] = alpha * x[ 9]; if (10 >= n) return; X += ldx;
  X[0] = alpha * x[10]; if (11 >= n) return; X += ldx;
  X[0] = alpha * x[11]; if (12 >= n) return; X += ldx;
  X[0] = alpha * x[12]; if (13 >= n) return; X += ldx;
  X[0] = alpha * x[13]; if (14 >= n) return; X += ldx;
  X[0] = alpha * x[14]; if (15 >= n) return; X += ldx;
  X[0] = alpha * x[15];
}

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void strmm2R(int m, int n,
                        float alpha, const float * __restrict__ A, int lda, const float * __restrict__ B, int ldb,
                        float * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + threadIdx.x : (bj + threadIdx.y) * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bi + ti : bj * ldb + bi + ti;
  }
  else {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + bj + threadIdx.x : threadIdx.y * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bj * ldb + bi + ti : bi + ti;
  }
  X += bj * ldx + bi + ti;

  __shared__ float a[kb][nb];

  float x[] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

  // For Upper/Trans and Lower/NoTrans process diagonal first
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {
    int k = min(n - bj, nb);
    int l = 0;
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by)
          a[threadIdx.x][j + threadIdx.y] = A[j * lda];
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by)
          a[l + threadIdx.y][threadIdx.x] = A[l * lda];
      }

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti <= l++)
            saxpy(B[0], a[ll], x);
          B += ldb;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, a[ll], x);
          else if (ti < l)
            saxpy(B[0], a[ll], x);
          B += ldb;
          l++;
        }
      }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti <= l++)
          saxpy(B[0], a[ll], x);
        B += ldb;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, a[ll], x);
        else if (ti < l)
          saxpy(B[0], a[ll], x);
        B += ldb;
        l++;
      }
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for SGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? n - bj - nb : bj)
                                  : ((uplo == CBlasUpper) ? bj : n - bj - nb);
  while (k > 0) {
    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int j = 0; j < nb; j += by)
        a[threadIdx.x][j + threadIdx.y] = A[j * lda];
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l += by)
        a[l + threadIdx.y][threadIdx.x] = A[l * lda];
    }

    __syncthreads();

    if (k < kb) break;

#pragma unroll
    for (int l = 0; l < kb; l++) {
      saxpy(B[0], a[l], x);
      B += ldb;
    }

    __syncthreads();

    A += (trans == CBlasNoTrans) ? kb : kb * lda;
    k -= kb;
  }

  for (int l = 0; l < k; l++) {
    saxpy(B[0], a[l], x);
    B += ldb;
  }

  // For Upper/NoTrans and Lower/Trans process diagonal last
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {

    __syncthreads();

    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by)
          a[threadIdx.x][j + threadIdx.y] = A[j * lda];
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by)
          a[l + threadIdx.y][threadIdx.x] = A[l * lda];
      }

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti >= l++)
            saxpy(B[0], a[ll], x);
          B += ldb;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, a[ll], x);
          else if (ti > l)
            saxpy(B[0], a[ll], x);
          B += ldb;
          l++;
        }
      }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti >= l++)
          saxpy(B[0], a[ll], x);
        B += ldb;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, a[ll], x);
        else if (ti > l)
          saxpy(B[0], a[ll], x);
        B += ldb;
        l++;
      }
    }
  }

  n -= bj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7]; if ( 8 >= n) return; X += ldx;
  X[0] = alpha * x[ 8]; if ( 9 >= n) return; X += ldx;
  X[0] = alpha * x[ 9]; if (10 >= n) return; X += ldx;
  X[0] = alpha * x[10]; if (11 >= n) return; X += ldx;
  X[0] = alpha * x[11]; if (12 >= n) return; X += ldx;
  X[0] = alpha * x[12]; if (13 >= n) return; X += ldx;
  X[0] = alpha * x[13]; if (14 >= n) return; X += ldx;
  X[0] = alpha * x[14]; if (15 >= n) return; X += ldx;
  X[0] = alpha * x[15];
}

template void strmm2L<CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasUpper, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasUpper, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasTrans,   CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasTrans,   CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasTrans,   CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasTrans,   CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
