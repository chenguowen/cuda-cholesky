#include "hip/hip_runtime.h"
#include "blas.h"

__device__ void saxpy(float alpha, const float * x, float * y) {
  y[ 0] += alpha * x[ 0]; y[ 1] += alpha * x[ 1]; y[ 2] += alpha * x[ 2]; y[ 3] += alpha * x[ 3];
  y[ 4] += alpha * x[ 4]; y[ 5] += alpha * x[ 5]; y[ 6] += alpha * x[ 6]; y[ 7] += alpha * x[ 7];
  y[ 8] += alpha * x[ 8]; y[ 9] += alpha * x[ 9]; y[10] += alpha * x[10]; y[11] += alpha * x[11];
  y[12] += alpha * x[12]; y[13] += alpha * x[13]; y[14] += alpha * x[14]; y[15] += alpha * x[15];
}

__device__ int min(int a, int b) { return (a < b) ? a : b; }

template <CBlasSide side, CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void strmm2(int m, int n,
                       float alpha, const float * __restrict__ A, int lda, const float * __restrict__ B, int ldb,
                       float * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;
  int tj = 0;
  if (trans != CBlasNoTrans) {
    tj = 16 * (ti / mb);
    ti = ti % mb;
  }

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? bi * lda + bi + ti : bi + ti;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + bi + threadIdx.x : (bj + threadIdx.y) * ldb + threadIdx.x;
  }
  else {
    A += (uplo == CBlasUpper) ? (bi + threadIdx.y) * lda + threadIdx.x : (bi + threadIdx.y) * lda + bi + threadIdx.x;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + threadIdx.x : (bj + threadIdx.y) * ldb + ldb + bi + threadIdx.x;
  }
  X += (bj + tj) * ldx + bi + ti;

  __shared__ float a[mb][kb + 1];
  __shared__ float b[kb][nb];

  float x[] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

  // For Upper/NoTrans and Lower/Trans process diagonal first
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {
    int k = min(m, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][threadIdx.x] = A[i * lda];
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti <= l++)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti < l)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti <= l++)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti < l)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        A += lda;
        l++;
      }
    }
  }

  // Process non-diagonal blocks as for SGEMM
  int k = (uplo == CBlasUpper) ? m - bi - mb : bi;
  while (k > 0) {
    if (trans != CBlasNoTrans) {
#pragma unroll
      for (int i = 0; i < mb; i += by)
        a[i + threadIdx.y][threadIdx.x] = A[i * lda];
      A += kb;
    }

#pragma unroll
    for (int j = 0; j < nb; j += by)
      b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

    __syncthreads();

    if (k < kb) break;

#pragma unroll
    for (int l = 0; l < kb; l++) {
      saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][l], (trans == CBlasNoTrans) ? b[l] : &b[l][tj], x);
      A += lda;
    }

    __syncthreads();

    B += kb;
    k -= kb;
  }

  for (int l = 0; l < k; l++) {
    saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][l], (trans == CBlasNoTrans) ? b[l] : &b[l][tj], x);
    A += lda;
  }

  // For Upper/Trans and Lower/NoTrans process diagonal last
  if (uplo == CBlasLower) {
    int k = min(m, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][threadIdx.x] = A[i * lda];
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti >= l++)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti > l)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti >= l++)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti > l)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        A += lda;
        l++;
      }
    }
  }

  n -= bj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7]; if ( 8 >= n) return; X += ldx;
  X[0] = alpha * x[ 8]; if ( 9 >= n) return; X += ldx;
  X[0] = alpha * x[ 9]; if (10 >= n) return; X += ldx;
  X[0] = alpha * x[10]; if (11 >= n) return; X += ldx;
  X[0] = alpha * x[11]; if (12 >= n) return; X += ldx;
  X[0] = alpha * x[12]; if (13 >= n) return; X += ldx;
  X[0] = alpha * x[13]; if (14 >= n) return; X += ldx;
  X[0] = alpha * x[14]; if (15 >= n) return; X += ldx;
  X[0] = alpha * x[15];
}

template void strmm2<CBlasLeft,  CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2<CBlasLeft,  CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2<CBlasLeft,  CBlasUpper, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2<CBlasLeft,  CBlasUpper, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2<CBlasLeft,  CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2<CBlasLeft,  CBlasLower, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2<CBlasLeft,  CBlasLower, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2<CBlasLeft,  CBlasLower, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasUpper, CBlasTrans,   CBlasUnit,    64, 64, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasUpper, CBlasTrans,   CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasLower, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasLower, CBlasTrans,   CBlasUnit,    64, 64, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm2<CBlasRight, CBlasLower, CBlasTrans,   CBlasNonUnit, 64, 64, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
