#include "hip/hip_runtime.h"
#include "blas.h"

// y(1:16) += alpha * x(1:16)
__device__ void saxpy(float alpha, const float * x, float * y) {
  y[ 0] += alpha * x[ 0]; y[ 1] += alpha * x[ 1]; y[ 2] += alpha * x[ 2]; y[ 3] += alpha * x[ 3];
//   y[ 4] += alpha * x[ 4]; y[ 5] += alpha * x[ 5]; y[ 6] += alpha * x[ 6]; y[ 7] += alpha * x[ 7];
//   y[ 8] += alpha * x[ 8]; y[ 9] += alpha * x[ 9]; y[10] += alpha * x[10]; y[11] += alpha * x[11];
//   y[12] += alpha * x[12]; y[13] += alpha * x[13]; y[14] += alpha * x[14]; y[15] += alpha * x[15];
}

// y(1:16) = x(1:16)
// __device__ void scopy(const float * x, float * y) {
//   y[ 0] = x[ 0]; y[ 1] = x[ 1]; y[ 2] = x[ 2]; y[ 3] = x[ 3];
//   y[ 4] = x[ 4]; y[ 5] = x[ 5]; y[ 6] = x[ 6]; y[ 7] = x[ 7];
//   y[ 8] = x[ 8]; y[ 9] = x[ 9]; y[10] = x[10]; y[11] = x[11];
//   y[12] = x[12]; y[13] = x[13]; y[14] = x[14]; y[15] = x[15];
// }

/**
 * This implementation is out-of-place.  Calling with X = B results in undefined
 * behaviour.
 *
 * STRMM:
 *   X = alpha * A B   for side == CBlasLeft and trans == CBlasNoTrans
 *   X = alpha * A'B   for side == CBlasLeft and trans == CBlasTrans
 *   X = alpha * B A   for side == CBlasRight and trans == CBlasNoTrans
 *   X = alpha * B A'  for side == CBlasRight and trans == CBlasTrans
 *
 * A is unit or non-unit upper or lower triangular (16 cases in total).
 *
 * @param side   side A multiplies B from (AB or BA).
 * @param uplo   whether A is upper or lower triangular.
 * @param trans  transpose for A.
 * @param diag   whether A has a unit or non-unit diagonal.
 * @param mb     the number of rows in the block of B/X.
 * @param nb     the number of columns in the block of B/X.
 * @param kb     how far to unroll the inner loop.
 * @param bx     blockDim.x.
 * @param by     blockDim.y.
 */
template <CBlasSide side, CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void strmm(int m, int n,
                      float alpha, const float * __restrict__ A, int lda,
                      const float * __restrict__ B, int ldb,
                      float * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  const int ti = threadIdx.y * bx + threadIdx.x;        // Unwrapped thread index [0, bx * by]

  /*
   * Compute our starting points in A, B and X.
   *
   * For trans != CBlasNoTrans A is cached in shared memory so the unwrapped
   * thread index can be re-wrapped around mb when calculating X.
   *
   * If trans == CBlasNoTrans then bx * by == mb (checked later on) so there
   * doesn't need to be a separate check for trans == CBlasNoTrans in
   * calculating the start of X here.
   */
  if (side == CBlasLeft) {
    if (trans == CBlasNoTrans) {
      if (uplo == CBlasUpper) {
        /* Left, Upper, NoTrans */
        A += bi * lda + bi + ti;        // Start on the diagonal
        B += (bj + threadIdx.y) * ldb + bi + threadIdx.x;  // Start halfway down
      }
      else {
        /* Left, Lower, NoTrans */
        A += bi + ti;   // Start on the left
        B += (bj + threadIdx.y) * ldb + threadIdx.x;    // Start at the top
      }
      X += bj * ldx + bi + ti;
      m -= bi;
      n -= bj;
    }
    else {
      if (uplo == CBlasUpper) {
      }
      else {
      }
    }
  }
  else {
  }

  /*
   * Blocks of A and B in shared memory and X in registers.
   */
  // A is optimised away when side == CBlasLeft and trans == CBlasNoTrans
//   __shared__ float a[(side == CBlasLeft) ? mb     : kb]
//                     [(side == CBlasLeft) ? kb + 1 : ((trans == CBlasNoTrans) ? nb + 1 : nb)];
  // B is optimised away when side == CBlasRight
  __shared__ float b[kb][nb + 1];

  float x[] = { 0.0f, 0.0f, 0.0f, 0.0f };//, 0.0f, 0.0f, 0.0f, 0.0f,
//                 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

  int k = 0;
  while (k < m) {
    // B will always be "transposed" w.r.t. X so must always be cached in shared
    // memory (i.e. it is read along the K or N dimensions when M is the
    // dimension being expanded).
#pragma unroll
    for (int j = 0; j < nb; j += by)
      b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

    __syncthreads();

    if (k + kb > m) break;

    // Read A straight from global memory.
    if (diag == CBlasNonUnit) {
#pragma unroll
      for (int l = 0; l < kb; l++) {
        if ((uplo == CBlasUpper && k + l >= ti) ||
            (uplo == CBlasLower && k + l <= ti))
          saxpy(A[0], b[l], x);
        A += lda;
      }
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l++) {
        if (k + l == ti)
          saxpy(1.0f, b[l], x);
        if ((uplo == CBlasUpper && k + l > ti) ||
            (uplo == CBlasLower && k + l < ti))
          saxpy(A[0], b[l], x);
        A += lda;
      }
    }

    __syncthreads();

    B += kb;
    k += kb;
  }

  int kk = m - k;
  if (diag == CBlasNonUnit) {
    for (int l = 0; l < kk; l++) {
      if ((uplo == CBlasUpper && k + l >= ti) ||
          (uplo == CBlasLower && k + l <= ti))
        saxpy(A[0], b[l], x);
      A += lda;
    }
  }
  else {
    for (int l = 0; l < kk; l++) {
      if (k + l == ti)
        saxpy(1.0f, b[l], x);
      if ((uplo == CBlasUpper && k + l > ti) ||
          (uplo == CBlasLower && k + l < ti))
        saxpy(A[0], b[l], x);
      A += lda;
    }
  }

  if (n <= 0) return;
  if (ti < m) {
    X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
    X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
    X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
    X[0] = alpha * x[ 3]; /*if ( 4 >= n) return; X += ldx;
    X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
    X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
    X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
    X[0] = alpha * x[ 7]; if ( 8 >= n) return; X += ldx;
    X[0] = alpha * x[ 8]; if ( 9 >= n) return; X += ldx;
    X[0] = alpha * x[ 9]; if (10 >= n) return; X += ldx;
    X[0] = alpha * x[10]; if (11 >= n) return; X += ldx;
    X[0] = alpha * x[11]; if (12 >= n) return; X += ldx;
    X[0] = alpha * x[12]; if (13 >= n) return; X += ldx;
    X[0] = alpha * x[13]; if (14 >= n) return; X += ldx;
    X[0] = alpha * x[14]; if (15 >= n) return; X += ldx;
    X[0] = alpha * x[15];*/
  }
}

/**
 * For D = aAB + bC:
 *   mb must be a multiple of the warp size (32) and less than or equal to the
 *        maximum number of threads per block (512).
 *   nb must be less than or equal to 20 (registers start spilling to global
 *        memory after 20).
 *   kb must be a multiple of the half-warp size (16) and such that
 *        (nb + 1)*kb*sizeof(float) is less than the amount of shared memory
 *        available per block (16384 bytes).
 *
 * mb and nb must be selected such that the bandwidth reduction is greater than
 * the flop:word ratio of the GPU.  The bandwidth reduction for all valid values
 * of mb and nb can be calculated with the following loop (bash):
 * echo -n " mb\nb"; for nb in {1..20}; do printf "%6d" ${nb}; done; echo; for mb in {32..512..32}; do printf "%6d"  ${mb}; for nb in {1..20}; do printf "%6.2f" $(echo 2 / \(1/${mb} + 1/${nb}\) | bc -l); done; echo; done
 *
 * Sample output:
 *  mb\nb     1     2     3     4     5     6     7     8     9    10    11    12    13    14    15    16    17    18    19    20
 *     32  1.94  3.76  5.49  7.11  8.65 10.11 11.49 12.80 14.05 15.24 16.37 17.45 18.49 19.48 20.43 21.33 22.20 23.04 23.84 24.62
 *     64  1.97  3.88  5.73  7.53  9.28 10.97 12.62 14.22 15.78 17.30 18.77 20.21 21.61 22.97 24.30 25.60 26.86 28.10 29.30 30.48
 *     96  1.98  3.92  5.82  7.68  9.50 11.29 13.05 14.77 16.46 18.11 19.74 21.33 22.90 24.44 25.95 27.43 28.88 30.32 31.72 33.10
 *    128  1.98  3.94  5.86  7.76  9.62 11.46 13.27 15.06 16.82 18.55 20.26 21.94 23.60 25.24 26.85 28.44 30.01 31.56 33.09 34.59
 *    160  1.99  3.95  5.89  7.80  9.70 11.57 13.41 15.24 17.04 18.82 20.58 22.33 24.05 25.75 27.43 29.09 30.73 32.36 33.97 35.56
 *    192  1.99  3.96  5.91  7.84  9.75 11.64 13.51 15.36 17.19 19.01 20.81 22.59 24.35 26.10 27.83 29.54 31.23 32.91 34.58 36.23
 *    224  1.99  3.96  5.92  7.86  9.78 11.69 13.58 15.45 17.30 19.15 20.97 22.78 24.57 26.35 28.12 29.87 31.60 33.32 35.03 36.72
 *    256  1.99  3.97  5.93  7.88  9.81 11.73 13.63 15.52 17.39 19.25 21.09 22.93 24.74 26.55 28.34 30.12 31.88 33.64 35.37 37.10
 *    288  1.99  3.97  5.94  7.89  9.83 11.76 13.67 15.57 17.45 19.33 21.19 23.04 24.88 26.70 28.51 30.32 32.10 33.88 35.65 37.40
 *    320  1.99  3.98  5.94  7.90  9.85 11.78 13.70 15.61 17.51 19.39 21.27 23.13 24.98 26.83 28.66 30.48 32.28 34.08 35.87 37.65
 *    352  1.99  3.98  5.95  7.91  9.86 11.80 13.73 15.64 17.55 19.45 21.33 23.21 25.07 26.93 28.77 30.61 32.43 34.25 36.05 37.85
 *    384  1.99  3.98  5.95  7.92  9.87 11.82 13.75 15.67 17.59 19.49 21.39 23.27 25.15 27.02 28.87 30.72 32.56 34.39 36.21 38.02
 *    416  2.00  3.98  5.96  7.92  9.88 11.83 13.77 15.70 17.62 19.53 21.43 23.33 25.21 27.09 28.96 30.81 32.67 34.51 36.34 38.17
 *    448  2.00  3.98  5.96  7.93  9.89 11.84 13.78 15.72 17.65 19.56 21.47 23.37 25.27 27.15 29.03 30.90 32.76 34.61 36.45 38.29
 *    480  2.00  3.98  5.96  7.93  9.90 11.85 13.80 15.74 17.67 19.59 21.51 23.41 25.31 27.21 29.09 30.97 32.84 34.70 36.55 38.40
 *    512  2.00  3.98  5.97  7.94  9.90 11.86 13.81 15.75 17.69 19.62 21.54 23.45 25.36 27.25 29.15 31.03 32.91 34.78 36.64 38.50
 *
 * The number of registers per block is mb*32 (compiled with -maxrregcount=32).
 * More threads == better performance (from flop-test) therefore mb is chosen to
 * be the largest number of threads such that the number of blocks per
 * multiprocessor is still limited by the register usage.
 * kb is chosen to be the largest multiple of 16 such that the number of blocks
 * per multiprocessor is limited by the register usage.
 */
template void strmm<CBlasLeft,  CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm<CBlasLeft,  CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasLeft,  CBlasUpper, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasLeft,  CBlasUpper, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm<CBlasLeft,  CBlasLower, CBlasNoTrans, CBlasNonUnit, 8, 4, 4, 4,  2>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm<CBlasLeft,  CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasLeft,  CBlasLower, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasLeft,  CBlasLower, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasUpper, CBlasTrans,   CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasUpper, CBlasTrans,   CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasLower, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasLower, CBlasTrans,   CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
// template void strmm<CBlasRight, CBlasLower, CBlasTrans,   CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
