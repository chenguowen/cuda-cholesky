#include "hip/hip_runtime.h"
#include "blas.h"

// y(1:16) += alpha * x(1:16)
__device__ void saxpy(float alpha, const float * x, float * y) {
  y[ 0] += alpha * x[ 0]; y[ 1] += alpha * x[ 1]; y[ 2] += alpha * x[ 2]; y[ 3] += alpha * x[ 3];
  y[ 4] += alpha * x[ 4]; y[ 5] += alpha * x[ 5]; y[ 6] += alpha * x[ 6]; y[ 7] += alpha * x[ 7];
  y[ 8] += alpha * x[ 8]; y[ 9] += alpha * x[ 9]; y[10] += alpha * x[10]; y[11] += alpha * x[11];
  y[12] += alpha * x[12]; y[13] += alpha * x[13]; y[14] += alpha * x[14]; y[15] += alpha * x[15];
}

// y(1:n) += alpha * x(1:n)
__device__ void saxpy(int n, float alpha, const float * x, float * y) {
  y[ 0] += alpha * x[ 0]; if ( 1 >= n) return; y[ 1] += alpha * x[ 1]; if ( 2 >= n) return;
  y[ 2] += alpha * x[ 2]; if ( 3 >= n) return; y[ 3] += alpha * x[ 3]; if ( 4 >= n) return;
  y[ 4] += alpha * x[ 4]; if ( 5 >= n) return; y[ 5] += alpha * x[ 5]; if ( 6 >= n) return;
  y[ 6] += alpha * x[ 6]; if ( 7 >= n) return; y[ 7] += alpha * x[ 7]; if ( 8 >= n) return;
  y[ 8] += alpha * x[ 8]; if ( 9 >= n) return; y[ 9] += alpha * x[ 9]; if (10 >= n) return;
  y[10] += alpha * x[10]; if (11 >= n) return; y[11] += alpha * x[11]; if (12 >= n) return;
  y[12] += alpha * x[12]; if (13 >= n) return; y[13] += alpha * x[13]; if (14 >= n) return;
  y[14] += alpha * x[14]; if (15 >= n) return; y[15] += alpha * x[15];
}

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void strmm2L(int m, int n,
                        float alpha, const float * __restrict__ A, int lda, const float * __restrict__ B, int ldb,
                        float * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;
  int tj = 0;
  if (trans != CBlasNoTrans) {
    tj = 16 * (ti / mb);
    ti = ti % mb;
  }

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? bi * lda + bi + ti : bi + ti;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + bi + threadIdx.x : (bj + threadIdx.y) * ldb + threadIdx.x;
  }
  else {
    A += (uplo == CBlasUpper) ? (bi + threadIdx.y) * lda + threadIdx.x : (bi + threadIdx.y) * lda + bi + threadIdx.x;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + threadIdx.x : (bj + threadIdx.y) * ldb + bi + threadIdx.x;
  }
  X += (bj + tj) * ldx + bi + ti;

  __shared__ float a[mb][kb + 1];
  __shared__ float b[kb][nb];

  float x[] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

  // For Upper/NoTrans and Lower/Trans process diagonal first
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {
    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int l = 0; l < kb; l += bx) {
#pragma unroll
          for (int i = 0; i < mb; i += by)
            a[i + threadIdx.y][l + threadIdx.x] = A[i * lda + l];
        }
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti <= l++)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti < l)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti <= l++)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti < l)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for SGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? m - bi - mb : bi)
                                  : ((uplo == CBlasUpper) ? bi : m - bi - mb);
  while (k > 0) {
    if (trans != CBlasNoTrans) {
#pragma unroll
      for (int l = 0; l < kb; l += bx) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][l + threadIdx.x] = A[i * lda + l];
      }
      A += kb;
    }

#pragma unroll
    for (int j = 0; j < nb; j += by)
      b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

    __syncthreads();

    if (k < kb) break;

    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int l = 0; l < kb; l++) {
        saxpy(A[0], b[l], x);
        A += lda;
      }
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l++)
        saxpy(a[ti][l], &b[l][tj], x);
    }

    __syncthreads();

    B += kb;
    k -= kb;
  }

  if (trans == CBlasNoTrans) {
    for (int l = 0; l < k; l++) {
      saxpy(A[0], b[l], x);
      A += lda;
    }
  }
  else {
    for (int l = 0; l < k; l++)
      saxpy(a[ti][l], &b[l][tj], x);
  }

  // For Upper/Trans and Lower/NoTrans process diagonal last
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {

    __syncthreads();

    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][threadIdx.x] = A[i * lda];
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti >= l++)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti > l)
            saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti >= l++)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          saxpy(1.0f, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti > l)
          saxpy((trans == CBlasNoTrans) ? A[0] : a[ti][ll], (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }
  }

  n -= bj + tj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7]; if ( 8 >= n) return; X += ldx;
  X[0] = alpha * x[ 8]; if ( 9 >= n) return; X += ldx;
  X[0] = alpha * x[ 9]; if (10 >= n) return; X += ldx;
  X[0] = alpha * x[10]; if (11 >= n) return; X += ldx;
  X[0] = alpha * x[11]; if (12 >= n) return; X += ldx;
  X[0] = alpha * x[12]; if (13 >= n) return; X += ldx;
  X[0] = alpha * x[13]; if (14 >= n) return; X += ldx;
  X[0] = alpha * x[14]; if (15 >= n) return; X += ldx;
  X[0] = alpha * x[15];
}

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void strmm2R(int m, int n,
                        float alpha, const float * __restrict__ A, int lda, const float * __restrict__ B, int ldb,
                        float * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + threadIdx.x : (bj + threadIdx.y) * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bi + ti : bj * ldb + bi + ti;
  }
  else {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + bj + threadIdx.x : threadIdx.y * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bj * ldb + bi + ti : bi + ti;
  }
  X += bj * ldx + bi + ti;

  __shared__ float a[kb][nb];

  float x[] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
                0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

  // For Upper/Trans and Lower/NoTrans process diagonal first
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {
    int k = min(n - bj, nb);
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by)
          a[threadIdx.x][j + threadIdx.y] = (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? 1.0f : A[j * lda];
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by)
          a[l + threadIdx.y][threadIdx.x] = (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? 1.0f : A[l * lda];
      }

      __syncthreads();

      if (k < kb) break;

// #pragma unroll
//       for (int ll = 0; ll < kb; ll++) {
//         saxpy(ll + 1, B[0], a[ll], x);
        saxpy( 1, B[0], a[ 0], x); B += ldb;
        saxpy( 2, B[0], a[ 1], x); B += ldb;
        saxpy( 3, B[0], a[ 2], x); B += ldb;
        saxpy( 4, B[0], a[ 3], x); B += ldb;
        saxpy( 5, B[0], a[ 4], x); B += ldb;
        saxpy( 6, B[0], a[ 5], x); B += ldb;
        saxpy( 7, B[0], a[ 6], x); B += ldb;
        saxpy( 8, B[0], a[ 7], x); B += ldb;
        saxpy( 9, B[0], a[ 8], x); B += ldb;
        saxpy(10, B[0], a[ 9], x); B += ldb;
        saxpy(11, B[0], a[10], x); B += ldb;
        saxpy(12, B[0], a[11], x); B += ldb;
        saxpy(13, B[0], a[12], x); B += ldb;
        saxpy(14, B[0], a[13], x); B += ldb;
        saxpy(15, B[0], a[14], x); B += ldb;
        saxpy(16, B[0], a[15], x); B += ldb;
//         B += ldb;
//       }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

    for (int ll = 0; ll < k; ll++) {
      saxpy(ll + 1, B[0], a[ll], x);
      B += ldb;
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for SGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? bj : n - bj - nb)
                                  : ((uplo == CBlasUpper) ? n - bj - nb : bj);
  while (k > 0) {
    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int j = 0; j < nb; j += by)
        a[threadIdx.x][j + threadIdx.y] = A[j * lda];
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l += by)
        a[l + threadIdx.y][threadIdx.x] = A[l * lda];
    }

    __syncthreads();

    if (k < kb) break;

#pragma unroll
    for (int l = 0; l < kb; l++) {
      saxpy(B[0], a[l], x);
      B += ldb;
    }

    __syncthreads();

    A += (trans == CBlasNoTrans) ? kb : kb * lda;
    k -= kb;
  }

  for (int l = 0; l < k; l++) {
    saxpy(B[0], a[l], x);
    B += ldb;
  }

  // For Upper/NoTrans and Lower/Trans process diagonal last
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {

    __syncthreads();

    int k = min(n - bj, nb);
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by)
          a[threadIdx.x][j + threadIdx.y] = (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? 1.0f : A[j * lda];
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by)
          a[l + threadIdx.y][threadIdx.x] = (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? 1.0f : A[l * lda];
      }

      __syncthreads();

      if (k < kb) break;

// #pragma unroll
//       for (int ll = 0; ll < kb; ll++) {
//         saxpy(nb - ll, B[0], &a[ll][ll], &x[ll]);
        saxpy(16, B[0], &a[ 0][ 0], &x[ 0]); B += ldb;
        saxpy(15, B[0], &a[ 1][ 1], &x[ 1]); B += ldb;
        saxpy(14, B[0], &a[ 2][ 2], &x[ 2]); B += ldb;
        saxpy(13, B[0], &a[ 3][ 3], &x[ 3]); B += ldb;
        saxpy(12, B[0], &a[ 4][ 4], &x[ 4]); B += ldb;
        saxpy(11, B[0], &a[ 5][ 5], &x[ 5]); B += ldb;
        saxpy(10, B[0], &a[ 6][ 6], &x[ 6]); B += ldb;
        saxpy( 9, B[0], &a[ 7][ 7], &x[ 7]); B += ldb;
        saxpy( 8, B[0], &a[ 8][ 8], &x[ 8]); B += ldb;
        saxpy( 7, B[0], &a[ 9][ 9], &x[ 9]); B += ldb;
        saxpy( 6, B[0], &a[10][10], &x[10]); B += ldb;
        saxpy( 5, B[0], &a[11][11], &x[11]); B += ldb;
        saxpy( 4, B[0], &a[12][12], &x[12]); B += ldb;
        saxpy( 3, B[0], &a[13][13], &x[13]); B += ldb;
        saxpy( 2, B[0], &a[14][14], &x[14]); B += ldb;
        saxpy( 1, B[0], &a[15][15], &x[15]); B += ldb;
//         B += ldb;
//       }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

//     for (int ll = 0; ll < k; ll++) {
//       saxpy(nb - ll, B[0], &a[ll][ll], &x[ll]);
//       B += ldb;
//     }
    if (k > 0) { saxpy(16, B[0], &a[ 0][ 0], &x[ 0]); B += ldb;
    if (k > 1) { saxpy(15, B[0], &a[ 1][ 1], &x[ 1]); B += ldb;
    if (k > 2) { saxpy(14, B[0], &a[ 2][ 2], &x[ 2]); B += ldb;
    if (k > 3) { saxpy(13, B[0], &a[ 3][ 3], &x[ 3]); B += ldb;
    if (k > 4) { saxpy(12, B[0], &a[ 4][ 4], &x[ 4]); B += ldb;
    if (k > 5) { saxpy(11, B[0], &a[ 5][ 5], &x[ 5]); B += ldb;
    if (k > 6) { saxpy(10, B[0], &a[ 6][ 6], &x[ 6]); B += ldb;
    if (k > 7) { saxpy( 9, B[0], &a[ 7][ 7], &x[ 7]); B += ldb;
    if (k > 8) { saxpy( 8, B[0], &a[ 8][ 8], &x[ 8]); B += ldb;
    if (k > 9) { saxpy( 7, B[0], &a[ 9][ 9], &x[ 9]); B += ldb;
    if (k >10) { saxpy( 6, B[0], &a[10][10], &x[10]); B += ldb;
    if (k >11) { saxpy( 5, B[0], &a[11][11], &x[11]); B += ldb;
    if (k >12) { saxpy( 4, B[0], &a[12][12], &x[12]); B += ldb;
    if (k >13) { saxpy( 3, B[0], &a[13][13], &x[13]); B += ldb;
    if (k >14) { saxpy( 2, B[0], &a[14][14], &x[14]); B += ldb;
    if (k >15) { saxpy( 1, B[0], &a[15][15], &x[15]); }}}}}}}}}}}}}}}}
  }

  n -= bj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7]; if ( 8 >= n) return; X += ldx;
  X[0] = alpha * x[ 8]; if ( 9 >= n) return; X += ldx;
  X[0] = alpha * x[ 9]; if (10 >= n) return; X += ldx;
  X[0] = alpha * x[10]; if (11 >= n) return; X += ldx;
  X[0] = alpha * x[11]; if (12 >= n) return; X += ldx;
  X[0] = alpha * x[12]; if (13 >= n) return; X += ldx;
  X[0] = alpha * x[13]; if (14 >= n) return; X += ldx;
  X[0] = alpha * x[14]; if (15 >= n) return; X += ldx;
  X[0] = alpha * x[15];
}

template void strmm2L<CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasUpper, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasUpper, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasTrans,   CBlasUnit,    32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2L<CBlasLower, CBlasTrans,   CBlasNonUnit, 32, 32,  8,  8,  8>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasTrans,   CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasUpper, CBlasTrans,   CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasNoTrans, CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasNoTrans, CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasTrans,   CBlasUnit,    64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
template void strmm2R<CBlasLower, CBlasTrans,   CBlasNonUnit, 64, 16, 16, 16,  4>(int, int, float, const float * __restrict__, int, const float * __restrict__, int, float * __restrict__, int);
