#include "hip/hip_runtime.h"
#include "blas.h"
#include <hip/hip_complex.h>

#if __CUDA_ARCH__ < 200
__device__ void caxpy(hipComplex a, float * b_real, float * b_imag, hipComplex * c) {
  c[0] = hipCfmaf(a, make_hipComplex(b_real[0], b_imag[0]), c[0]);
  c[1] = hipCfmaf(a, make_hipComplex(b_real[1], b_imag[1]), c[1]);
  c[2] = hipCfmaf(a, make_hipComplex(b_real[2], b_imag[2]), c[2]);
  c[3] = hipCfmaf(a, make_hipComplex(b_real[3], b_imag[3]), c[3]);
  c[4] = hipCfmaf(a, make_hipComplex(b_real[4], b_imag[4]), c[4]);
  c[5] = hipCfmaf(a, make_hipComplex(b_real[5], b_imag[5]), c[5]);
  c[6] = hipCfmaf(a, make_hipComplex(b_real[6], b_imag[6]), c[6]);
  c[7] = hipCfmaf(a, make_hipComplex(b_real[7], b_imag[7]), c[7]);
}
#else
__device__ void caxpy(hipComplex a, hipComplex * b, hipComplex * c) {
  c[0] = hipCfmaf(a, b[0], c[0]); c[1] = hipCfmaf(a, b[1], c[1]);
  c[2] = hipCfmaf(a, b[2], c[2]); c[3] = hipCfmaf(a, b[3], c[3]);
  c[4] = hipCfmaf(a, b[4], c[4]); c[5] = hipCfmaf(a, b[5], c[5]);
  c[6] = hipCfmaf(a, b[6], c[6]); c[7] = hipCfmaf(a, b[7], c[7]);
}
#endif

/**
 *
 * @param transA  transpose for A.
 * @param transB  transpose for B.
 * @param mb      the number of rows in the block of C.
 * @param nb      the number of columns in the block of C.
 * @param kb      how far to unroll the inner loop.
 * @param bx      blockDim.x.
 * @param by      blockDim.y.
 */
template <CBlasTranspose transA, CBlasTranspose transB,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void cgemm(int m, int n, int k, hipComplex alpha, const hipComplex * A, int lda,
                      const hipComplex * B, int ldb, hipComplex beta, hipComplex * C, int ldc) {

  const int bi = blockIdx.x * mb;       // Starting row of block of C
  const int bj = blockIdx.y * nb;       // Starting column of block of C
  const int ti = threadIdx.y * bx + threadIdx.x;        // Unwrapped thread index [0, bx * by]

  /*
   * Compute our starting points in A, B and C.
   *
   * For transA != CBlasNoTrans A is cached in shared memory so the unwrapped
   * thread index can be re-wrapped around mb when calculating C.
   *
   * If transA == CBlasNoTrans then bx * by == mb (checked later on) so there
   * doesn't need to be a separate check for transA == CBlasNoTrans in
   * calculating the start of C here.
   */
  A += (transA == CBlasNoTrans) ? bi + ti : (bi + threadIdx.y) * lda + threadIdx.x;
  B += (transB == CBlasNoTrans) ? (bj + threadIdx.y) * ldb + threadIdx.x : threadIdx.y * ldb + bj + threadIdx.x;
  C += (bx * by == mb) ? bj * ldc + bi + ti : (bj + 8 * (ti / mb)) * ldc + bi + ti % mb;

  /*
   * Blocks of A and B in shared memory and C in registers.
   */
#if __CUDA_ARCH__ < 200
  __shared__ float a_real[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ float a_imag[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ float b_real[kb][(transB == CBlasNoTrans) ? nb + 1 : nb];
  __shared__ float b_imag[kb][(transB == CBlasNoTrans) ? nb + 1 : nb];
#else
  __shared__ hipComplex a[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ hipComplex b[kb][(transB == CBlasNoTrans) ? nb + 1 : nb];
#endif

  hipComplex c[] = { { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f },
                    { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f } };

  while (k > 0) {
    // If A is to be transposed cache it in shared memory
    if (transA != CBlasNoTrans) {
//       typedef char x[(kb % bx == 0) ? 1 : -1];  // bx must be a multiple of kb
//       typedef char y[(mb % by == 0) ? 1 : -1];  // by must be a multiple of mb
      // If bx or by is equal to kb or mb then nvcc will optimise one of these
      // loops away.  This is the source of the "warning: expression has no
      // effect" compiler messages.
      if (transA == CBlasConjTrans) {
#pragma unroll
        for (int l = 0; l < kb; l += bx) {
#pragma unroll
          for (int i = 0; i < mb; i += by) {
#if __CUDA_ARCH__ < 200
            a_real[i + threadIdx.y][l + threadIdx.x] =  hipCrealf(A[i * lda + l]);
            a_imag[i + threadIdx.y][l + threadIdx.x] = -hipCimagf(A[i * lda + l]);
#else
            a[i + threadIdx.y][l + threadIdx.x] = hipConjf(A[i * lda + l]);
#endif
          }
        }
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += bx) {
#pragma unroll
          for (int i = 0; i < mb; i += by) {
#if __CUDA_ARCH__ < 200
            a_real[i + threadIdx.y][l + threadIdx.x] = hipCrealf(A[i * lda + l]);
            a_imag[i + threadIdx.y][l + threadIdx.x] = hipCimagf(A[i * lda + l]);
#else
            a[i + threadIdx.y][l + threadIdx.x] = A[i * lda + l];
#endif
          }
        }
      }
      A += kb;
    }

    // B will always be "transposed" w.r.t. C so must always be cached in shared
    // memory (i.e. it is read along the K or N dimensions when M is the
    // dimension being expanded).
    if (transB == CBlasNoTrans) {
//       typedef char x[(kb % bx == 0) ? 1 : -1];  // bx must be a multiple of kb
//       typedef char y[(nb % by == 0) ? 1 : -1];  // by must be a multiple of nb
#pragma unroll
      for (int l = 0; l < kb; l += bx) {
#pragma unroll
        for (int j = 0; j < nb; j += by) {
#if __CUDA_ARCH__ < 200
          b_real[l + threadIdx.x][j + threadIdx.y] = hipCrealf(B[j * ldb + l]);
          b_imag[l + threadIdx.x][j + threadIdx.y] = hipCimagf(B[j * ldb + l]);
#else
          b[l + threadIdx.x][j + threadIdx.y] = B[j * ldb + l];
#endif
        }
      }
    }
    else if (transB == CBlasConjTrans) {
//       typedef char x[(nb % bx == 0) ? 1 : -1];  // bx must be a multiple of nb
//       typedef char y[(kb % by == 0) ? 1 : -1];  // by must be a multiple of kb
#pragma unroll
      for (int l = 0; l < kb; l += by) {
#pragma unroll
        for (int j = 0; j < nb; j += bx) {
#if __CUDA_ARCH__ < 200
          b_real[l + threadIdx.y][j + threadIdx.x] =  hipCrealf(B[l * ldb + j]);
          b_imag[l + threadIdx.y][j + threadIdx.x] = -hipCimagf(B[l * ldb + j]);
#else
          b[l + threadIdx.y][j + threadIdx.x] = hipConjf(B[l * ldb + j]);
#endif
        }
      }
    }
    else {
//       typedef char x[(nb % bx == 0) ? 1 : -1];  // bx must be a multiple of nb
//       typedef char y[(kb % by == 0) ? 1 : -1];  // by must be a multiple of kb
#pragma unroll
      for (int l = 0; l < kb; l += by) {
#pragma unroll
        for (int j = 0; j < nb; j += bx) {
#if __CUDA_ARCH__ < 200
          b_real[l + threadIdx.y][j + threadIdx.x] = hipCrealf(B[l * ldb + j]);
          b_imag[l + threadIdx.y][j + threadIdx.x] = hipCimagf(B[l * ldb + j]);
#else
          b[l + threadIdx.y][j + threadIdx.x] = B[l * ldb + j];
#endif
        }
      }
    }

    __syncthreads();

    if (k < kb) break;

    if (transA == CBlasNoTrans) {
      // Read A straight from global memory.
//       typedef char x[(bx * by == mb) ? 1 : -1]; // There must be mb unrolled threads
//       typedef char y[(nb == 8) ? 1 : -1]; // nb must equal the size of row per thread
#pragma unroll
      for (int l = 0; l < kb; l++) {
#if __CUDA_ARCH__ < 200
        caxpy(A[0], b_real[l], b_imag[l], c);
#else
        caxpy(A[0], b[l], c);
#endif
        A += lda;
      }
    }
    else {
      // Read A from shared memory.
      // Need to check for thread wrapping so that the correct column of A is
      // matched with the correct row/column of B.
//       typedef char x[(bx * by % mb == 0) ? 1 : -1];     // bx * by must be a multiple of mb
//       typedef char y[((bx * by * 8) / mb == nb) ? 1 : -1];     // when the threads are wrapped around mb they must spread along to nb
#pragma unroll
      for (int l = 0; l < kb; l++)
#if __CUDA_ARCH__ < 200
        caxpy(make_hipComplex(a_real[(bx * by == mb) ? ti : ti % mb][l],
                             a_imag[(bx * by == mb) ? ti : ti % mb][l]),
              &b_real[l][(bx * by == mb) ? 0 : 8 * (ti / mb)],
              &b_imag[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], c);
#else
        caxpy(a[(bx * by == mb) ? ti : ti % mb][l],
              &b[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], c);
#endif
    }

    __syncthreads();

    B += (transB == CBlasNoTrans) ? kb : kb * ldb;
    k -= kb;
  }

  if (transA == CBlasNoTrans) {
    for (int l = 0; l < k; l++) {
#if __CUDA_ARCH__ < 200
      caxpy(A[0], b_real[l], b_imag[l], c);
#else
      caxpy(A[0], b[l], c);
#endif
      A += lda;
    }
  }
  else {
    for (int l = 0; l < k; l++)
#if __CUDA_ARCH__ < 200
      caxpy(make_hipComplex(a_real[(bx * by == mb) ? ti : ti % mb][l],
                           a_imag[(bx * by == mb) ? ti : ti % mb][l]),
            &b_real[l][(bx * by == mb) ? 0 : 8 * (ti / mb)],
            &b_imag[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], c);
#else
      caxpy(a[(bx * by == mb) ? ti : ti % mb][l],
            &b[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], c);
#endif
  }

  if (bx * by == mb)
    n -= bj;
  else {
    n -= bj + 8 * (ti / mb);
    if (n == 0) return;
  }
  if ((bx * by == mb && bi + ti < m) || (bx * by > mb && bi + ti % mb < m)) {
    if (hipCrealf(beta) == 0.0f && hipCimagf(beta) == 0.0f) {
      C[0] = hipCmulf(alpha, c[0]); if (1 >= n) return; C += ldc;
      C[0] = hipCmulf(alpha, c[1]); if (2 >= n) return; C += ldc;
      C[0] = hipCmulf(alpha, c[2]); if (3 >= n) return; C += ldc;
      C[0] = hipCmulf(alpha, c[3]); if (4 >= n) return; C += ldc;
      C[0] = hipCmulf(alpha, c[4]); if (5 >= n) return; C += ldc;
      C[0] = hipCmulf(alpha, c[5]); if (6 >= n) return; C += ldc;
      C[0] = hipCmulf(alpha, c[6]); if (7 >= n) return; C += ldc;
      C[0] = hipCmulf(alpha, c[7]);
    }
    else {
      C[0] = hipCfmaf(alpha, c[0], hipCmulf(beta, C[0])); if (1 >= n) return; C += ldc;
      C[0] = hipCfmaf(alpha, c[1], hipCmulf(beta, C[0])); if (2 >= n) return; C += ldc;
      C[0] = hipCfmaf(alpha, c[2], hipCmulf(beta, C[0])); if (3 >= n) return; C += ldc;
      C[0] = hipCfmaf(alpha, c[3], hipCmulf(beta, C[0])); if (4 >= n) return; C += ldc;
      C[0] = hipCfmaf(alpha, c[4], hipCmulf(beta, C[0])); if (5 >= n) return; C += ldc;
      C[0] = hipCfmaf(alpha, c[5], hipCmulf(beta, C[0])); if (6 >= n) return; C += ldc;
      C[0] = hipCfmaf(alpha, c[6], hipCmulf(beta, C[0])); if (7 >= n) return; C += ldc;
      C[0] = hipCfmaf(alpha, c[7], hipCmulf(beta, C[0]));
    }
  }
}

/**
 * For C = aAB + bC:
 *   mb must be a multiple of the warp size (32) and less than or equal to the
 *        maximum number of threads per block (512).
 *   nb must be less than or equal to 20 (registers start spilling to global
 *        memory after 20).
 *   kb must be a multiple of the half-warp size (16) and such that
 *        (nb + 1)*kb*sizeof(float) is less than the amount of shared memory
 *        available per block (16384 bytes).
 *
 * mb and nb must be selected such that the bandwidth reduction is greater than
 * the flop:word ratio of the GPU.  The bandwidth reduction for all valid values
 * of mb and nb can be calculated with the following loop (bash):
 * echo -n " mb\nb"; for nb in {1..20}; do printf "%6d" ${nb}; done; echo; for mb in {32..512..32}; do printf "%6d"  ${mb}; for nb in {1..20}; do printf "%6.2f" $(echo 2 / \(1/${mb} + 1/${nb}\) | bc -l); done; echo; done
 *
 * Sample output:
 *  mb\nb     1     2     3     4     5     6     7     8     9    10    11    12    13    14    15    16    17    18    19    20
 *     32  1.94  3.76  5.49  7.11  8.65 10.11 11.49 12.80 14.05 15.24 16.37 17.45 18.49 19.48 20.43 21.33 22.20 23.04 23.84 24.62
 *     64  1.97  3.88  5.73  7.53  9.28 10.97 12.62 14.22 15.78 17.30 18.77 20.21 21.61 22.97 24.30 25.60 26.86 28.10 29.30 30.48
 *     96  1.98  3.92  5.82  7.68  9.50 11.29 13.05 14.77 16.46 18.11 19.74 21.33 22.90 24.44 25.95 27.43 28.88 30.32 31.72 33.10
 *    128  1.98  3.94  5.86  7.76  9.62 11.46 13.27 15.06 16.82 18.55 20.26 21.94 23.60 25.24 26.85 28.44 30.01 31.56 33.09 34.59
 *    160  1.99  3.95  5.89  7.80  9.70 11.57 13.41 15.24 17.04 18.82 20.58 22.33 24.05 25.75 27.43 29.09 30.73 32.36 33.97 35.56
 *    192  1.99  3.96  5.91  7.84  9.75 11.64 13.51 15.36 17.19 19.01 20.81 22.59 24.35 26.10 27.83 29.54 31.23 32.91 34.58 36.23
 *    224  1.99  3.96  5.92  7.86  9.78 11.69 13.58 15.45 17.30 19.15 20.97 22.78 24.57 26.35 28.12 29.87 31.60 33.32 35.03 36.72
 *    256  1.99  3.97  5.93  7.88  9.81 11.73 13.63 15.52 17.39 19.25 21.09 22.93 24.74 26.55 28.34 30.12 31.88 33.64 35.37 37.10
 *    288  1.99  3.97  5.94  7.89  9.83 11.76 13.67 15.57 17.45 19.33 21.19 23.04 24.88 26.70 28.51 30.32 32.10 33.88 35.65 37.40
 *    320  1.99  3.98  5.94  7.90  9.85 11.78 13.70 15.61 17.51 19.39 21.27 23.13 24.98 26.83 28.66 30.48 32.28 34.08 35.87 37.65
 *    352  1.99  3.98  5.95  7.91  9.86 11.80 13.73 15.64 17.55 19.45 21.33 23.21 25.07 26.93 28.77 30.61 32.43 34.25 36.05 37.85
 *    384  1.99  3.98  5.95  7.92  9.87 11.82 13.75 15.67 17.59 19.49 21.39 23.27 25.15 27.02 28.87 30.72 32.56 34.39 36.21 38.02
 *    416  2.00  3.98  5.96  7.92  9.88 11.83 13.77 15.70 17.62 19.53 21.43 23.33 25.21 27.09 28.96 30.81 32.67 34.51 36.34 38.17
 *    448  2.00  3.98  5.96  7.93  9.89 11.84 13.78 15.72 17.65 19.56 21.47 23.37 25.27 27.15 29.03 30.90 32.76 34.61 36.45 38.29
 *    480  2.00  3.98  5.96  7.93  9.90 11.85 13.80 15.74 17.67 19.59 21.51 23.41 25.31 27.21 29.09 30.97 32.84 34.70 36.55 38.40
 *    512  2.00  3.98  5.97  7.94  9.90 11.86 13.81 15.75 17.69 19.62 21.54 23.45 25.36 27.25 29.15 31.03 32.91 34.78 36.64 38.50
 *
 * The number of registers per block is mb*32 (compiled with -maxrregcount=32).
 * More threads == better performance (from flop-test) therefore mb is chosen to
 * be the largest number of threads such that the number of blocks per
 * multiprocessor is still limited by the register usage.
 * kb is chosen to be the largest multiple of 16 such that the number of blocks
 * per multiprocessor is limited by the register usage.
 */
template void cgemm<CBlasNoTrans,   CBlasNoTrans,   64,  8, 16, 16,  4>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasNoTrans,   CBlasTrans,     64,  8, 16,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasNoTrans,   CBlasConjTrans, 64,  8, 16,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasTrans,     CBlasNoTrans,   32, 16,  8,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasTrans,     CBlasTrans,     32, 16,  8,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasTrans,     CBlasConjTrans, 32, 16,  8,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasConjTrans, CBlasNoTrans,   32, 16,  8,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasConjTrans, CBlasTrans,     32, 16,  8,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
template void cgemm<CBlasConjTrans, CBlasConjTrans, 32, 16,  8,  8,  8>(int, int, int, hipComplex, const hipComplex *, int, const hipComplex *, int, hipComplex, hipComplex *, int);
