#include "hip/hip_runtime.h"
#include "blas.h"
#include <hip/hip_complex.h>

__host__ __device__ static __inline__ hipComplex hipCmulf(float a, hipComplex b) {
  return make_hipComplex(a * hipCrealf(b), a * hipCimagf(b));
}

__host__ __device__ static __inline__ hipComplex hipCfmaf(float a, hipComplex b, hipComplex c) {
  return make_hipComplex(a * hipCrealf(b) + hipCrealf(c), a * hipCimagf(b) + hipCimagf(c));
}

#if __CUDA_ARCH__ < 200 && (!defined(__BANK_CONFLICTS__) || __BANK_CONFLICTS__ <= 1)

// y(1:8) += alpha * x(1:8)
__device__ void caxpy(hipComplex alpha, const float * x_real, const float * x_imag, hipComplex * y) {
  y[0] = hipCfmaf(alpha, make_hipComplex(x_real[0], x_imag[0]), y[0]);
  y[1] = hipCfmaf(alpha, make_hipComplex(x_real[1], x_imag[1]), y[1]);
  y[2] = hipCfmaf(alpha, make_hipComplex(x_real[2], x_imag[2]), y[2]);
  y[3] = hipCfmaf(alpha, make_hipComplex(x_real[3], x_imag[3]), y[3]);
  y[4] = hipCfmaf(alpha, make_hipComplex(x_real[4], x_imag[4]), y[4]);
  y[5] = hipCfmaf(alpha, make_hipComplex(x_real[5], x_imag[5]), y[5]);
  y[6] = hipCfmaf(alpha, make_hipComplex(x_real[6], x_imag[6]), y[6]);
  y[7] = hipCfmaf(alpha, make_hipComplex(x_real[7], x_imag[7]), y[7]);
}

/**
 * CHERK:
 *   C := alpha * A'A + beta * C for trans == CBlasNoTrans; or
 *   C := alpha * AA' + beta * C for trans != CBlasNoTrans.
 *
 * Only the upper or lower triangle of C is updated.
 *
 * @param uplo   uplo for C.
 * @param trans  transpose for A.
 * @param mb     the number of rows in the block of C.
 * @param nb     the number of columns in the block of C.
 * @param kb     how far to unroll the inner loop.
 * @param bx     blockDim.x.
 * @param by     blockDim.y.
 */
template <CBlasUplo uplo, CBlasTranspose trans,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void cherk(const hipComplex * __restrict__ A,
                      hipComplex * __restrict__ C,
                      float alpha, float beta,
                      int lda, int ldc,
                      int n, int k) {

//   int bi, bj, nnb = (n + nb - 1) / nb;
//   if (uplo == CBlasLower) {
//     bi = blockIdx.x % nnb;
//     bj = blockIdx.x / nnb;
//     if (bi < bj) {
//       bi = nnb - bi - 1;
//       bj = nnb - bj;
//     }
//   }
//   else {
//     bi = blockIdx.x / nnb;
//     bj = blockIdx.x % nnb;
//     if (bj < bi) {
//       bi = nnb - bi;
//       bj = nnb - bj - 1;
//     }
//   }
//
//   bi *= mb;
//   bj *= nb;

  const int bi = blockIdx.x * mb;       // Starting row of block of C
  const int bj = blockIdx.y * nb;       // Starting column of block of C

  /*
   * Cause blocks that are entirely above or below the diagonal to exit now.
   */
  if (uplo == CBlasUpper) {
    if (bj + nb - 1 < bi)
      return;
  }
  else if (uplo == CBlasLower) {
    if (bi + mb - 1 < bj)
      return;
  }

  // Using a CGEMM kernel, CHERK is:
  // C = alpha * A * B + beta * C
  // with A = A and B = A' when trans == CBlasNoTrans, and
  // with A = A' and B = A when trans == CBlasTrans
  const hipComplex * __restrict__ B = A;

  int ti = threadIdx.y * bx + threadIdx.x;        // Unwrapped thread index [0, bx * by]
  int tj = 0;
  if (trans != CBlasNoTrans) {
    tj = 8 * (ti / mb);
    ti = ti % mb;
  }

  /*
   * Compute our starting points in A, "B" and C.
   *
   * For trans != CBlasNoTrans A is cached in shared memory so the unwrapped
   * thread index can be re-wrapped around mb when calculating C.
   *
   * If trans == CBlasNoTrans then bx * by == mb (checked later on) so there
   * doesn't need to be a separate check for trans == CBlasNoTrans in
   * calculating the start of C here.
   */
  if (trans == CBlasNoTrans) {
    A += bi + ti;
    B += threadIdx.y * lda + bj + threadIdx.x;
  }
  else {
    A += (bi + threadIdx.y) * lda + threadIdx.x;
    B += (bj + threadIdx.y) * lda + threadIdx.x;
  }
  C += (bj + tj) * ldc + bi + ti;
  int m = n - bi - ti;
  n -= bj + tj;

  /*
   * Blocks of A and "B" in shared memory and C in registers.
   */
  __shared__ float a_real[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ float a_imag[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ float b_real[kb][(trans == CBlasNoTrans) ? nb : nb + 1];
  __shared__ float b_imag[kb][(trans == CBlasNoTrans) ? nb : nb + 1];

  hipComplex c[] = { { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f },
                    { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f } };

  while (k > 0) {
    if (trans == CBlasNoTrans) {
      // C = aAA' + bC so read B into shared memory and transpose leaving A
      // untransposed in global memory
#pragma unroll
      for (int l = 0; l < kb; l += by) {
        b_real[l + threadIdx.y][threadIdx.x] =  hipCrealf(B[l * lda]);
        b_imag[l + threadIdx.y][threadIdx.x] = -hipCimagf(B[l * lda]);
      }
    }
    else {
      // C = aA'A + bC so read A into shared memory and transpose before reading
      // B into shared memory untransposed
#pragma unroll
      for (int i = 0; i < mb; i += by) {
        a_real[i + threadIdx.y][threadIdx.x] =  hipCrealf(A[i * lda]);
        a_imag[i + threadIdx.y][threadIdx.x] = -hipCimagf(A[i * lda]);
      }
      A += kb;

#pragma unroll
      for (int j = 0; j < nb; j += by) {
        b_real[threadIdx.x][j + threadIdx.y] = hipCrealf(B[j * lda]);
        b_imag[threadIdx.x][j + threadIdx.y] = hipCimagf(B[j * lda]);
      }
    }

    __syncthreads();

    if (k < kb) break;

    if (trans == CBlasNoTrans) {
      // Read A from global memory
#pragma unroll
      for (int l = 0; l < kb; l++) {
        caxpy(A[0], b_real[l], b_imag[l], c);
        A += lda;
      }
    }
    else {
      // Read A' from shared memory
#pragma unroll
      for (int l = 0; l < kb; l++)
        caxpy(make_hipComplex(a_real[ti][l], a_imag[ti][l]),
              &b_real[l][tj], &b_imag[l][tj], c);
    }

    __syncthreads();

    B += (trans != CBlasNoTrans) ? kb : kb * lda;
    k -= kb;
  }

  if (trans == CBlasNoTrans) {
    // Read A from global memory
    for (int l = 0; l < k; l++) {
      caxpy(A[0], b_real[l], b_imag[l], c);
      A += lda;
    }
  }
  else {
    // Read A' from shared memory
    for (int l = 0; l < k; l++)
        caxpy(make_hipComplex(a_real[ti][l], a_imag[ti][l]),
              &b_real[l][tj], &b_imag[l][tj], c);
  }

  if (m <= 0 || n <= 0) return;
  int i = bi + ti;
  int j = bj + tj;
  if (beta == 0.0f) {
    if (uplo == CBlasUpper) {
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[0]), 0.0f) : hipCmulf(alpha, c[0]); if (1 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[1]), 0.0f) : hipCmulf(alpha, c[1]); if (2 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[2]), 0.0f) : hipCmulf(alpha, c[2]); if (3 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[3]), 0.0f) : hipCmulf(alpha, c[3]); if (4 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[4]), 0.0f) : hipCmulf(alpha, c[4]); if (5 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[5]), 0.0f) : hipCmulf(alpha, c[5]); if (6 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[6]), 0.0f) : hipCmulf(alpha, c[6]); if (7 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[7]), 0.0f) : hipCmulf(alpha, c[7]);
    }
    else {
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[0]), 0.0f) : hipCmulf(alpha, c[0]); if (1 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[1]), 0.0f) : hipCmulf(alpha, c[1]); if (2 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[2]), 0.0f) : hipCmulf(alpha, c[2]); if (3 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[3]), 0.0f) : hipCmulf(alpha, c[3]); if (4 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[4]), 0.0f) : hipCmulf(alpha, c[4]); if (5 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[5]), 0.0f) : hipCmulf(alpha, c[5]); if (6 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[6]), 0.0f) : hipCmulf(alpha, c[6]); if (7 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[7]), 0.0f) : hipCmulf(alpha, c[7]);
    }
  }
  else {
    if (uplo == CBlasUpper) {
      if (i <= j) C[0] = hipCfmaf(alpha, c[0], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (1 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[1], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (2 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[2], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (3 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[3], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (4 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[4], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (5 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[5], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (6 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[6], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (7 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[7], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0])));
    }
    else {
      if (i >= j) C[0] = hipCfmaf(alpha, c[0], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (1 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[1], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (2 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[2], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (3 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[3], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (4 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[4], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (5 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[5], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (6 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[6], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (7 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[7], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0])));
    }
  }
}

#else

// y(1:8) += alpha * x(1:8)
__device__ void caxpy(hipComplex alpha, const hipComplex * x, hipComplex * y) {
  y[0] = hipCfmaf(alpha, x[0], y[0]); y[1] = hipCfmaf(alpha, x[1], y[1]);
  y[2] = hipCfmaf(alpha, x[2], y[2]); y[3] = hipCfmaf(alpha, x[3], y[3]);
  y[4] = hipCfmaf(alpha, x[4], y[4]); y[5] = hipCfmaf(alpha, x[5], y[5]);
  y[6] = hipCfmaf(alpha, x[6], y[6]); y[7] = hipCfmaf(alpha, x[7], y[7]);
}

/**
 * CHERK:
 *   C := alpha * A'A + beta * C for trans == CBlasNoTrans; or
 *   C := alpha * AA' + beta * C for trans != CBlasNoTrans.
 *
 * Only the upper or lower triangle of C is updated.
 *
 * @param uplo   uplo for C.
 * @param trans  transpose for A.
 * @param mb     the number of rows in the block of C.
 * @param nb     the number of columns in the block of C.
 * @param kb     how far to unroll the inner loop.
 * @param bx     blockDim.x.
 * @param by     blockDim.y.
 */
template <CBlasUplo uplo, CBlasTranspose trans,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void cherk(const hipComplex * __restrict__ A,
                      hipComplex * __restrict__ C,
                      float alpha, float beta,
                      int lda, int ldc,
                      int n, int k) {

//   int bi, bj, nnb = (n + nb - 1) / nb;
//   if (uplo == CBlasLower) {
//     bi = blockIdx.x % nnb;
//     bj = blockIdx.x / nnb;
//     if (bi < bj) {
//       bi = nnb - bi - 1;
//       bj = nnb - bj;
//     }
//   }
//   else {
//     bi = blockIdx.x / nnb;
//     bj = blockIdx.x % nnb;
//     if (bj < bi) {
//       bi = nnb - bi;
//       bj = nnb - bj - 1;
//     }
//   }
//
//   bi *= mb;
//   bj *= nb;

  const int bi = blockIdx.x * mb;       // Starting row of block of C
  const int bj = blockIdx.y * nb;       // Starting column of block of C

  /*
   * Cause blocks that are entirely above or below the diagonal to exit now.
   */
  if (uplo == CBlasUpper) {
    if (bj + nb - 1 < bi)
      return;
  }
  else if (uplo == CBlasLower) {
    if (bi + mb - 1 < bj)
      return;
  }

  // Using a CGEMM kernel, CHERK is:
  // C = alpha * A * B + beta * C
  // with A = A and B = A' when trans == CBlasNoTrans, and
  // with A = A' and B = A when trans == CBlasTrans
  const hipComplex * __restrict__ B = A;

  int ti = threadIdx.y * bx + threadIdx.x;        // Unwrapped thread index [0, bx * by]
  int tj = 0;
  if (trans != CBlasNoTrans) {
    tj = 8 * (ti / mb);
    ti = ti % mb;
  }

  /*
   * Compute our starting points in A, "B" and C.
   *
   * For trans != CBlasNoTrans A is cached in shared memory so the unwrapped
   * thread index can be re-wrapped around mb when calculating C.
   *
   * If trans == CBlasNoTrans then bx * by == mb (checked later on) so there
   * doesn't need to be a separate check for trans == CBlasNoTrans in
   * calculating the start of C here.
   */
  if (trans == CBlasNoTrans) {
    A += bi + ti;
    B += threadIdx.y * lda + bj + threadIdx.x;
  }
  else {
    A += (bi + threadIdx.y) * lda + threadIdx.x;
    B += (bj + threadIdx.y) * lda + threadIdx.x;
  }
  C += (bj + tj) * ldc + bi + ti;
  int m = n - bi - ti;
  n -= bj + tj;

  /*
   * Blocks of A and "B" in shared memory and C in registers.
   */
  __shared__ hipComplex a[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ hipComplex b[kb][(trans == CBlasNoTrans) ? nb : nb + 1];

  hipComplex c[] = { { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f },
                    { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f }, { 0.0f, 0.0f } };

  while (k > 0) {
    if (trans == CBlasNoTrans) {
      // C = aAA' + bC so read B into shared memory and transpose leaving A
      // untransposed in global memory
#pragma unroll
      for (int l = 0; l < kb; l += by)
        b[l + threadIdx.y][threadIdx.x] = hipConjf(B[l * lda]);
    }
    else {
     // C = aA'A + bC so read A into shared memory and transpose before reading
     // B into shared memory untransposed
#pragma unroll
      for (int i = 0; i < mb; i += by)
        a[i + threadIdx.y][threadIdx.x] = hipConjf(A[i * lda]);
      A += kb;

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * lda];
    }

    __syncthreads();

    if (k < kb) break;

    if (trans == CBlasNoTrans) {
      // Read A from global memory
#pragma unroll
      for (int l = 0; l < kb; l++) {
        caxpy(A[0], b[l], c);
        A += lda;
      }
    }
    else {
      // Read A' from shared memory
#pragma unroll
      for (int l = 0; l < kb; l++)
        caxpy(a[ti][l], &b[l][tj], c);
    }

    __syncthreads();

    B += (trans != CBlasNoTrans) ? kb : kb * lda;
    k -= kb;
  }

  if (trans == CBlasNoTrans) {
    // Read A from global memory
    for (int l = 0; l < k; l++) {
      caxpy(A[0], b[l], c);
      A += lda;
    }
  }
  else {
    // Read A' from shared memory
    for (int l = 0; l < k; l++)
        caxpy(a[ti][l], &b[l][tj], c);
  }

  if (m <= 0 || n <= 0) return;
  int i = bi + ti;
  int j = bj + tj;
  if (beta == 0.0f) {
    if (uplo == CBlasUpper) {
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[0]), 0.0f) : hipCmulf(alpha, c[0]); if (1 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[1]), 0.0f) : hipCmulf(alpha, c[1]); if (2 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[2]), 0.0f) : hipCmulf(alpha, c[2]); if (3 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[3]), 0.0f) : hipCmulf(alpha, c[3]); if (4 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[4]), 0.0f) : hipCmulf(alpha, c[4]); if (5 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[5]), 0.0f) : hipCmulf(alpha, c[5]); if (6 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[6]), 0.0f) : hipCmulf(alpha, c[6]); if (7 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[7]), 0.0f) : hipCmulf(alpha, c[7]);
    }
    else {
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[0]), 0.0f) : hipCmulf(alpha, c[0]); if (1 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[1]), 0.0f) : hipCmulf(alpha, c[1]); if (2 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[2]), 0.0f) : hipCmulf(alpha, c[2]); if (3 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[3]), 0.0f) : hipCmulf(alpha, c[3]); if (4 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[4]), 0.0f) : hipCmulf(alpha, c[4]); if (5 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[5]), 0.0f) : hipCmulf(alpha, c[5]); if (6 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[6]), 0.0f) : hipCmulf(alpha, c[6]); if (7 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = (i == j) ? make_hipComplex(alpha * hipCrealf(c[7]), 0.0f) : hipCmulf(alpha, c[7]);
    }
  }
  else {
    if (uplo == CBlasUpper) {
      if (i <= j) C[0] = hipCfmaf(alpha, c[0], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (1 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[1], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (2 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[2], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (3 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[3], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (4 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[4], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (5 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[5], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (6 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[6], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (7 >= n) return; j++; C += ldc;
      if (i <= j) C[0] = hipCfmaf(alpha, c[7], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0])));
    }
    else {
      if (i >= j) C[0] = hipCfmaf(alpha, c[0], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (1 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[1], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (2 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[2], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (3 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[3], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (4 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[4], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (5 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[5], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (6 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[6], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0]))); if (7 >= n) return; j++; C += ldc;
      if (i >= j) C[0] = hipCfmaf(alpha, c[7], hipCmulf(beta, ((i == j) ? make_hipComplex(hipCrealf(C[0]), 0.0f) : C[0])));
    }
  }
}

#endif

/**
 * For C = aAB + bC:
 *   mb must be a multiple of the warp size (32) and less than or equal to the
 *        maximum number of threads per block (512).
 *   nb must be less than or equal to 20 (registers start spilling to global
 *        memory after 20).
 *   kb must be a multiple of the half-warp size (16) and such that
 *        (nb + 1)*kb*sizeof(hipComplex) is less than the amount of shared memory
 *        available per block (16384 bytes).
 *
 * mb and nb must be selected such that the bandwidth reduction is greater than
 * the flop:word ratio of the GPU.  The bandwidth reduction for all valid values
 * of mb and nb can be calculated with the following loop (bash):
 * echo -n " mb\nb"; for nb in {1..20}; do printf "%6d" ${nb}; done; echo; for mb in {32..512..32}; do printf "%6d"  ${mb}; for nb in {1..20}; do printf "%6.2f" $(echo 2 / \(1/${mb} + 1/${nb}\) | bc -l); done; echo; done
 *
 * Sample output:
 *  mb\nb     1     2     3     4     5     6     7     8     9    10    11    12    13    14    15    16    17    18    19    20
 *     32  1.94  3.76  5.49  7.11  8.65 10.11 11.49 12.80 14.05 15.24 16.37 17.45 18.49 19.48 20.43 21.33 22.20 23.04 23.84 24.62
 *     64  1.97  3.88  5.73  7.53  9.28 10.97 12.62 14.22 15.78 17.30 18.77 20.21 21.61 22.97 24.30 25.60 26.86 28.10 29.30 30.48
 *     96  1.98  3.92  5.82  7.68  9.50 11.29 13.05 14.77 16.46 18.11 19.74 21.33 22.90 24.44 25.95 27.43 28.88 30.32 31.72 33.10
 *    128  1.98  3.94  5.86  7.76  9.62 11.46 13.27 15.06 16.82 18.55 20.26 21.94 23.60 25.24 26.85 28.44 30.01 31.56 33.09 34.59
 *    160  1.99  3.95  5.89  7.80  9.70 11.57 13.41 15.24 17.04 18.82 20.58 22.33 24.05 25.75 27.43 29.09 30.73 32.36 33.97 35.56
 *    192  1.99  3.96  5.91  7.84  9.75 11.64 13.51 15.36 17.19 19.01 20.81 22.59 24.35 26.10 27.83 29.54 31.23 32.91 34.58 36.23
 *    224  1.99  3.96  5.92  7.86  9.78 11.69 13.58 15.45 17.30 19.15 20.97 22.78 24.57 26.35 28.12 29.87 31.60 33.32 35.03 36.72
 *    256  1.99  3.97  5.93  7.88  9.81 11.73 13.63 15.52 17.39 19.25 21.09 22.93 24.74 26.55 28.34 30.12 31.88 33.64 35.37 37.10
 *    288  1.99  3.97  5.94  7.89  9.83 11.76 13.67 15.57 17.45 19.33 21.19 23.04 24.88 26.70 28.51 30.32 32.10 33.88 35.65 37.40
 *    320  1.99  3.98  5.94  7.90  9.85 11.78 13.70 15.61 17.51 19.39 21.27 23.13 24.98 26.83 28.66 30.48 32.28 34.08 35.87 37.65
 *    352  1.99  3.98  5.95  7.91  9.86 11.80 13.73 15.64 17.55 19.45 21.33 23.21 25.07 26.93 28.77 30.61 32.43 34.25 36.05 37.85
 *    384  1.99  3.98  5.95  7.92  9.87 11.82 13.75 15.67 17.59 19.49 21.39 23.27 25.15 27.02 28.87 30.72 32.56 34.39 36.21 38.02
 *    416  2.00  3.98  5.96  7.92  9.88 11.83 13.77 15.70 17.62 19.53 21.43 23.33 25.21 27.09 28.96 30.81 32.67 34.51 36.34 38.17
 *    448  2.00  3.98  5.96  7.93  9.89 11.84 13.78 15.72 17.65 19.56 21.47 23.37 25.27 27.15 29.03 30.90 32.76 34.61 36.45 38.29
 *    480  2.00  3.98  5.96  7.93  9.90 11.85 13.80 15.74 17.67 19.59 21.51 23.41 25.31 27.21 29.09 30.97 32.84 34.70 36.55 38.40
 *    512  2.00  3.98  5.97  7.94  9.90 11.86 13.81 15.75 17.69 19.62 21.54 23.45 25.36 27.25 29.15 31.03 32.91 34.78 36.64 38.50
 *
 * The number of registers per block is mb*32 (compiled with -maxrregcount=32).
 * More threads == better performance (from flop-test) therefore mb is chosen to
 * be the largest number of threads such that the number of blocks per
 * multiprocessor is still limited by the register usage.
 * kb is chosen to be the largest multiple of 16 such that the number of blocks
 * per multiprocessor is limited by the register usage.
 */
template void cherk<CBlasUpper, CBlasNoTrans,   64,  8,  8,  8,  8>(const hipComplex *, hipComplex *, float, float, int, int, int, int);
template void cherk<CBlasLower, CBlasNoTrans,   64,  8,  8,  8,  8>(const hipComplex *, hipComplex *, float, float, int, int, int, int);
template void cherk<CBlasUpper, CBlasConjTrans, 32, 16,  8,  8,  8>(const hipComplex *, hipComplex *, float, float, int, int, int, int);
template void cherk<CBlasLower, CBlasConjTrans, 32, 16,  8,  8,  8>(const hipComplex *, hipComplex *, float, float, int, int, int, int);
