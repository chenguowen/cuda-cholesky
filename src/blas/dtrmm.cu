#include "hip/hip_runtime.h"
#include "blas.h"

#if __CUDA_ARCH__ < 200 && !defined(__BANK_CONFLICT__)

// y(1:8) += alpha * x(1:8)
__device__ void daxpy(double alpha, const int * x_hi, const int * x_lo, double * y) {
  y[0] += alpha * __hiloint2double(x_hi[0], x_lo[0]);
  y[1] += alpha * __hiloint2double(x_hi[1], x_lo[1]);
  y[2] += alpha * __hiloint2double(x_hi[2], x_lo[2]);
  y[3] += alpha * __hiloint2double(x_hi[3], x_lo[3]);
  y[4] += alpha * __hiloint2double(x_hi[4], x_lo[4]);
  y[5] += alpha * __hiloint2double(x_hi[5], x_lo[5]);
  y[6] += alpha * __hiloint2double(x_hi[6], x_lo[6]);
  y[7] += alpha * __hiloint2double(x_hi[7], x_lo[7]);
}

// y(1:n) += alpha * x(1:n)
__device__ void daxpy(int n, double alpha, const int * x_hi, const int * x_lo, double * y) {
  y[0] += alpha * __hiloint2double(x_hi[0], x_lo[0]); if (1 >= n) return;
  y[1] += alpha * __hiloint2double(x_hi[1], x_lo[1]); if (2 >= n) return;
  y[2] += alpha * __hiloint2double(x_hi[2], x_lo[2]); if (3 >= n) return;
  y[3] += alpha * __hiloint2double(x_hi[3], x_lo[3]); if (4 >= n) return;
  y[4] += alpha * __hiloint2double(x_hi[4], x_lo[4]); if (5 >= n) return;
  y[5] += alpha * __hiloint2double(x_hi[5], x_lo[5]); if (6 >= n) return;
  y[6] += alpha * __hiloint2double(x_hi[6], x_lo[6]); if (7 >= n) return;
  y[7] += alpha * __hiloint2double(x_hi[7], x_lo[7]);
}

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void dtrmm2L(int m, int n,
                        double alpha, const double * __restrict__ A, int lda, const double * __restrict__ B, int ldb,
                        double * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;
  int tj = 0;
  if (trans != CBlasNoTrans) {
    tj = 8 * (ti / mb);
    ti = ti % mb;
  }

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? bi * lda + bi + ti : bi + ti;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + bi + threadIdx.x
                              : (bj + threadIdx.y) * ldb + threadIdx.x;
  }
  else {
    A += (uplo == CBlasUpper) ? (bi + threadIdx.y) * lda + threadIdx.x
                              : (bi + threadIdx.y) * lda + bi + threadIdx.x;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + threadIdx.x
                              : (bj + threadIdx.y) * ldb + bi + threadIdx.x;
  }
  X += (bj + tj) * ldx + bi + ti;

  __shared__ int a_hi[mb][kb + 1];
  __shared__ int a_lo[mb][kb + 1];
  __shared__ int b_hi[kb][nb];
  __shared__ int b_lo[kb][nb];

  double x[] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };

  // For Upper/NoTrans and Lower/Trans process diagonal first
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {
    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by) {
          a_hi[i + threadIdx.y][threadIdx.x] = __double2hiint(A[i * lda]);
          a_lo[i + threadIdx.y][threadIdx.x] = __double2loint(A[i * lda]);
        }
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by) {
        b_hi[threadIdx.x][j + threadIdx.y] = __double2hiint(B[j * ldb]);
        b_lo[threadIdx.x][j + threadIdx.y] = __double2loint(B[j * ldb]);
      }

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti <= l++)
            daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                  (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                  (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            daxpy(1.0, (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                       (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
          else if (ti < l)
            daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                  (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                  (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti <= l++)
          daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          daxpy(1.0, (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                     (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
        else if (ti < l)
          daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for DGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? m - bi - mb : bi)
                                  : ((uplo == CBlasUpper) ? bi : m - bi - mb);
  while (k > 0) {
    if (trans != CBlasNoTrans) {
#pragma unroll
      for (int i = 0; i < mb; i += by) {
        a_hi[i + threadIdx.y][threadIdx.x] = __double2hiint(A[i * lda]);
        a_lo[i + threadIdx.y][threadIdx.x] = __double2loint(A[i * lda]);
      }
      A += kb;
    }

#pragma unroll
    for (int j = 0; j < nb; j += by) {
      b_hi[threadIdx.x][j + threadIdx.y] = __double2hiint(B[j * ldb]);
      b_lo[threadIdx.x][j + threadIdx.y] = __double2loint(B[j * ldb]);
    }

    __syncthreads();

    if (k < kb) break;

    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int l = 0; l < kb; l++) {
        daxpy(A[0], b_hi[l], b_lo[l], x);
        A += lda;
      }
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l++)
        daxpy(__hiloint2double(a_hi[ti][l], a_lo[ti][l]), &b_hi[l][tj], &b_lo[l][tj], x);
    }

    __syncthreads();

    B += kb;
    k -= kb;
  }

  if (trans == CBlasNoTrans) {
    for (int l = 0; l < k; l++) {
      daxpy(A[0], b_hi[l], b_lo[l], x);
      A += lda;
    }
  }
  else {
    for (int l = 0; l < k; l++)
      daxpy(__hiloint2double(a_hi[ti][l], a_lo[ti][l]), &b_hi[l][tj], &b_lo[l][tj], x);
  }

  // For Upper/Trans and Lower/NoTrans process diagonal last
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {

    __syncthreads();

    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
    if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by) {
          a_hi[i + threadIdx.y][threadIdx.x] = __double2hiint(A[i * lda]);
          a_lo[i + threadIdx.y][threadIdx.x] = __double2loint(A[i * lda]);
        }
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by) {
        b_hi[threadIdx.x][j + threadIdx.y] = __double2hiint(B[j * ldb]);
        b_lo[threadIdx.x][j + threadIdx.y] = __double2loint(B[j * ldb]);
      }

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti >= l++)
            daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                  (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                  (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            daxpy(1.0, (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                       (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
          else if (ti > l)
            daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                  (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                  (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti >= l++)
          daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          daxpy(1.0, (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                      (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
        else if (ti > l)
          daxpy((trans == CBlasNoTrans) ? A[0] : __hiloint2double(a_hi[ti][ll], a_lo[ti][ll]),
                (trans == CBlasNoTrans) ? b_hi[ll] : &b_hi[ll][tj],
                (trans == CBlasNoTrans) ? b_lo[ll] : &b_lo[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }
  }

  n -= bj + tj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7];
}

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void dtrmm2R(int m, int n,
                        double alpha, const double * __restrict__ A, int lda, const double * __restrict__ B, int ldb,
                        double * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + threadIdx.x
                              : (bj + threadIdx.y) * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bi + ti : bj * ldb + bi + ti;
  }
  else {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + bj + threadIdx.x
                              : threadIdx.y * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bj * ldb + bi + ti : bi + ti;
  }
  X += bj * ldx + bi + ti;

  __shared__ int a_hi[kb][nb];
  __shared__ int a_lo[kb][nb];

  double x[] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };

  // For Upper/Trans and Lower/NoTrans process diagonal first
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {
    int k = min(n - bj, nb);
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by) {
          a_hi[threadIdx.x][j + threadIdx.y] =
            (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? __double2hiint(1.0)
                                                                     : __double2hiint(A[j * lda]);
          a_lo[threadIdx.x][j + threadIdx.y] =
            (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? __double2loint(1.0)
                                                                     : __double2loint(A[j * lda]);
        }
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by) {
          a_hi[l + threadIdx.y][threadIdx.x] =
            (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? __double2hiint(1.0)
                                                                     : __double2hiint(A[l * lda]);
          a_lo[l + threadIdx.y][threadIdx.x] =
            (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? __double2loint(1.0)
                                                                     : __double2loint(A[l * lda]);
        }
      }

      __syncthreads();

      if (k < kb) break;

// #pragma unroll
//       for (int ll = 0; ll < kb; ll++) {
//         daxpy(ll + 1, B[0], a_hi[ll], a_lo[ll], x);
        daxpy( 1, B[0], a_hi[ 0], a_lo[ 0], x); B += ldb;
        daxpy( 2, B[0], a_hi[ 1], a_lo[ 1], x); B += ldb;
        daxpy( 3, B[0], a_hi[ 2], a_lo[ 2], x); B += ldb;
        daxpy( 4, B[0], a_hi[ 3], a_lo[ 3], x); B += ldb;
        daxpy( 5, B[0], a_hi[ 4], a_lo[ 4], x); B += ldb;
        daxpy( 6, B[0], a_hi[ 5], a_lo[ 5], x); B += ldb;
        daxpy( 7, B[0], a_hi[ 6], a_lo[ 6], x); B += ldb;
        daxpy( 8, B[0], a_hi[ 7], a_lo[ 7], x); B += ldb;
//         B += ldb;
//       }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

    for (int ll = 0; ll < k; ll++) {
      daxpy(ll + 1, B[0], a_hi[ll], a_lo[ll], x);
      B += ldb;
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for DGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? bj : n - bj - nb)
                                  : ((uplo == CBlasUpper) ? n - bj - nb : bj);
  while (k > 0) {
    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int j = 0; j < nb; j += by) {
        a_hi[threadIdx.x][j + threadIdx.y] = __double2hiint(A[j * lda]);
        a_lo[threadIdx.x][j + threadIdx.y] = __double2loint(A[j * lda]);
      }
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l += by) {
        a_hi[l + threadIdx.y][threadIdx.x] = __double2hiint(A[l * lda]);
        a_lo[l + threadIdx.y][threadIdx.x] = __double2loint(A[l * lda]);
      }
    }

      __syncthreads();

    if (k < kb) break;

#pragma unroll
    for (int l = 0; l < kb; l++) {
      daxpy(B[0], a_hi[l], a_lo[l], x);
      B += ldb;
    }

    __syncthreads();

    A += (trans == CBlasNoTrans) ? kb : kb * lda;
    k -= kb;
  }

  for (int l = 0; l < k; l++) {
    daxpy(B[0], a_hi[l], a_lo[l], x);
    B += ldb;
  }

  // For Upper/NoTrans and Lower/Trans process diagonal last
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {

    __syncthreads();

    int k = min(n - bj, nb);
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by) {
          a_hi[threadIdx.x][j + threadIdx.y] =
            (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? __double2hiint(1.0)
                                                                     : __double2hiint(A[j * lda]);
          a_lo[threadIdx.x][j + threadIdx.y] =
            (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? __double2loint(1.0)
                                                                     : __double2loint(A[j * lda]);
        }
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by) {
          a_hi[l + threadIdx.y][threadIdx.x] =
            (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? __double2hiint(1.0)
                                                                     : __double2hiint(A[l * lda]);
          a_lo[l + threadIdx.y][threadIdx.x] =
            (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? __double2loint(1.0)
                                                                     : __double2loint(A[l * lda]);
        }
      }

      __syncthreads();

      if (k < kb) break;

// #pragma unroll
//       for (int ll = 0; ll < kb; ll++) {
//         daxpy(nb - ll, B[0], &a_hi[ll][ll], &a_lo[ll][ll], &x[ll]);
        daxpy(8, B[0], &a_hi[ 0][ 0], &a_lo[ 0][ 0], &x[ 0]); B += ldb;
        daxpy(7, B[0], &a_hi[ 1][ 1], &a_lo[ 1][ 1], &x[ 1]); B += ldb;
        daxpy(6, B[0], &a_hi[ 2][ 2], &a_lo[ 2][ 2], &x[ 2]); B += ldb;
        daxpy(5, B[0], &a_hi[ 3][ 3], &a_lo[ 3][ 3], &x[ 3]); B += ldb;
        daxpy(4, B[0], &a_hi[ 4][ 4], &a_lo[ 4][ 4], &x[ 4]); B += ldb;
        daxpy(3, B[0], &a_hi[ 5][ 5], &a_lo[ 5][ 5], &x[ 5]); B += ldb;
        daxpy(2, B[0], &a_hi[ 6][ 6], &a_lo[ 6][ 6], &x[ 6]); B += ldb;
        daxpy(1, B[0], &a_hi[ 7][ 7], &a_lo[ 7][ 7], &x[ 7]); B += ldb;
//         B += ldb;
//       }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

//     for (int ll = 0; ll < k; ll++) {
//       saxpy(nb - ll, B[0], &a[ll][ll], &x[ll]);
//       B += ldb;
//     }
    if (k > 0) { daxpy(8, B[0], &a_hi[ 0][ 0], &a_lo[ 0][ 0], &x[ 0]); B += ldb;
    if (k > 1) { daxpy(7, B[0], &a_hi[ 1][ 1], &a_lo[ 1][ 1], &x[ 1]); B += ldb;
    if (k > 2) { daxpy(6, B[0], &a_hi[ 2][ 2], &a_lo[ 2][ 2], &x[ 2]); B += ldb;
    if (k > 3) { daxpy(5, B[0], &a_hi[ 3][ 3], &a_lo[ 3][ 3], &x[ 3]); B += ldb;
    if (k > 4) { daxpy(4, B[0], &a_hi[ 4][ 4], &a_lo[ 4][ 4], &x[ 4]); B += ldb;
    if (k > 5) { daxpy(3, B[0], &a_hi[ 5][ 5], &a_lo[ 5][ 5], &x[ 5]); B += ldb;
    if (k > 6) { daxpy(2, B[0], &a_hi[ 6][ 6], &a_lo[ 6][ 6], &x[ 6]); B += ldb;
    if (k > 7) { daxpy(1, B[0], &a_hi[ 7][ 7], &a_lo[ 7][ 7], &x[ 7]); B += ldb; }}}}}}}}
  }

  n -= bj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7];
}

#else

// y(1:8) += alpha * x(1:8)
__device__ void daxpy(double alpha, const double * x, double * y) {
  y[ 0] += alpha * x[ 0]; y[ 1] += alpha * x[ 1];
  y[ 2] += alpha * x[ 2]; y[ 3] += alpha * x[ 3];
  y[ 4] += alpha * x[ 4]; y[ 5] += alpha * x[ 5];
  y[ 6] += alpha * x[ 6]; y[ 7] += alpha * x[ 7];
}

// y(1:n) += alpha * x(1:n)
__device__ void daxpy(int n, double alpha, const double * x, double * y) {
  y[ 0] += alpha * x[ 0]; if ( 1 >= n) return;
  y[ 1] += alpha * x[ 1]; if ( 2 >= n) return;
  y[ 2] += alpha * x[ 2]; if ( 3 >= n) return;
  y[ 3] += alpha * x[ 3]; if ( 4 >= n) return;
  y[ 4] += alpha * x[ 4]; if ( 5 >= n) return;
  y[ 5] += alpha * x[ 5]; if ( 6 >= n) return;
  y[ 6] += alpha * x[ 6]; if ( 7 >= n) return;
  y[ 7] += alpha * x[ 7];
}

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void dtrmm2L(int m, int n,
                        double alpha, const double * __restrict__ A, int lda, const double * __restrict__ B, int ldb,
                        double * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;
  int tj = 0;
  if (trans != CBlasNoTrans) {
    tj = 8 * (ti / mb);
    ti = ti % mb;
  }

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? bi * lda + bi + ti : bi + ti;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + bi + threadIdx.x
                              : (bj + threadIdx.y) * ldb + threadIdx.x;
  }
  else {
    A += (uplo == CBlasUpper) ? (bi + threadIdx.y) * lda + threadIdx.x
                              : (bi + threadIdx.y) * lda + bi + threadIdx.x;
    B += (uplo == CBlasUpper) ? (bj + threadIdx.y) * ldb + threadIdx.x
                              : (bj + threadIdx.y) * ldb + bi + threadIdx.x;
  }
  X += (bj + tj) * ldx + bi + ti;

  __shared__ double a[mb][kb + 1];
  __shared__ double b[kb][nb];

  double x[] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };

  // For Upper/NoTrans and Lower/Trans process diagonal first
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {
    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][threadIdx.x] = A[i * lda];
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti <= l++)
            daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                  (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            daxpy(1.0, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti < l)
            daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                  (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti <= l++)
          daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          daxpy(1.0, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti < l)
          daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for DGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? m - bi - mb : bi)
                                  : ((uplo == CBlasUpper) ? bi : m - bi - mb);
  while (k > 0) {
    if (trans != CBlasNoTrans) {
#pragma unroll
      for (int i = 0; i < mb; i += by)
        a[i + threadIdx.y][threadIdx.x] = A[i * lda];
      A += kb;
    }

#pragma unroll
    for (int j = 0; j < nb; j += by)
      b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

    __syncthreads();

    if (k < kb) break;

    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int l = 0; l < kb; l++) {
        daxpy(A[0], b[l], x);
        A += lda;
      }
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l++)
        daxpy(a[ti][l], &b[l][tj], x);
    }

    __syncthreads();

    B += kb;
    k -= kb;
  }

  if (trans == CBlasNoTrans) {
    for (int l = 0; l < k; l++) {
      daxpy(A[0], b[l], x);
      A += lda;
    }
  }
  else {
    for (int l = 0; l < k; l++)
      daxpy(a[ti][l], &b[l][tj], x);
  }

  // For Upper/Trans and Lower/NoTrans process diagonal last
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {

    __syncthreads();

    int k = min(m - bi, mb);
    int l = 0;
    while (k > 0) {
      if (trans != CBlasNoTrans) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][threadIdx.x] = A[i * lda];
        A += kb;
      }

#pragma unroll
      for (int j = 0; j < nb; j += by)
        b[threadIdx.x][j + threadIdx.y] = B[j * ldb];

      __syncthreads();

      if (k < kb) break;

      if (diag == CBlasNonUnit) {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti >= l++)
            daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                  (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
        }
      }
      else {
#pragma unroll
        for (int ll = 0; ll < kb; ll++) {
          if (ti == l)
            daxpy(1.0, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          else if (ti > l)
            daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                  (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
          if (trans == CBlasNoTrans)
            A += lda;
          l++;
        }
      }

      __syncthreads();

      B += kb;
      k -= kb;
    }

    if (diag == CBlasNonUnit) {
      for (int ll = 0; ll < k; ll++) {
        if (ti >= l++)
          daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
      }
    }
    else {
      for (int ll = 0; ll < k; ll++) {
        if (ti == l)
          daxpy(1.0, (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        else if (ti > l)
          daxpy((trans == CBlasNoTrans) ? A[0]  :  a[ti][ll],
                (trans == CBlasNoTrans) ? b[ll] : &b[ll][tj], x);
        if (trans == CBlasNoTrans)
          A += lda;
        l++;
      }
    }
  }

  n -= bj + tj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7];
}

template <CBlasUplo uplo, CBlasTranspose trans, CBlasDiag diag,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void dtrmm2R(int m, int n,
                        double alpha, const double * __restrict__ A, int lda, const double * __restrict__ B, int ldb,
                        double * __restrict__ X, int ldx) {

  const int bi = blockIdx.x * mb;       // Starting row of block of X
  const int bj = blockIdx.y * nb;       // Starting column of block of X
  int ti = threadIdx.y * bx + threadIdx.x;

  if (trans == CBlasNoTrans) {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + threadIdx.x
                              : (bj + threadIdx.y) * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bi + ti : bj * ldb + bi + ti;
  }
  else {
    A += (uplo == CBlasUpper) ? (bj + threadIdx.y) * lda + bj + threadIdx.x
                              : threadIdx.y * lda + bj + threadIdx.x;
    B += (uplo == CBlasUpper) ? bj * ldb + bi + ti : bi + ti;
  }
  X += bj * ldx + bi + ti;

  __shared__ double a[kb][nb];

  double x[] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };

  // For Upper/Trans and Lower/NoTrans process diagonal first
  if (uplo == CBlasUpper && trans != CBlasNoTrans ||
      uplo == CBlasLower && trans == CBlasNoTrans) {
    int k = min(n - bj, nb);
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by)
          a[threadIdx.x][j + threadIdx.y] =
            (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? 1.0 : A[j * lda];
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by)
          a[l + threadIdx.y][threadIdx.x] =
            (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? 1.0 : A[l * lda];
      }

      __syncthreads();

      if (k < kb) break;

// #pragma unroll
//       for (int ll = 0; ll < kb; ll++) {
//         daxpy(ll + 1, B[0], a[ll], x);
        daxpy( 1, B[0], a[ 0], x); B += ldb;
        daxpy( 2, B[0], a[ 1], x); B += ldb;
        daxpy( 3, B[0], a[ 2], x); B += ldb;
        daxpy( 4, B[0], a[ 3], x); B += ldb;
        daxpy( 5, B[0], a[ 4], x); B += ldb;
        daxpy( 6, B[0], a[ 5], x); B += ldb;
        daxpy( 7, B[0], a[ 6], x); B += ldb;
        daxpy( 8, B[0], a[ 7], x); B += ldb;
//         B += ldb;
//       }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

    for (int ll = 0; ll < k; ll++) {
      daxpy(ll + 1, B[0], a[ll], x);
      B += ldb;
    }

    __syncthreads();
  }

  // Process non-diagonal blocks as for DGEMM
  int k = (trans == CBlasNoTrans) ? ((uplo == CBlasUpper) ? bj : n - bj - nb)
                                  : ((uplo == CBlasUpper) ? n - bj - nb : bj);
  while (k > 0) {
    if (trans == CBlasNoTrans) {
#pragma unroll
      for (int j = 0; j < nb; j += by)
        a[threadIdx.x][j + threadIdx.y] = A[j * lda];
    }
    else {
#pragma unroll
      for (int l = 0; l < kb; l += by)
        a[l + threadIdx.y][threadIdx.x] = A[l * lda];
    }

    __syncthreads();

    if (k < kb) break;

#pragma unroll
    for (int l = 0; l < kb; l++) {
      daxpy(B[0], a[l], x);
      B += ldb;
    }

    __syncthreads();

    A += (trans == CBlasNoTrans) ? kb : kb * lda;
    k -= kb;
  }

  for (int l = 0; l < k; l++) {
    daxpy(B[0], a[l], x);
    B += ldb;
  }

  // For Upper/NoTrans and Lower/Trans process diagonal last
  if (uplo == CBlasUpper && trans == CBlasNoTrans ||
      uplo == CBlasLower && trans != CBlasNoTrans) {

    __syncthreads();

    int k = min(n - bj, nb);
    while (k > 0) {
      if (trans == CBlasNoTrans) {
#pragma unroll
        for (int j = 0; j < nb; j += by)
          a[threadIdx.x][j + threadIdx.y] =
            (diag != CBlasNonUnit && threadIdx.x == j + threadIdx.y) ? 1.0 : A[j * lda];
      }
      else {
#pragma unroll
        for (int l = 0; l < kb; l += by)
          a[l + threadIdx.y][threadIdx.x] =
            (diag != CBlasNonUnit && threadIdx.x == l + threadIdx.y) ? 1.0 : A[l * lda];
      }

      __syncthreads();

      if (k < kb) break;

// #pragma unroll
//       for (int ll = 0; ll < kb; ll++) {
//         daxpy(nb - ll, B[0], &a[ll][ll], &x[ll]);
        daxpy(8, B[0], &a[ 0][ 0], &x[ 0]); B += ldb;
        daxpy(7, B[0], &a[ 1][ 1], &x[ 1]); B += ldb;
        daxpy(6, B[0], &a[ 2][ 2], &x[ 2]); B += ldb;
        daxpy(5, B[0], &a[ 3][ 3], &x[ 3]); B += ldb;
        daxpy(4, B[0], &a[ 4][ 4], &x[ 4]); B += ldb;
        daxpy(3, B[0], &a[ 5][ 5], &x[ 5]); B += ldb;
        daxpy(2, B[0], &a[ 6][ 6], &x[ 6]); B += ldb;
        daxpy(1, B[0], &a[ 7][ 7], &x[ 7]); B += ldb;
//         B += ldb;
//       }

      __syncthreads();

      A += (trans == CBlasNoTrans) ? kb : kb * lda;
      k -= kb;
    }

//     for (int ll = 0; ll < k; ll++) {
//       daxpy(nb - ll, B[0], &a[ll][ll], &x[ll]);
//       B += ldb;
//     }
    if (k > 0) { daxpy(8, B[0], &a[ 0][ 0], &x[ 0]); B += ldb;
    if (k > 1) { daxpy(7, B[0], &a[ 1][ 1], &x[ 1]); B += ldb;
    if (k > 2) { daxpy(6, B[0], &a[ 2][ 2], &x[ 2]); B += ldb;
    if (k > 3) { daxpy(5, B[0], &a[ 3][ 3], &x[ 3]); B += ldb;
    if (k > 4) { daxpy(4, B[0], &a[ 4][ 4], &x[ 4]); B += ldb;
    if (k > 5) { daxpy(3, B[0], &a[ 5][ 5], &x[ 5]); B += ldb;
    if (k > 6) { daxpy(2, B[0], &a[ 6][ 6], &x[ 6]); B += ldb;
    if (k > 7) { daxpy(1, B[0], &a[ 7][ 7], &x[ 7]); B += ldb; }}}}}}}}
  }

  n -= bj;
  m -= bi + ti;
  if (n <= 0 || m <= 0) return;
  X[0] = alpha * x[ 0]; if ( 1 >= n) return; X += ldx;
  X[0] = alpha * x[ 1]; if ( 2 >= n) return; X += ldx;
  X[0] = alpha * x[ 2]; if ( 3 >= n) return; X += ldx;
  X[0] = alpha * x[ 3]; if ( 4 >= n) return; X += ldx;
  X[0] = alpha * x[ 4]; if ( 5 >= n) return; X += ldx;
  X[0] = alpha * x[ 5]; if ( 6 >= n) return; X += ldx;
  X[0] = alpha * x[ 6]; if ( 7 >= n) return; X += ldx;
  X[0] = alpha * x[ 7];
}

#endif

template void dtrmm2L<CBlasUpper, CBlasNoTrans, CBlasUnit,    64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2L<CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2L<CBlasUpper, CBlasTrans,   CBlasUnit,    32, 16,  8,  8,  8>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2L<CBlasUpper, CBlasTrans,   CBlasNonUnit, 32, 16,  8,  8,  8>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2L<CBlasLower, CBlasNoTrans, CBlasUnit,    64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2L<CBlasLower, CBlasNoTrans, CBlasNonUnit, 64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2L<CBlasLower, CBlasTrans,   CBlasUnit,    32, 16,  8,  8,  8>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2L<CBlasLower, CBlasTrans,   CBlasNonUnit, 32, 16,  8,  8,  8>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasUpper, CBlasNoTrans, CBlasUnit,    64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasUpper, CBlasNoTrans, CBlasNonUnit, 64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasUpper, CBlasTrans,   CBlasUnit,    64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasUpper, CBlasTrans,   CBlasNonUnit, 64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasLower, CBlasNoTrans, CBlasUnit,    64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasLower, CBlasNoTrans, CBlasNonUnit, 64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasLower, CBlasTrans,   CBlasUnit,    64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
template void dtrmm2R<CBlasLower, CBlasTrans,   CBlasNonUnit, 64,  8, 16, 16,  4>(int, int, double, const double * __restrict__, int, const double * __restrict__, int, double * __restrict__, int);
