#include "hip/hip_runtime.h"
#include "blas.h"

#if __CUDA_ARCH__ < 200 && !defined(__BANK_CONFLICT__)

// y(1:8) += alpha * x(1:8)
__device__ void daxpy(double alpha, int * x_hi, int * x_lo, double * y) {
  y[0] += alpha * __hiloint2double(x_hi[0], x_lo[0]);
  y[1] += alpha * __hiloint2double(x_hi[1], x_lo[1]);
  y[2] += alpha * __hiloint2double(x_hi[2], x_lo[2]);
  y[3] += alpha * __hiloint2double(x_hi[3], x_lo[3]);
  y[4] += alpha * __hiloint2double(x_hi[4], x_lo[4]);
  y[5] += alpha * __hiloint2double(x_hi[5], x_lo[5]);
  y[6] += alpha * __hiloint2double(x_hi[6], x_lo[6]);
  y[7] += alpha * __hiloint2double(x_hi[7], x_lo[7]);
}

/**
 * This implementation is out-of-place.  For in-place call with D = C and ldd = ldc.
 *
 * DGEMM:
 *   D := alpha * AB   + beta * C for transA == CBlasNoTrans and transB == CBlasNoTrans
 *   D := alpha * AB'  + beta * C for transA == CBlasNoTrans and transB == CBlasTrans
 *   D := alpha * A'B  + beta * C for transA == CBlasTrans and transB == CBlasNoTrans
 *   D := alpha * A'B' + beta * C for transA == CBlasTrans and transB == CBlasTrans
 *
 * @param transA  transpose for A.
 * @param transB  transpose for B.
 * @param mb      the number of rows in the block of C/D.
 * @param nb      the number of columns in the block of C/D.
 * @param kb      how far to unroll the inner loop.
 * @param bx      blockDim.x.
 * @param by      blockDim.y.
 */
template <CBlasTranspose transA, CBlasTranspose transB,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void dgemm(int m, int n, int k,
                      double alpha, const double * A, int lda, const double * B, int ldb,
                      double beta, const double * C, int ldc, double * D, int ldd) {

  const int bi = blockIdx.x * mb;       // Starting row of block of C/D
  const int bj = blockIdx.y * nb;       // Starting column of block of C/D
  const int ti = threadIdx.y * bx + threadIdx.x;        // Unwrapped thread index [0, bx * by]

  /*
   * Compute our starting points in A, B, C and D.
   *
   * For transA != CBlasNoTrans A is cached in shared memory so the unwrapped
   * thread index can be re-wrapped around mb when calculating D.
   *
   * If transA == CBlasNoTrans then bx * by == mb (checked later on) so there
   * doesn't need to be a separate check for transA == CBlasNoTrans in
   * calculating the start of C/D here.
   */
  A += (transA == CBlasNoTrans) ? bi + ti : (bi + threadIdx.y) * lda + threadIdx.x;
  B += (transB == CBlasNoTrans) ? (bj + threadIdx.y) * ldb + threadIdx.x : threadIdx.y * ldb + bj + threadIdx.x;
  C += (bx * by == mb) ? bj * ldc + bi + ti : (bj + 8 * (ti / mb)) * ldc + bi + ti % mb;
  D += (bx * by == mb) ? bj * ldd + bi + ti : (bj + 8 * (ti / mb)) * ldd + bi + ti % mb;

  /*
   * Blocks of A and B in shared memory and D in registers.
   */
  __shared__ int a_hi[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ int a_lo[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ int b_hi[kb][(transB == CBlasNoTrans) ? nb + 1 : nb];
  __shared__ int b_lo[kb][(transB == CBlasNoTrans) ? nb + 1 : nb];

  double d[] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };

  while (k > 0) {
    // If A is to be transposed cache it in shared memory
    if (transA != CBlasNoTrans) {
//       typedef char x[(kb % bx == 0) ? 1 : -1];  // bx must be a multiple of kb
//       typedef char y[(mb % by == 0) ? 1 : -1];  // by must be a multiple of mb
      // If bx or by is equal to kb or mb then nvcc will optimise one of these
      // loops away.  This is the source of the "warning: expression has no
      // effect" compiler messages.
#pragma unroll
      for (int l = 0; l < kb; l += bx) {
#pragma unroll
        for (int i = 0; i < mb; i += by) {
          a_hi[i + threadIdx.y][l + threadIdx.x] = __double2hiint(A[i * lda + l]);
          a_lo[i + threadIdx.y][l + threadIdx.x] = __double2loint(A[i * lda + l]);
        }
      }
      A += kb;
    }

    // B will always be "transposed" w.r.t. C so must always be cached in shared
    // memory (i.e. it is read along the K or N dimensions when M is the
    // dimension being expanded).
    if (transB == CBlasNoTrans) {
//       typedef char x[(kb % bx == 0) ? 1 : -1];  // bx must be a multiple of kb
//       typedef char y[(nb % by == 0) ? 1 : -1];  // by must be a multiple of nb
#pragma unroll
      for (int l = 0; l < kb; l += bx) {
#pragma unroll
        for (int j = 0; j < nb; j += by) {
          b_hi[l + threadIdx.x][j + threadIdx.y] = __double2hiint(B[j * ldb + l]);
          b_lo[l + threadIdx.x][j + threadIdx.y] = __double2loint(B[j * ldb + l]);
        }
      }
    }
    else {
//       typedef char x[(nb % bx == 0) ? 1 : -1];  // bx must be a multiple of nb
//       typedef char y[(kb % by == 0) ? 1 : -1];  // by must be a multiple of kb
#pragma unroll
      for (int l = 0; l < kb; l += by) {
#pragma unroll
        for (int j = 0; j < nb; j += bx) {
          b_hi[l + threadIdx.y][j + threadIdx.x] = __double2hiint(B[l * ldb + j]);
          b_lo[l + threadIdx.y][j + threadIdx.x] = __double2loint(B[l * ldb + j]);
        }
      }
    }

    __syncthreads();

    if (k < kb) break;

    if (transA == CBlasNoTrans) {
      // Read A straight from global memory.
//       typedef char x[(bx * by == mb) ? 1 : -1]; // There must be mb unrolled threads
//       typedef char y[(nb == 8) ? 1 : -1]; // nb must equal the size of row per thread
#pragma unroll
      for (int l = 0; l < kb; l++) {
        daxpy(A[0], b_hi[l], b_lo[l], d);
        A += lda;
      }
    }
    else {
      // Read A from shared memory.
      // Need to check for thread wrapping so that the correct column of A is
      // matched with the correct row/column of B.
//       typedef char x[(bx * by % mb == 0) ? 1 : -1];     // bx * by must be a multiple of mb
//       typedef char y[((bx * by * 8) / mb == nb) ? 1 : -1];     // when the threads are wrapped around mb they must spread along to nb
#pragma unroll
      for (int l = 0; l < kb; l++)
        daxpy(__hiloint2double(a_hi[(bx * by == mb) ? ti : ti % mb][l],
                               a_lo[(bx * by == mb) ? ti : ti % mb][l]),
              &b_hi[l][(bx * by == mb) ? 0 : 8 * (ti / mb)],
              &b_lo[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], d);
    }

    __syncthreads();

    B += (transB == CBlasNoTrans) ? kb : kb * ldb;
    k -= kb;
  }

  if (transA == CBlasNoTrans) {
    for (int l = 0; l < k; l++) {
      daxpy(A[0], b_hi[l], b_lo[l], d);
      A += lda;
    }
  }
  else {
    for (int l = 0; l < k; l++)
      daxpy(__hiloint2double(a_hi[(bx * by == mb) ? ti : ti % mb][l],
                             a_lo[(bx * by == mb) ? ti : ti % mb][l]),
            &b_hi[l][(bx * by == mb) ? 0 : 8 * (ti / mb)],
            &b_lo[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], d);
  }

  if (bx * by == mb)
    n -= bj;
  else {
    n -= bj + 8 * (ti / mb);
    if (n == 0) return;
  }
  if ((bx * by == mb && bi + ti < m) || (bx * by > mb && bi + ti % mb < m)) {
    if (beta == 0.0) {
      D[0] = alpha * d[0]; if (1 >= n) return; D += ldd;
      D[0] = alpha * d[1]; if (2 >= n) return; D += ldd;
      D[0] = alpha * d[2]; if (3 >= n) return; D += ldd;
      D[0] = alpha * d[3]; if (4 >= n) return; D += ldd;
      D[0] = alpha * d[4]; if (5 >= n) return; D += ldd;
      D[0] = alpha * d[5]; if (6 >= n) return; D += ldd;
      D[0] = alpha * d[6]; if (7 >= n) return; D += ldd;
      D[0] = alpha * d[7];
    }
    else {
      D[0] = alpha * d[0] + beta * C[0]; if (1 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[1] + beta * C[0]; if (2 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[2] + beta * C[0]; if (3 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[3] + beta * C[0]; if (4 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[4] + beta * C[0]; if (5 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[5] + beta * C[0]; if (6 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[6] + beta * C[0]; if (7 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[7] + beta * C[0];
    }
  }
}

#else

// y(1:8) += alpha * x(1:8)
__device__ void daxpy(double alpha, double * x, double * y) {
  y[0] += alpha * x[0]; y[1] += alpha * x[1]; y[2] += alpha * x[2]; y[3] += alpha * x[3];
  y[4] += alpha * x[4]; y[5] += alpha * x[5]; y[6] += alpha * x[6]; y[7] += alpha * x[7];
}

/**
 * This implementation is out-of-place.  For in-place call with D = C and ldd = ldc.
 *
 * DGEMM:
 *   D := alpha * AB   + beta * C for transA == CBlasNoTrans and transB == CBlasNoTrans
 *   D := alpha * AB'  + beta * C for transA == CBlasNoTrans and transB == CBlasTrans
 *   D := alpha * A'B  + beta * C for transA == CBlasTrans and transB == CBlasNoTrans
 *   D := alpha * A'B' + beta * C for transA == CBlasTrans and transB == CBlasTrans
 *
 * @param transA  transpose for A.
 * @param transB  transpose for B.
 * @param mb      the number of rows in the block of C/D.
 * @param nb      the number of columns in the block of C/D.
 * @param kb      how far to unroll the inner loop.
 * @param bx      blockDim.x.
 * @param by      blockDim.y.
 */
template <CBlasTranspose transA, CBlasTranspose transB,
          unsigned int mb, unsigned int nb, unsigned int kb,
          unsigned int bx, unsigned int by>
__global__ void dgemm(int m, int n, int k,
                      double alpha, const double * A, int lda, const double * B, int ldb,
                      double beta, const double * C, int ldc, double * D, int ldd) {

  const int bi = blockIdx.x * mb;       // Starting row of block of C/D
  const int bj = blockIdx.y * nb;       // Starting column of block of C/D
  const int ti = threadIdx.y * bx + threadIdx.x;        // Unwrapped thread index [0, bx * by]

  /*
   * Compute our starting points in A, B, C and D.
   *
   * For transA != CBlasNoTrans A is cached in shared memory so the unwrapped
   * thread index can be re-wrapped around mb when calculating D.
   *
   * If transA == CBlasNoTrans then bx * by == mb (checked later on) so there
   * doesn't need to be a separate check for transA == CBlasNoTrans in
   * calculating the start of C/D here.
   */
  A += (transA == CBlasNoTrans) ? bi + ti : (bi + threadIdx.y) * lda + threadIdx.x;
  B += (transB == CBlasNoTrans) ? (bj + threadIdx.y) * ldb + threadIdx.x : threadIdx.y * ldb + bj + threadIdx.x;
  C += (bx * by == mb) ? bj * ldc + bi + ti : (bj + 8 * (ti / mb)) * ldc + bi + ti % mb;
  D += (bx * by == mb) ? bj * ldd + bi + ti : (bj + 8 * (ti / mb)) * ldd + bi + ti % mb;

  /*
   * Blocks of A and B in shared memory and D in registers.
   */
  __shared__ double a[mb][kb + 1];       // Optimised away when transA == CBlasNoTrans
  __shared__ double b[kb][(transB == CBlasNoTrans) ? nb + 1 : nb];

  double d[] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };

  while (k > 0) {
    // If A is to be transposed cache it in shared memory
    if (transA != CBlasNoTrans) {
//       typedef char x[(kb % bx == 0) ? 1 : -1];  // bx must be a multiple of kb
//       typedef char y[(mb % by == 0) ? 1 : -1];  // by must be a multiple of mb
      // If bx or by is equal to kb or mb then nvcc will optimise one of these
      // loops away.  This is the source of the "warning: expression has no
      // effect" compiler messages.
#pragma unroll
      for (int l = 0; l < kb; l += bx) {
#pragma unroll
        for (int i = 0; i < mb; i += by)
          a[i + threadIdx.y][l + threadIdx.x] = A[i * lda + l];
      }
      A += kb;
    }

    // B will always be "transposed" w.r.t. C so must always be cached in shared
    // memory (i.e. it is read along the K or N dimensions when M is the
    // dimension being expanded).
    if (transB == CBlasNoTrans) {
//       typedef char x[(kb % bx == 0) ? 1 : -1];  // bx must be a multiple of kb
//       typedef char y[(nb % by == 0) ? 1 : -1];  // by must be a multiple of nb
#pragma unroll
      for (int l = 0; l < kb; l += bx) {
#pragma unroll
        for (int j = 0; j < nb; j += by)
          b[l + threadIdx.x][j + threadIdx.y] = B[j * ldb + l];
      }
    }
    else {
//       typedef char x[(nb % bx == 0) ? 1 : -1];  // bx must be a multiple of nb
//       typedef char y[(kb % by == 0) ? 1 : -1];  // by must be a multiple of kb
#pragma unroll
      for (int l = 0; l < kb; l += by) {
#pragma unroll
        for (int j = 0; j < nb; j += bx)
          b[l + threadIdx.y][j + threadIdx.x] = B[l * ldb + j];
      }
    }

    __syncthreads();

    if (k < kb) break;

    if (transA == CBlasNoTrans) {
      // Read A straight from global memory.
//       typedef char x[(bx * by == mb) ? 1 : -1]; // There must be mb unrolled threads
//       typedef char y[(nb == 8) ? 1 : -1]; // nb must equal the size of row per thread
#pragma unroll
      for (int l = 0; l < kb; l++) {
        daxpy(A[0], b[l], d);
        A += lda;
      }
    }
    else {
      // Read A from shared memory.
      // Need to check for thread wrapping so that the correct column of A is
      // matched with the correct row/column of B.
//       typedef char x[(bx * by % mb == 0) ? 1 : -1];     // bx * by must be a multiple of mb
//       typedef char y[((bx * by * 8) / mb == nb) ? 1 : -1];     // when the threads are wrapped around mb they must spread along to nb
#pragma unroll
      for (int l = 0; l < kb; l++)
        daxpy(a[(bx * by == mb) ? ti : ti % mb][l],
              &b[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], d);
    }

    __syncthreads();

    B += (transB == CBlasNoTrans) ? kb : kb * ldb;
    k -= kb;
  }

  if (transA == CBlasNoTrans) {
    for (int l = 0; l < k; l++) {
      daxpy(A[0], b[l], d);
      A += lda;
    }
  }
  else {
    for (int l = 0; l < k; l++)
      daxpy(a[(bx * by == mb) ? ti : ti % mb][l],
            &b[l][(bx * by == mb) ? 0 : 8 * (ti / mb)], d);
  }

  if (bx * by == mb)
    n -= bj;
  else {
    n -= bj + 8 * (ti / mb);
    if (n == 0) return;
  }
  if ((bx * by == mb && bi + ti < m) || (bx * by > mb && bi + ti % mb < m)) {
    if (beta == 0.0) {
      D[0] = alpha * d[0]; if (1 >= n) return; D += ldd;
      D[0] = alpha * d[1]; if (2 >= n) return; D += ldd;
      D[0] = alpha * d[2]; if (3 >= n) return; D += ldd;
      D[0] = alpha * d[3]; if (4 >= n) return; D += ldd;
      D[0] = alpha * d[4]; if (5 >= n) return; D += ldd;
      D[0] = alpha * d[5]; if (6 >= n) return; D += ldd;
      D[0] = alpha * d[6]; if (7 >= n) return; D += ldd;
      D[0] = alpha * d[7];
    }
    else {
      D[0] = alpha * d[0] + beta * C[0]; if (1 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[1] + beta * C[0]; if (2 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[2] + beta * C[0]; if (3 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[3] + beta * C[0]; if (4 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[4] + beta * C[0]; if (5 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[5] + beta * C[0]; if (6 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[6] + beta * C[0]; if (7 >= n) return; C += ldc; D += ldd;
      D[0] = alpha * d[7] + beta * C[0];
    }
  }
}

#endif

/**
 * For D = aAB + bC:
 *   mb must be a multiple of the warp size (32) and less than or equal to the
 *        maximum number of threads per block (512).
 *   nb must be less than or equal to 20 (registers start spilling to global
 *        memory after 20).
 *   kb must be a multiple of the half-warp size (16) and such that
 *        (nb + 1)*kb*sizeof(float) is less than the amount of shared memory
 *        available per block (16384 bytes).
 *
 * mb and nb must be selected such that the bandwidth reduction is greater than
 * the flop:word ratio of the GPU.  The bandwidth reduction for all valid values
 * of mb and nb can be calculated with the following loop (bash):
 * echo -n " mb\nb"; for nb in {1..20}; do printf "%6d" ${nb}; done; echo; for mb in {32..512..32}; do printf "%6d"  ${mb}; for nb in {1..20}; do printf "%6.2f" $(echo 2 / \(1/${mb} + 1/${nb}\) | bc -l); done; echo; done
 *
 * Sample output:
 *  mb\nb     1     2     3     4     5     6     7     8     9    10    11    12    13    14    15    16    17    18    19    20
 *     32  1.94  3.76  5.49  7.11  8.65 10.11 11.49 12.80 14.05 15.24 16.37 17.45 18.49 19.48 20.43 21.33 22.20 23.04 23.84 24.62
 *     64  1.97  3.88  5.73  7.53  9.28 10.97 12.62 14.22 15.78 17.30 18.77 20.21 21.61 22.97 24.30 25.60 26.86 28.10 29.30 30.48
 *     96  1.98  3.92  5.82  7.68  9.50 11.29 13.05 14.77 16.46 18.11 19.74 21.33 22.90 24.44 25.95 27.43 28.88 30.32 31.72 33.10
 *    128  1.98  3.94  5.86  7.76  9.62 11.46 13.27 15.06 16.82 18.55 20.26 21.94 23.60 25.24 26.85 28.44 30.01 31.56 33.09 34.59
 *    160  1.99  3.95  5.89  7.80  9.70 11.57 13.41 15.24 17.04 18.82 20.58 22.33 24.05 25.75 27.43 29.09 30.73 32.36 33.97 35.56
 *    192  1.99  3.96  5.91  7.84  9.75 11.64 13.51 15.36 17.19 19.01 20.81 22.59 24.35 26.10 27.83 29.54 31.23 32.91 34.58 36.23
 *    224  1.99  3.96  5.92  7.86  9.78 11.69 13.58 15.45 17.30 19.15 20.97 22.78 24.57 26.35 28.12 29.87 31.60 33.32 35.03 36.72
 *    256  1.99  3.97  5.93  7.88  9.81 11.73 13.63 15.52 17.39 19.25 21.09 22.93 24.74 26.55 28.34 30.12 31.88 33.64 35.37 37.10
 *    288  1.99  3.97  5.94  7.89  9.83 11.76 13.67 15.57 17.45 19.33 21.19 23.04 24.88 26.70 28.51 30.32 32.10 33.88 35.65 37.40
 *    320  1.99  3.98  5.94  7.90  9.85 11.78 13.70 15.61 17.51 19.39 21.27 23.13 24.98 26.83 28.66 30.48 32.28 34.08 35.87 37.65
 *    352  1.99  3.98  5.95  7.91  9.86 11.80 13.73 15.64 17.55 19.45 21.33 23.21 25.07 26.93 28.77 30.61 32.43 34.25 36.05 37.85
 *    384  1.99  3.98  5.95  7.92  9.87 11.82 13.75 15.67 17.59 19.49 21.39 23.27 25.15 27.02 28.87 30.72 32.56 34.39 36.21 38.02
 *    416  2.00  3.98  5.96  7.92  9.88 11.83 13.77 15.70 17.62 19.53 21.43 23.33 25.21 27.09 28.96 30.81 32.67 34.51 36.34 38.17
 *    448  2.00  3.98  5.96  7.93  9.89 11.84 13.78 15.72 17.65 19.56 21.47 23.37 25.27 27.15 29.03 30.90 32.76 34.61 36.45 38.29
 *    480  2.00  3.98  5.96  7.93  9.90 11.85 13.80 15.74 17.67 19.59 21.51 23.41 25.31 27.21 29.09 30.97 32.84 34.70 36.55 38.40
 *    512  2.00  3.98  5.97  7.94  9.90 11.86 13.81 15.75 17.69 19.62 21.54 23.45 25.36 27.25 29.15 31.03 32.91 34.78 36.64 38.50
 *
 * The number of registers per block is mb*32 (compiled with -maxrregcount=32).
 * More threads == better performance (from flop-test) therefore mb is chosen to
 * be the largest number of threads such that the number of blocks per
 * multiprocessor is still limited by the register usage.
 * kb is chosen to be the largest multiple of 16 such that the number of blocks
 * per multiprocessor is limited by the register usage.
 */
template void dgemm<CBlasNoTrans, CBlasNoTrans, 64,  8, 16, 16,  4>(int, int, int, double, const double *, int, const double *, int, double, const double *, int, double *, int);
template void dgemm<CBlasNoTrans, CBlasTrans,   64,  8, 16,  8,  8>(int, int, int, double, const double *, int, const double *, int, double, const double *, int, double *, int);
template void dgemm<CBlasTrans,   CBlasNoTrans, 32, 16,  8,  8,  8>(int, int, int, double, const double *, int, const double *, int, double, const double *, int, double *, int);
template void dgemm<CBlasTrans,   CBlasTrans,   32, 16,  8,  8,  8>(int, int, int, double, const double *, int, const double *, int, double, const double *, int, double *, int);
