#include "../hipComplex.cuh"
#include "vector.cu"

template void           all<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double>, size_t);
template void         basis<hipComplex<double> >(hipComplex<double> *, size_t, size_t, size_t);
template void           add<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double> *, size_t, size_t);
template void      addConst<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double>, size_t);
template void      subtract<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double> *, size_t, size_t);
template void subtractConst<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double>, size_t);
template void      multiply<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double> *, size_t, size_t);
template void multiplyConst<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double>, size_t);
template void        divide<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double> *, size_t, size_t);
template void   divideConst<hipComplex<double> >(hipComplex<double> *, size_t, hipComplex<double>, size_t);
