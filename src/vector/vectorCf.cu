#include "../hipComplex.cuh"
#include "vector.cu"

template void           all<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t);
template void         basis<hipComplex<float> >(hipComplex<float> *, size_t, size_t, size_t);
template void           add<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t);
template void      addConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t);
template void      subtract<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t);
template void subtractConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t);
template void      multiply<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t);
template void multiplyConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t);
template void        divide<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float> *, size_t, size_t);
template void   divideConst<hipComplex<float> >(hipComplex<float> *, size_t, hipComplex<float>, size_t);
