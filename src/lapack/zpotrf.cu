#include "hip/hip_runtime.h"
// nvcc -I../../include -O2 -arch=compute_13 -code=sm_13 -use_fast_math -Xptxas=-v -maxrregcount=32 -cubin zpotrf.cu
#include "blas.h"
#include <hip/hip_complex.h>

template <unsigned int bs>
__device__ hipDoubleComplex zdotc(int ti, int n, const hipDoubleComplex * x, const hipDoubleComplex * y) {
  __shared__ double temp_real[bs], temp_imag[bs];

  hipDoubleComplex res = make_hipDoubleComplex(0.0, 0.0);

  for (int i = ti; i < n; i += bs * 2) {
    res = hipCfma(hipConj(x[i]), y[i], res);
    if (i + bs < n)
      res = hipCfma(hipConj(x[i + bs]), y[i + bs], res);
  }

  temp_real[ti] = hipCreal(res);
  temp_imag[ti] = hipCimag(res);
  __syncthreads();

  if (bs >= 512) { if (ti < 256) res = make_hipDoubleComplex(temp_real[ti] = hipCreal(res) + temp_real[ti + 256], temp_imag[ti] = hipCimag(res) + temp_imag[ti + 256]); __syncthreads(); }
  if (bs >= 256) { if (ti < 128) res = make_hipDoubleComplex(temp_real[ti] = hipCreal(res) + temp_real[ti + 128], temp_imag[ti] = hipCimag(res) + temp_imag[ti + 128]); __syncthreads(); }
  if (bs >= 128) { if (ti <  64) res = make_hipDoubleComplex(temp_real[ti] = hipCreal(res) + temp_real[ti +  64], temp_imag[ti] = hipCimag(res) + temp_imag[ti +  64]); __syncthreads(); }

  if (ti < 32) {
    volatile double * vtemp_real = temp_real;
    volatile double * vtemp_imag = temp_imag;
    if (bs >= 64) { res = make_hipDoubleComplex(vtemp_real[ti] = hipCreal(res) + vtemp_real[ti + 32], vtemp_imag[ti] = hipCimag(res) + vtemp_imag[ti + 32]); }
    if (bs >= 32) { res = make_hipDoubleComplex(vtemp_real[ti] = hipCreal(res) + vtemp_real[ti + 16], vtemp_imag[ti] = hipCimag(res) + vtemp_imag[ti + 16]); }
    if (bs >= 16) { res = make_hipDoubleComplex(vtemp_real[ti] = hipCreal(res) + vtemp_real[ti +  8], vtemp_imag[ti] = hipCimag(res) + vtemp_imag[ti +  8]); }
    if (bs >=  8) { res = make_hipDoubleComplex(vtemp_real[ti] = hipCreal(res) + vtemp_real[ti +  4], vtemp_imag[ti] = hipCimag(res) + vtemp_imag[ti +  4]); }
    if (bs >=  4) { res = make_hipDoubleComplex(vtemp_real[ti] = hipCreal(res) + vtemp_real[ti +  2], vtemp_imag[ti] = hipCimag(res) + vtemp_imag[ti +  2]); }
    if (bs >=  2) { res = make_hipDoubleComplex(vtemp_real[ti] = hipCreal(res) + vtemp_real[ti +  1], vtemp_imag[ti] = hipCimag(res) + vtemp_imag[ti +  1]); }
  }

  return res;
}

template <CBlasUplo uplo, unsigned int bx, unsigned int by>
__global__ void zpotf2(int n, hipDoubleComplex * A, int lda, int * info) {
  const int ti = threadIdx.y * bx + threadIdx.x;

  __shared__ int s_info;
  if (ti == 0)
    s_info = 0;

  if (uplo == CBlasUpper) {
    for (int i = 0; i < n; i++) {
      hipDoubleComplex temp = zdotc<bx * by>(ti, i, &A[i * lda], &A[i * lda]);

      double aii;
      if (ti == 0) {
        aii = hipCreal(A[i * lda + i]) - hipCreal(temp);
        if (aii <= 0.0 || isnan(aii)) {
          A[i * lda + i] = temp;
          *info = s_info = i;
        }
        else
          A[i * lda + i] = make_hipDoubleComplex(aii = sqrt(aii), 0.0);
      }

      __syncthreads();

      if (s_info != 0)
        return;

      for (int j = i + 1; j < n; j++) {
        temp = zdotc<bx * by>(ti, i, &A[i * lda], &A[j * lda]);
        if (ti == 0)
          A[j * lda + i] = make_hipDoubleComplex((hipCreal(A[j * lda + i]) - hipCreal(temp)) / aii, (hipCimag(A[j * lda + i]) - hipCimag(temp)) / aii);
      }

      __syncthreads();
    }
  }
  else {
    __shared__ double ajj;
    for (int j = 0; j < n; j++) {
      if (j + ti < n) {
        hipDoubleComplex temp = A[j * lda + j + ti];
        for (int k = 0; k < j; k++)
          temp = hipCsub(temp, hipCmul(hipConj(A[k * lda + j]), A[k * lda + j + ti]));

        if (ti == 0) {
          if (hipCreal(temp) <= 0.0 || isnan(hipCreal(temp))) {
            A[j * lda + j] = temp;
            *info = s_info = j;
          }
          else
            A[j * lda + j] = make_hipDoubleComplex(ajj = sqrt(hipCreal(temp)), 0.0);
        }

        __syncthreads();

        if (s_info != 0)
          return;

        if (ti > 0)
          A[j * lda + j + ti] = make_hipDoubleComplex(hipCreal(temp) / ajj, hipCimag(temp) / ajj);
      }

      for (int i = j + bx * by + ti; i < n; i += bx * by) {
        hipDoubleComplex temp = A[j * lda + i];
        for (int k = 0; k < j; k++)
          temp = hipCsub(temp, hipCmul(hipConj(A[k * lda + j]), A[k * lda + i]));
        A[j * lda + i] = make_hipDoubleComplex(hipCreal(temp) / ajj, hipCimag(temp) / ajj);
      }

      __syncthreads();
    }
  }
}

template void zpotf2<CBlasUpper,  8, 8>(int, hipDoubleComplex *, int, int *);
template void zpotf2<CBlasLower, 16, 4>(int, hipDoubleComplex *, int, int *);
